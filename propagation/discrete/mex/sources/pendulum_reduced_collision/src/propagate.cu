#include "hip/hip_runtime.h"
#include "propagate.cuh"

#include <math.h>
#include <stdio.h>
#include "omp.h"

void get_df(myReal* df, const myReal* f, const myReal* lambda, myReal* const* fcL, const int numR, const int* indR, int* const* lambda_indx, const int* lambda_numx, int* const* fcL_indx1, const int* fcL_numx1, int* const* fcL_indx2, int* const* fcL_numx2, const Size_f* size_f)
{
    // number of threads
    int nthread = 32;
    
    hipStream_t cudaStreams[nthread];
    for (int i = 0; i < nthread; i++) {
        cudaErrorHandle(hipStreamCreate(&cudaStreams[i]));
    }

    hipblasHandle_t cublasHandles[nthread];
    for (int i = 0; i < nthread; i++) {
        cublasErrorHandle(hipblasCreate(&cublasHandles[i]));
    }

    // maximum memory needed
    int max_numx2 = 0;
    for (int iR = 0; iR < numR; iR++) {
        max_numx2 = (max_numx2 < fcL_numx2[iR][fcL_numx1[iR]]) ? fcL_numx2[iR][fcL_numx1[iR]] : max_numx2;
    }

    // density in
    myReal* fcL_dev;
    cudaErrorHandle(hipMalloc(&fcL_dev, max_numx2*sizeof(myReal)));

    int* fcL_indx2_dev;
    cudaErrorHandle(hipMalloc(&fcL_indx2_dev, max_numx2*sizeof(int)));

    myReal* f_dev;
    cudaErrorHandle(hipMalloc(&f_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(f_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));

    Size_f* size_f_dev;
    cudaErrorHandle(hipMalloc(&size_f_dev, sizeof(Size_f)));
    cudaErrorHandle(hipMemcpy(size_f_dev, size_f, sizeof(Size_f), hipMemcpyHostToDevice));

    myReal* f_temp_dev;
    cudaErrorHandle(hipMalloc(&f_temp_dev, max_numx2*sizeof(myReal)));

    myReal* df_dev;
    cudaErrorHandle(hipMalloc(&df_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemset(df_dev, 0, size_f->nTot*sizeof(myReal)));

    hipblasHandle_t handle_cublas;
    hipblasCreate(&handle_cublas);

    for (int iR = 0; iR < numR; iR++) {
        int numx2 = fcL_numx2[iR][fcL_numx1[iR]];
        cudaErrorHandle(hipMemcpy(fcL_dev, fcL[iR], numx2*sizeof(myReal), hipMemcpyHostToDevice));
        cudaErrorHandle(hipMemcpy(fcL_indx2_dev, fcL_indx2[iR], numx2*sizeof(int), hipMemcpyHostToDevice));

        get_fold <<<(int)numx2/128+1, 128>>> (f_temp_dev, f_dev+indR[iR], fcL_indx2_dev, numx2, size_f_dev);

        #pragma omp parallel for num_threads(nthread)
        for (int ix1 = 0; ix1 < fcL_numx1[iR]; ix1++) {
            int tid = omp_get_thread_num();
            cublasErrorHandle(hipblasSetStream(cublasHandles[tid], cudaStreams[tid]));

            int n = fcL_numx2[iR][ix1+1] - fcL_numx2[iR][ix1];
            cublasErrorHandle(mycublasdot(cublasHandles[tid], n, fcL_dev+fcL_numx2[iR][ix1], 1, f_temp_dev+fcL_numx2[iR][ix1], 1, df_dev+indR[iR]+size_f->nR*fcL_indx1[iR][ix1]));
        }

        printf("No. %d finished, total: %d\n", iR, numR);
    }

    // density out
    int* lambda_indx_dev;
    cudaErrorHandle(hipMalloc(&lambda_indx_dev, size_f->nx*sizeof(int)));

    for (int iR = 0; iR < numR; iR++) {
        cudaErrorHandle(hipMemcpy(lambda_indx_dev, lambda_indx[iR], lambda_numx[iR]*sizeof(int), hipMemcpyHostToDevice));
        get_fout <<<(int)lambda_numx[iR]/128+1, 128>>> (df_dev+indR[iR], f_dev+indR[iR], lambda[iR], lambda_indx_dev, lambda_numx[iR], size_f_dev);
    }

    cudaErrorHandle(hipMemcpy(df, df_dev, size_f->nTot*sizeof(myReal), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(fcL_dev));
    cudaErrorHandle(hipFree(fcL_indx2_dev));
    cudaErrorHandle(hipFree(f_dev));
    cudaErrorHandle(hipFree(size_f_dev));
    cudaErrorHandle(hipFree(f_temp_dev));
    cudaErrorHandle(hipFree(df_dev));
    cudaErrorHandle(hipFree(lambda_indx_dev));

    for (int i = 0; i < nthread; i++) {
        cublasErrorHandle(hipblasDestroy(cublasHandles[i]));
    }
}

__global__ void get_fold(myReal* f_temp, const myReal* f, const int* fcL_indx2, const int fcL_numx2, const Size_f* size_f)
{
    int ind_temp = threadIdx.x + blockIdx.x*blockDim.x;

    if (ind_temp < fcL_numx2) {
        int indf = fcL_indx2[ind_temp]*size_f->nR;
        f_temp[ind_temp] = f[indf];
    }
}

__global__ void get_fout(myReal* df, const myReal* f, const myReal lambda, const int* lambda_indx, const int lambda_numx, const Size_f* size_f)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    if (indx < lambda_numx) {
        int indf = lambda_indx[indx]*size_f->nR;
        df[indf] = df[indf] - f[indf]*lambda;
    }
}


