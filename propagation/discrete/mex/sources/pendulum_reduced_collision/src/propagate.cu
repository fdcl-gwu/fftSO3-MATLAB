#include "hip/hip_runtime.h"
#include "propagate.cuh"

#include <math.h>
#include <stdio.h>
#include "omp.h"

void get_df_noise(myReal* df, const myReal* f, const myReal* lambda, myReal* const* fcL, const int numR, const int* indR, int* const* lambda_indx, const int* lambda_numx, int* const* fcL_indx1, const int* fcL_numx1, int* const* fcL_indx2, int* const* fcL_numx2, const Size_f* size_f)
{
    // number of threads
    int nthread = size_f->nx;
    hipStream_t cudaStreams[nthread];
    for (int i = 0; i < nthread; i++) {
        cudaErrorHandle(hipStreamCreate(&cudaStreams[i]));
    }

    // maximum memory needed
    int max_numx2 = 0;
    for (int iR = 0; iR < numR; iR++) {
        max_numx2 = (max_numx2 < fcL_numx2[iR][fcL_numx1[iR]]) ? fcL_numx2[iR][fcL_numx1[iR]] : max_numx2;
    }

    // density in
    myReal* fcL_dev;
    cudaErrorHandle(hipMalloc(&fcL_dev, max_numx2*sizeof(myReal)));

    int* fcL_indx2_dev;
    cudaErrorHandle(hipMalloc(&fcL_indx2_dev, max_numx2*sizeof(int)));

    myReal* f_dev;
    cudaErrorHandle(hipMalloc(&f_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(f_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));

    Size_f* size_f_dev;
    cudaErrorHandle(hipMalloc(&size_f_dev, sizeof(Size_f)));
    cudaErrorHandle(hipMemcpy(size_f_dev, size_f, sizeof(Size_f), hipMemcpyHostToDevice));

    myReal* f_temp_dev;
    cudaErrorHandle(hipMalloc(&f_temp_dev, max_numx2*sizeof(myReal)));

    myReal* df_dev;
    cudaErrorHandle(hipMalloc(&df_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemset(df_dev, 0, size_f->nTot*sizeof(myReal)));

    hipblasHandle_t handle_cublas;
    hipblasCreate(&handle_cublas);

    for (int iR = 0; iR < numR; iR++) {
        int numx2 = fcL_numx2[iR][fcL_numx1[iR]];
        cudaErrorHandle(hipMemcpy(fcL_dev, fcL[iR], numx2*sizeof(myReal), hipMemcpyHostToDevice));
        cudaErrorHandle(hipMemcpy(fcL_indx2_dev, fcL_indx2[iR], numx2*sizeof(int), hipMemcpyHostToDevice));

        get_fold_noise <<<(int)numx2/128+1, 128>>> (f_temp_dev, f_dev+indR[iR], fcL_indx2_dev, numx2, size_f_dev);

        for (int ix1 = 0; ix1 < fcL_numx1[iR]; ix1++) {
            int n = fcL_numx2[iR][ix1+1] - fcL_numx2[iR][ix1];

            cublasErrorHandle(hipblasSetStream(handle_cublas, cudaStreams[ix1]));
            cublasErrorHandle(mycublasdot(handle_cublas, n, fcL_dev+fcL_numx2[iR][ix1], 1, f_temp_dev+fcL_numx2[iR][ix1], 1, df_dev+indR[iR]+size_f->nR*fcL_indx1[iR][ix1]));
        }

        printf("No. %d finished, total: %d\n", iR, numR);
    }

    // density out
    int* lambda_indx_dev;
    cudaErrorHandle(hipMalloc(&lambda_indx_dev, size_f->nx*sizeof(int)));

    for (int iR = 0; iR < numR; iR++) {
        cudaErrorHandle(hipMemcpy(lambda_indx_dev, lambda_indx[iR], lambda_numx[iR]*sizeof(int), hipMemcpyHostToDevice));
        get_fout <<<(int)lambda_numx[iR]/128+1, 128>>> (df_dev+indR[iR], f_dev+indR[iR], lambda[iR], lambda_indx_dev, lambda_numx[iR], size_f_dev);
    }

    cudaErrorHandle(hipMemcpy(df, df_dev, size_f->nTot*sizeof(myReal), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(fcL_dev));
    cudaErrorHandle(hipFree(fcL_indx2_dev));
    cudaErrorHandle(hipFree(f_dev));
    cudaErrorHandle(hipFree(size_f_dev));
    cudaErrorHandle(hipFree(f_temp_dev));
    cudaErrorHandle(hipFree(df_dev));
    cudaErrorHandle(hipFree(lambda_indx_dev));

    cublasErrorHandle(hipblasDestroy(handle_cublas));
}

void get_df_nonoise(myReal* df, const myReal* f, const myReal* lambda, const int numR, const int* indR, int* const* lambda_indx, const int* lambda_numx, const int* ind_interp, const myReal* coeff_interp, const Size_f* size_f)
{
    // compute fin
    dim3 blocksize_n0Rx(size_f->const_2Bx, 1, 1);
    dim3 gridsize_n0Rx(size_f->const_2Bx, numR, 1);

    myReal* f_dev;
    cudaErrorHandle(hipMalloc(&f_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(f_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));

    myReal* lambda_dev;
    cudaErrorHandle(hipMalloc(&lambda_dev, numR*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(lambda_dev, lambda, numR*sizeof(myReal), hipMemcpyHostToDevice));

    int* indR_dev;
    cudaErrorHandle(hipMalloc(&indR_dev, numR*sizeof(int)));
    cudaErrorHandle(hipMemcpy(indR_dev, indR, numR*sizeof(int), hipMemcpyHostToDevice));

    int* ind_interp_dev;
    cudaErrorHandle(hipMalloc(&ind_interp_dev, 4*numR*size_f->nx*sizeof(int)));
    cudaErrorHandle(hipMemcpy(ind_interp_dev, ind_interp, 4*numR*size_f->nx*sizeof(int), hipMemcpyHostToDevice));
    
    myReal* coeff_interp_dev;
    cudaErrorHandle(hipMalloc(&coeff_interp_dev, 4*numR*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(coeff_interp_dev, coeff_interp, 4*numR*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
    
    Size_f* size_f_dev;
    cudaErrorHandle(hipMalloc(&size_f_dev, sizeof(Size_f)));
    cudaErrorHandle(hipMemcpy(size_f_dev, size_f, sizeof(Size_f), hipMemcpyHostToDevice));

    myReal* df_dev;
    cudaErrorHandle(hipMalloc(&df_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemset(df_dev, 0, size_f->nTot*sizeof(myReal)));

    get_fin_nonoise <<<gridsize_n0Rx, blocksize_n0Rx>>> (df_dev, f_dev, lambda_dev, indR_dev, ind_interp_dev, coeff_interp_dev, size_f_dev);    

    // compute fout
    int* lambda_indx_dev;
    cudaErrorHandle(hipMalloc(&lambda_indx_dev, size_f->nx*sizeof(int)));

    for (int iR = 0; iR < numR; iR++) {
        cudaErrorHandle(hipMemcpy(lambda_indx_dev, lambda_indx[iR], lambda_numx[iR]*sizeof(int), hipMemcpyHostToDevice));
        get_fout <<<(int)lambda_numx[iR]/128+1, 128>>> (df_dev+indR[iR], f_dev+indR[iR], lambda[iR], lambda_indx_dev, lambda_numx[iR], size_f_dev);
    }

    cudaErrorHandle(hipMemcpy(df, df_dev, size_f->nTot*sizeof(myReal), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(f_dev));
    cudaErrorHandle(hipFree(lambda_dev));
    cudaErrorHandle(hipFree(indR_dev));
    cudaErrorHandle(hipFree(ind_interp_dev));
    cudaErrorHandle(hipFree(coeff_interp_dev));
    cudaErrorHandle(hipFree(size_f_dev));
    cudaErrorHandle(hipFree(df_dev));
    cudaErrorHandle(hipFree(lambda_indx_dev));
}

__global__ void get_fold_noise(myReal* f_temp, const myReal* f, const int* fcL_indx2, const int fcL_numx2, const Size_f* size_f)
{
    int ind_temp = threadIdx.x + blockIdx.x*blockDim.x;

    if (ind_temp < fcL_numx2) {
        int indf = fcL_indx2[ind_temp]*size_f->nR;
        f_temp[ind_temp] = f[indf];
    }
}

__global__ void get_fin_nonoise(myReal* df, const myReal* f, const myReal* lambda, const int* indR, const int* ind_interp, const myReal* coeff_interp, const Size_f* size_f)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    int indfR = indR[blockIdx.y];
    int indf = indfR + indx*size_f->nR;
    int indInterp = 4*(blockIdx.y + indx*gridDim.y);

    if (isnan(coeff_interp[indInterp])) {
        df[indf] = 0;
    } else {
        int indf_interp[4];
        for (int i = 0; i < 4; i++) {
            indf_interp[i] = indfR + ind_interp[indInterp+i]*size_f->nR;
        }

        myReal f_interp = 0.0;
        for (int i = 0; i < 4; i++) {
            f_interp += f[indf_interp[i]]*coeff_interp[indInterp+i];
        }

        df[indf] = f_interp*lambda[blockIdx.y];
    }
}

__global__ void get_fout(myReal* df, const myReal* f, const myReal lambda, const int* lambda_indx, const int lambda_numx, const Size_f* size_f)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    if (indx < lambda_numx) {
        int indf = lambda_indx[indx]*size_f->nR;
        df[indf] = df[indf] - f[indf]*lambda;
    }
}


