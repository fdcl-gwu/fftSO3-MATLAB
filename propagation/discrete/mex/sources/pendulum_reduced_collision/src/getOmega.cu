#include "hip/hip_runtime.h"
#include "getOmega.cuh"

#include <stdio.h>

void getOmega(myReal* Omega, const myReal* R, const myReal* x, const int* ind_n0, const int nn0, const myReal* epsilon, const Size_f* size_f)
{
    // get t
    myReal* R_dev;
    cudaErrorHandle(hipMalloc(&R_dev, 9*size_f->nR*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(R_dev, R, 9*size_f->nR*sizeof(myReal), hipMemcpyHostToDevice));

    Size_f* size_f_dev;
    cudaErrorHandle(hipMalloc(&size_f_dev, sizeof(Size_f)));
    cudaErrorHandle(hipMemcpy(size_f_dev, size_f, sizeof(Size_f), hipMemcpyHostToDevice));

    myReal* t_dev;
    cudaErrorHandle(hipMalloc(&t_dev, 2*size_f->nR*sizeof(myReal)));

    dim3 blocksize_R(size_f->const_2BR, 1, 1);
    dim3 gridsize_R(size_f->const_2BRs, 1, 1);

    getT <<<gridsize_R, blocksize_R>>> (t_dev, R_dev, size_f_dev);

    // get Omega
    myReal* x_dev;
    cudaErrorHandle(hipMalloc(&x_dev, 2*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(x_dev, x, 2*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));

    int* ind_n0_dev;
    cudaErrorHandle(hipMalloc(&ind_n0_dev, nn0*sizeof(int)));
    cudaErrorHandle(hipMemcpy(ind_n0_dev, ind_n0, nn0*sizeof(int), hipMemcpyHostToDevice));

    myReal* Omega_dev;
    cudaErrorHandle(hipMalloc(&Omega_dev, 2*nn0*size_f->nx*sizeof(myReal)));

    dim3 blocksize_n0Rx(size_f->const_2Bx, 1, 1);
    dim3 gridsize_n0Rx(size_f->const_2Bx, nn0, 1);

    compute_Omega <<<gridsize_n0Rx, blocksize_n0Rx>>> (Omega_dev, R_dev, x_dev, t_dev, ind_n0_dev, *epsilon, size_f_dev);

    cudaErrorHandle(hipMemcpy(Omega, Omega_dev, 2*nn0*size_f->nx*sizeof(myReal), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(R_dev));
    cudaErrorHandle(hipFree(size_f_dev));
    cudaErrorHandle(hipFree(t_dev));
    cudaErrorHandle(hipFree(x_dev));
    cudaErrorHandle(hipFree(ind_n0_dev));
    cudaErrorHandle(hipFree(Omega_dev));
}

__global__ void getT(myReal* t, const myReal* R, const Size_f* size_f)
{
    int indR = threadIdx.x + blockIdx.x*size_f->const_2BR;

    int indR2 = indR*2;
    int indR9 = indR*9;

    myReal normT = mysqrt(R[indR9+7]*R[indR9+7] + R[indR9+8]*R[indR9+8]);
    t[indR2] = R[indR9+8]/normT;
    t[indR2+1] = -R[indR9+7]/normT;
}

__global__ void compute_Omega(myReal* Omega, const myReal* R, const myReal* x, const myReal* t, const int* ind_n0, const myReal epsilon, const Size_f* size_f)
{
    int indR = ind_n0[blockIdx.y];
    int indx = threadIdx.x + blockIdx.x*size_f->const_2Bx;
    int indTot = (blockIdx.y + indx*gridDim.y)*2;

    int indR9 = indR*9;
    int indR2 = indR*2;
    indx = indx*2;

    myReal omega[3];
    omega[0] = R[indR9]*x[indx] + R[indR9+3]*x[indx+1];
    omega[1] = R[indR9+1]*x[indx] + R[indR9+4]*x[indx+1];
    omega[2] = R[indR9+2]*x[indx] + R[indR9+5]*x[indx+1];

    myReal ot = (1.0+epsilon) * (omega[1]*t[indR2] + omega[2]*t[indR2+1]);

    omega[1] = omega[1] - ot*t[indR2];
    omega[2] = omega[2] - ot*t[indR2+1];

    Omega[indTot] = R[indR9]*omega[0] + R[indR9+1]*omega[1] + R[indR9+2]*omega[2];
    Omega[indTot+1] = R[indR9+3]*omega[0] + R[indR9+4]*omega[1] + R[indR9+5]*omega[2];
}

