#include "hip/hip_runtime.h"
#include "getFcL.cuh"

#include <stdio.h>
#include <math.h>

void getFcL(myReal*** fcL, int** fcL_indx1, int* fcL_numx1, int*** fcL_indx2, int** fcL_numx2, const myReal* x, const myReal* Omega, const myReal* lambda, const int nn0, int* const* lambda_indx, const int* lambda_numx, const myReal* Gd, const Size_f* size_f)
{
    // pre-calculations
    myReal detGd = Gd[0]*Gd[3] - Gd[2]*Gd[1];
    myReal c_normal = 1/(2*PI*mysqrt(detGd));

    myReal invGd[4];
    invGd[0] = Gd[3]/detGd;
    invGd[1] = -Gd[2]/detGd;
    invGd[2] = -Gd[1]/detGd;
    invGd[3] = Gd[0]/detGd;

    myReal dx2 = (x[2]-x[0]) * (x[2]-x[0]);

    // calculate fc*lambda
    myReal* x_dev;
    cudaErrorHandle(hipMalloc(&x_dev, 2*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(x_dev, x, 2*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
    
    myReal* Omega_dev;
    cudaErrorHandle(hipMalloc(&Omega_dev, 2*nn0*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(Omega_dev, Omega, 2*nn0*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
    
    myReal* invGd_dev;
    cudaErrorHandle(hipMalloc(&invGd_dev, 4*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(invGd_dev, invGd, 4*sizeof(myReal), hipMemcpyHostToDevice));

    int* lambda_indx_dev;
    cudaErrorHandle(hipMalloc(&lambda_indx_dev, size_f->nx*sizeof(int)));

    myReal* fcL_x_dev;
    cudaErrorHandle(hipMalloc(&fcL_x_dev, size_f->nx*sizeof(myReal)));

    myReal* fcL_x_temp = (myReal*) malloc(size_f->nx*sizeof(myReal));

    for (int iR = 0; iR < nn0; iR++) {
        dim3 blocksize_x(256, 1, 1);
        dim3 gridsize_x((int)lambda_numx[iR]/256+1, 1, 1);

        cudaErrorHandle(hipMemcpy(lambda_indx_dev, lambda_indx[iR], lambda_numx[iR]*sizeof(int), hipMemcpyHostToDevice));

        fcL[iR] = (myReal**) malloc(size_f->nx*sizeof(myReal*));

        fcL_indx1[iR] = (int*) malloc(size_f->nx*sizeof(int));
        fcL_indx2[iR] = (int**) malloc(size_f->nx*sizeof(int*));

        fcL_numx1[iR] = 0;
        fcL_numx2[iR] = (int*) malloc(size_f->nx*sizeof(int));

        for (int ix = 0; ix < size_f->nx; ix++) {
            get_fcL_x <<<blocksize_x, gridsize_x>>> (fcL_x_dev, x_dev+2*ix, Omega_dev+2*iR, lambda[iR], lambda_indx_dev, invGd_dev, nn0, lambda_numx[iR], dx2, c_normal);
            
            fcL[iR][fcL_numx1[iR]] = (myReal*) malloc(lambda_numx[iR]*sizeof(myReal));
            fcL_indx2[iR][fcL_numx1[iR]] = (int*) malloc(lambda_numx[iR]*sizeof(int));

            cudaErrorHandle(hipMemcpy(fcL_x_temp, fcL_x_dev, lambda_numx[iR]*sizeof(myReal), hipMemcpyDeviceToHost));

            fcL_numx2[iR][fcL_numx1[iR]] = 0;
            for (int ix2 = 0; ix2 < lambda_numx[iR]; ix2++) {
                if (fcL_x_temp[ix2] > 1e-6) {
                    fcL[iR][fcL_numx1[iR]][fcL_numx2[iR][fcL_numx1[iR]]] = fcL_x_temp[ix2];
                    fcL_indx2[iR][fcL_numx1[iR]][fcL_numx2[iR][fcL_numx1[iR]]] = lambda_indx[iR][ix2];
                    fcL_numx2[iR][fcL_numx1[iR]]++;
                }
            }
            
            if (fcL_numx2[iR][fcL_numx1[iR]] == 0) {
                free(fcL[iR][fcL_numx1[iR]]);
                free(fcL_indx2[iR][fcL_numx1[iR]]);
            } else {
                fcL[iR][fcL_numx1[iR]] = (myReal*) realloc(fcL[iR][fcL_numx1[iR]], fcL_numx2[iR][fcL_numx1[iR]]*sizeof(myReal));
                fcL_indx2[iR][fcL_numx1[iR]] = (int*) realloc(fcL_indx2[iR][fcL_numx1[iR]], fcL_numx2[iR][fcL_numx1[iR]]*sizeof(int));
                
                fcL_indx1[iR][fcL_numx1[iR]] = ix;
                fcL_numx1[iR]++;
            }
        }

        fcL[iR] = (myReal**) realloc(fcL[iR], fcL_numx1[iR]*sizeof(myReal*));
        fcL_indx1[iR] = (int*) realloc(fcL_indx1[iR], fcL_numx1[iR]*sizeof(int));
        fcL_indx2[iR] = (int**) realloc(fcL_indx2[iR], fcL_numx1[iR]*sizeof(int*));
        fcL_numx2[iR] = (int*) realloc(fcL_numx2[iR], fcL_numx1[iR]*sizeof(int));

        printf("No. %d finished, totol: %d.\n", iR+1, nn0);
    }

    // free memory
    cudaErrorHandle(hipFree(x_dev));
    cudaErrorHandle(hipFree(Omega_dev));
    cudaErrorHandle(hipFree(invGd_dev));
    cudaErrorHandle(hipFree(lambda_indx_dev));
    cudaErrorHandle(hipFree(fcL_x_dev));

    free(fcL_x_temp);
}

__global__ void get_fcL_x(myReal* fcL_x, const myReal* x, const myReal* Omega, const myReal lambda, const int* lambda_indx, const myReal* invGd, const int nn0, const int lambda_numx, const myReal dx2, const myReal c_normal)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    
    if (indx < lambda_numx) {
        int indOmega = 2*lambda_indx[indx]*nn0;

        myReal dOmega[2];
        dOmega[0] = x[0] - Omega[indOmega];
        dOmega[1] = x[1] - Omega[indOmega+1];

        myReal fc_local = invGd[0]*dOmega[0]*dOmega[0] + (invGd[1]+invGd[2])*dOmega[0]*dOmega[1] + invGd[3]*dOmega[1]*dOmega[1];
        fc_local = myexp(-0.5*fc_local)*c_normal;
        fc_local = fc_local*lambda*dx2;

        fcL_x[indx] = fc_local;
    }
}

