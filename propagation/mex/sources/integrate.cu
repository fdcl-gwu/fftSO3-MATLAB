#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

__global__ void flip_shift(hipDoubleComplex* X, hipDoubleComplex* X_ijk, int is, int js, int ks, Size_F* size_F)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if (i < size_F[0].const_2Bx && j < size_F[0].const_2Bx && k < size_F[0].const_2Bx) {
		int iout = is-i;
		if (iout < 0)
			iout += size_F[0].const_2Bx;
		else if (iout >= size_F[0].const_2Bx)
			iout -= size_F[0].const_2Bx;

		int jout = js-j;
		if (jout < 0)
			jout += size_F[0].const_2Bx;
		else if (jout >= size_F[0].const_2Bx)
			jout -= size_F[0].const_2Bx;

		int kout = ks-k;
		if (kout < 0)
			kout += size_F[0].const_2Bx;
		else if (kout >= size_F[0].const_2Bx)
			kout -= size_F[0].const_2Bx;

		int X_ind = i + j*size_F[0].const_2Bx + k*size_F[0].const_2Bxs;
		int X_ijk_ind = iout + jout*size_F[0].const_2Bx + kout*size_F[0].const_2Bxs;

		for (int m = 0; m < 3; m++)
			X_ijk[X_ijk_ind + m*size_F[0].nx] = X[X_ind + m*size_F[0].nx];
	}
}

__global__ void addup_F(hipDoubleComplex* Fnew, hipDoubleComplex* Fold, double dt, Size_F* size_F)
{
	int ind1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind1 < size_F[0].nR_compact) {
		ind1 += blockIdx.y*size_F[0].nR_compact;

		int ind2 = ind1 + size_F[0].nTot_compact;
		int ind3 = ind2 + size_F[0].nTot_compact;

		Fnew[ind1] = hipCadd(Fnew[ind1], Fnew[ind2]);
		Fnew[ind1] = hipCadd(Fnew[ind1], Fnew[ind3]);

		Fnew[ind1].x = Fold[ind1].x + dt*Fnew[ind1].x;
		Fnew[ind1].y = Fold[ind1].y + dt*Fnew[ind1].y;
	}
}

__global__ void add(hipDoubleComplex*F1, hipDoubleComplex* F2, Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nTot_compact)
		F1[ind] = hipCadd(F1[ind], F2[ind]);
}

__global__ void get_c(double* c, int i, int j, double* G, double dt, Size_F* size_F)
{
	if (i == j) {
		int ix = threadIdx.x;
		if (ix < size_F[0].Bx)
			c[ix] = -PI*PI * ix*ix * G[i+3*j] * dt;
		else
			c[ix] = -PI*PI * (ix-size_F[0].const_2Bx)*(ix-size_F[0].const_2Bx) * G[i+3*j] * dt;
	} else {
		int ix = threadIdx.x;
		int jx = threadIdx.y;

		double c1;
		if (ix < size_F[0].Bx)
			c1 = PI * ix;
		else if (ix == size_F[0].Bx)
			c1 = 0;
		else
			c1 = PI * (ix-size_F[0].const_2Bx);

		double c2;
		if (jx < size_F[0].Bx)
			c2 = PI * jx;
		else if (jx == size_F[0].Bx)
			c2 = 0;
		else
			c2 = PI * (jx-size_F[0].const_2Bx);

		int indc = ix + jx*size_F[0].const_2Bx;
		c[indc] = -c1*c2*G[i+3*j]*dt;
	}
}

__global__ void add_biasRW(hipDoubleComplex* dF, hipDoubleComplex* Fold, double* c, int i, int j, Size_F* size_F)
{
	int indR = threadIdx.x + blockIdx.x*blockDim.x;
	if (indR < size_F[0].nR_compact) {
		unsigned int indx = blockIdx.y;
		int ijk[3];

		ijk[2] = (int) indx / size_F[0].const_2Bxs;
		int ijx = indx % size_F[0].const_2Bxs;
		ijk[1] = (int) ijx / size_F[0].const_2Bx;
		ijk[0] = ijx % size_F[0].const_2Bx;

		int ind = indR + indx*size_F[0].nR_compact;

		if (i==j) {
			dF[ind].x = Fold[ind].x * c[ijk[i]];
			dF[ind].y = Fold[ind].y * c[ijk[i]];
		} else {
			int indc = ijk[i] + ijk[j]*size_F[0].const_2Bx;
			dF[ind].x = Fold[ind].x * c[indc];
			dF[ind].y = Fold[ind].y * c[indc];
		}
	}
}

__host__ void modify_F(hipDoubleComplex* F, hipDoubleComplex* F_modify, bool reduce,Size_F* size_F)
{
	if (reduce) {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F_reduced] = F[ind_F];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	} else {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F] = F[ind_F_reduced];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	}
}

__host__ void modify_u(hipDoubleComplex* u, hipDoubleComplex* u_modify, Size_F* size_F)
{
	int ind_u_reduced = 0;
	for (int i = 0; i < 3; i++) {
		for (int l = 0; l <= size_F[0].lmax; l++) {
			for (int m = -l; m <= l; m++) {
				for (int n = -l; n <= l; n++) {
					int ind_u = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + l*size_F[0].l_cum1 + i*size_F[0].l_cum2;
					u_modify[ind_u_reduced] = u[ind_u];

					ind_u_reduced++;
				}
			}
		}
	}
}

__host__ void cudaErrorHandle(const hipError_t& err)
{
	if (err != hipSuccess) {
		std::cout << "Cuda Error: " << hipGetErrorString(err) << std::endl;
	}
}

__host__ void cutensorErrorHandle(const hiptensorStatus_t& err)
{
	if (err != HIPTENSOR_STATUS_SUCCESS) {
		std::cout << "cuTensor Error: " << hiptensorGetErrorString(err) << std::endl;
	}
}

__host__ void cublasErrorHandle(const hipblasStatus_t& err)
{
	if (err != HIPBLAS_STATUS_SUCCESS) {
		std::cout << "cuBlas Error: " << err << std::endl;
	}
}

__host__ void init_Size_F(Size_F* size_F, int BR, int Bx)
{
	size_F->BR = BR;
	size_F->Bx = Bx;
	size_F->lmax = BR-1;

	size_F->nR = (2*size_F->lmax+1) * (2*size_F->lmax+1) * (size_F->lmax+1);
	size_F->nx = (2*Bx) * (2*Bx) * (2*Bx);
	size_F->nTot = size_F->nR * size_F->nx;

	size_F->nR_compact = (size_F->lmax+1) * (2*size_F->lmax+1) * (2*size_F->lmax+3) / 3;
	size_F->nTot_compact = size_F->nR_compact * size_F->nx;

	size_F->const_2Bx = 2*Bx;
	size_F->const_2Bxs = (2*Bx) * (2*Bx);
	size_F->const_2lp1 = 2*size_F->lmax+1;
	size_F->const_lp1 = size_F->lmax+1;
	size_F->const_2lp1s = (2*size_F->lmax+1) * (2*size_F->lmax+1);

	size_F->l_cum0 = size_F->const_2lp1;
	size_F->l_cum1 = size_F->l_cum0*size_F->const_2lp1;
	size_F->l_cum2 = size_F->l_cum1*size_F->const_lp1;
	size_F->l_cum3 = size_F->l_cum2*size_F->const_2Bx;
	size_F->l_cum4 = size_F->l_cum3*size_F->const_2Bx;
}

