#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

__global__ void flip_shift(hipDoubleComplex* X, hipDoubleComplex* X_ijk, int is, int js, int ks, Size_F* size_F)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if (i < size_F[0].const_2Bx && j < size_F[0].const_2Bx && k < size_F[0].const_2Bx) {
		int iout = is-i;
		if (iout < 0)
			iout += size_F[0].const_2Bx;
		else if (iout >= size_F[0].const_2Bx)
			iout -= size_F[0].const_2Bx;

		int jout = js-j;
		if (jout < 0)
			jout += size_F[0].const_2Bx;
		else if (jout >= size_F[0].const_2Bx)
			jout -= size_F[0].const_2Bx;

		int kout = ks-k;
		if (kout < 0)
			kout += size_F[0].const_2Bx;
		else if (kout >= size_F[0].const_2Bx)
			kout -= size_F[0].const_2Bx;

		int X_ind = i + j*size_F[0].const_2Bx + k*size_F[0].const_2Bxs;
		int X_ijk_ind = iout + jout*size_F[0].const_2Bx + kout*size_F[0].const_2Bxs;

		for (int m = 0; m < 3; m++)
			X_ijk[X_ijk_ind + m*size_F[0].nx] = X[X_ind + m*size_F[0].nx];
	}
}

__global__ void addup_F(hipDoubleComplex* Fnew, hipDoubleComplex* Fold, double dt, Size_F* size_F)
{
	int ind1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind1 < size_F[0].nR_compact) {
		ind1 += blockIdx.y*size_F[0].nR_compact;

		int ind2 = ind1 + size_F[0].nTot_compact;
		int ind3 = ind2 + size_F[0].nTot_compact;

		Fnew[ind1] = hipCadd(Fnew[ind1], Fnew[ind2]);
		Fnew[ind1] = hipCadd(Fnew[ind1], Fnew[ind3]);

		Fnew[ind1].x = Fold[ind1].x + dt*Fnew[ind1].x;
		Fnew[ind1].y = Fold[ind1].y + dt*Fnew[ind1].y;
	}
}

__host__ void modify_F(hipDoubleComplex* F, hipDoubleComplex* F_modify, bool reduce,Size_F* size_F)
{
	if (reduce) {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F_reduced] = F[ind_F];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	} else {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F] = F[ind_F_reduced];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	}
}

__host__ void modify_u(hipDoubleComplex* u, hipDoubleComplex* u_modify, Size_F* size_F)
{
	int ind_u_reduced = 0;
	for (int i = 0; i < 3; i++) {
		for (int l = 0; l <= size_F[0].lmax; l++) {
			for (int m = -l; m <= l; m++) {
				for (int n = -l; n <= l; n++) {
					int ind_u = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + l*size_F[0].l_cum1 + i*size_F[0].l_cum2;
					u_modify[ind_u_reduced] = u[ind_u];

					ind_u_reduced++;
				}
			}
		}
	}
}

__host__ void cudaErrorHandle(const hipError_t& err)
{
	if (err != hipSuccess) {
		std::cout << "Cuda Error: " << hipGetErrorString(err) << std::endl;
	}
}

__host__ void cutensorErrorHandle(const hiptensorStatus_t& err)
{
	if (err != HIPTENSOR_STATUS_SUCCESS) {
		std::cout << "cuTensor Error: " << hiptensorGetErrorString(err) << std::endl;
	}
}

__host__ void cublasErrorHandle(const hipblasStatus_t& err)
{
	if (err != HIPBLAS_STATUS_SUCCESS) {
		std::cout << "cuBlas Error: " << err << std::endl;
	}
}

__host__ void init_Size_F(Size_F* size_F, int BR, int Bx)
{
	size_F->BR = BR;
	size_F->Bx = Bx;
	size_F->lmax = BR-1;

	size_F->nR = (2*size_F->lmax+1) * (2*size_F->lmax+1) * (size_F->lmax+1);
	size_F->nx = (2*Bx) * (2*Bx) * (2*Bx);
	size_F->nTot = size_F->nR * size_F->nx;

	size_F->nR_compact = (size_F->lmax+1) * (2*size_F->lmax+1) * (2*size_F->lmax+3) / 3;
	size_F->nTot_compact = size_F->nR_compact * size_F->nx;

	size_F->const_2Bx = 2*Bx;
	size_F->const_2Bxs = (2*Bx) * (2*Bx);
	size_F->const_2lp1 = 2*size_F->lmax+1;
	size_F->const_lp1 = size_F->lmax+1;
	size_F->const_2lp1s = (2*size_F->lmax+1) * (2*size_F->lmax+1);

	size_F->l_cum0 = size_F->const_2lp1;
	size_F->l_cum1 = size_F->l_cum0*size_F->const_2lp1;
	size_F->l_cum2 = size_F->l_cum1*size_F->const_lp1;
	size_F->l_cum3 = size_F->l_cum2*size_F->const_2Bx;
	size_F->l_cum4 = size_F->l_cum3*size_F->const_2Bx;
}

