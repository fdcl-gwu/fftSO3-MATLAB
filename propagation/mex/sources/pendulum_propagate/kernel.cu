
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

#include <string.h>

#include "mex.h"

void mexFunction (int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
    ////////////////////////////
    // get arrays from Matlab //
    ////////////////////////////

    // get Fold from matlab
    hipDoubleComplex* Fold = (hipDoubleComplex*) mxGetComplexDoubles(prhs[0]);
    const mwSize* size_Fold = mxGetDimensions(prhs[0]);

    Size_F size_F;
    init_Size_F(&size_F, (int)size_Fold[2], (int)size_Fold[3]/2);

    hipDoubleComplex* Fold_compact = new hipDoubleComplex[size_F.nTot_compact];
    modify_F(Fold, Fold_compact, true, &size_F);

    Size_F* size_F_dev;
    cudaErrorHandle(hipMalloc(&size_F_dev, sizeof(Size_F)));
    cudaErrorHandle(hipMemcpy(size_F_dev, &size_F, sizeof(Size_F), hipMemcpyHostToDevice));

    // set up output Fnew
    plhs[0] = mxCreateUninitNumericArray(6, (size_t*) size_Fold, mxDOUBLE_CLASS, mxCOMPLEX);
    hipDoubleComplex* Fnew = (hipDoubleComplex*) mxGetComplexDoubles(plhs[0]);

    hipDoubleComplex* Fnew_compact = new hipDoubleComplex[size_F.nTot_compact];
    
    // get X from matlab
    hipDoubleComplex* X = (hipDoubleComplex*) mxGetComplexDoubles(prhs[1]);

    // get OJO from matlab
    hipDoubleComplex* OJO = (hipDoubleComplex*) mxGetComplexDoubles(prhs[2]);

    // get MR from matlab
    hipDoubleComplex* MR = (hipDoubleComplex*) mxGetComplexDoubles(prhs[3]);

    hipDoubleComplex* MR_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(MR, MR_compact, &size_F);

    // get dt from matlab
    double* dt = mxGetDoubles(prhs[4]);

    // get L from matlab
    double* L = mxGetDoubles(prhs[5]);

    // get u from matlab
    hipDoubleComplex* u = (hipDoubleComplex*) mxGetComplexDoubles(prhs[6]);

    hipDoubleComplex* u_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(u, u_compact, &size_F);

    // get CG from matlab
    double** CG = new double* [size_F.BR*size_F.BR];
    for (int l1 = 0; l1 < size_F.BR; l1++) {
        for (int l2 = 0; l2 < size_F.BR; l2++) {
            int ind_CG = l1+l2*size_F.BR;
            CG[ind_CG] = mxGetDoubles(mxGetCell(prhs[7], ind_CG));
        }
    }
    
    // get method from matlab
    char* method;
    method = mxArrayToString(prhs[8]);

    //////////////////
    // calculate dF //
    //////////////////

    // set up arrays
    hipDoubleComplex* dF1;
    hipDoubleComplex* dF2;
    hipDoubleComplex* dF3;
    hipDoubleComplex* dF4;

    hipDoubleComplex* Fold_dev;
    cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // set up blocksize and gridsize
    dim3 blocksize_512_nTot(512, 1, 1);
    dim3 gridsize_512_nTot((int)size_F.nTot_compact/512+1, 1, 1);

    // calculate
    // dF1
    dF1 = new hipDoubleComplex[size_F.nTot_compact];
    get_dF(dF1, Fold_compact, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

    if (stricmp(method,"midpoint") == 0 || stricmp(method,"runge-kutta") == 0) {
        // dF2
        hipDoubleComplex* F2_dev;
        cudaErrorHandle(hipMalloc(&F2_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

        hipDoubleComplex* dF1_dev;
        cudaErrorHandle(hipMalloc(&dF1_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));
        cudaErrorHandle(hipMemcpy(dF1_dev, dF1, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F2_dev, Fold_dev, dF1_dev, dt[0]/2, size_F_dev);

        hipDoubleComplex* F2 = new hipDoubleComplex[size_F.nTot_compact];
        cudaErrorHandle(hipMemcpy(F2, F2_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

        dF2 = new hipDoubleComplex[size_F.nTot_compact];
        get_dF(dF2, F2, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

        delete[] F2;
        cudaErrorHandle(hipFree(F2_dev));
        cudaErrorHandle(hipFree(dF1_dev));
    }
    
    if (stricmp(method,"runge-kutta") == 0) {
        // dF3
        hipDoubleComplex* F3_dev;
        cudaErrorHandle(hipMalloc(&F3_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

        hipDoubleComplex* dF2_dev;
        cudaErrorHandle(hipMalloc(&dF2_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));
        cudaErrorHandle(hipMemcpy(dF2_dev, dF2, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F3_dev, Fold_dev, dF2_dev, dt[0]/2, size_F_dev);

        hipDoubleComplex* F3 = new hipDoubleComplex[size_F.nTot_compact];
        cudaErrorHandle(hipMemcpy(F3, F3_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

        dF3 = new hipDoubleComplex[size_F.nTot_compact];
        get_dF(dF3, F3, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

        delete[] F3;
        cudaErrorHandle(hipFree(F3_dev));
        cudaErrorHandle(hipFree(dF2_dev));

        // dF4
        hipDoubleComplex* F4_dev;
        cudaErrorHandle(hipMalloc(&F4_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

        hipDoubleComplex* dF3_dev;
        cudaErrorHandle(hipMalloc(&dF3_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));
        cudaErrorHandle(hipMemcpy(dF3_dev, dF3, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F4_dev, Fold_dev, dF3_dev, dt[0], size_F_dev);

        hipDoubleComplex* F4 = new hipDoubleComplex[size_F.nTot_compact];
        cudaErrorHandle(hipMemcpy(F4, F4_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

        dF4 = new hipDoubleComplex[size_F.nTot_compact];
        get_dF(dF4, F4, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

        delete[] F4;
        cudaErrorHandle(hipFree(F4_dev));
        cudaErrorHandle(hipFree(dF3_dev));
    }

    ///////////////
    // integrate //
    ///////////////

    // Fnew = Fold + dt*dF

    // set up arrays
    hipDoubleComplex* Fnew_dev;
    cudaErrorHandle(hipMalloc(&Fnew_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

    hipDoubleComplex* dF_dev;
    cudaErrorHandle(hipMalloc(&dF_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

    // calculate
    if (stricmp(method,"euler") == 0) {
        cudaErrorHandle(hipMemcpy(dF_dev, dF1, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F_dev);

        delete[] dF1;
    } else if (stricmp(method,"midpoint") == 0) {
        cudaErrorHandle(hipMemcpy(dF_dev, dF2, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F_dev);

        delete[] dF1;
        delete[] dF2;
    } else if (stricmp(method,"runge-kutta") == 0) {
        cudaErrorHandle(hipMemcpy(dF_dev, dF1, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0]/6, size_F_dev);

        cudaErrorHandle(hipMemcpy(dF_dev, dF2, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fnew_dev, dF_dev, dt[0]/3, size_F_dev);

        cudaErrorHandle(hipMemcpy(dF_dev, dF3, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fnew_dev, dF_dev, dt[0]/3, size_F_dev);

        cudaErrorHandle(hipMemcpy(dF_dev, dF4, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fnew_dev, dF_dev, dt[0]/6, size_F_dev);

        delete[] dF1;
        delete[] dF2;
        delete[] dF3;
        delete[] dF4;
    } else {
        mexPrintf("'method' must be 'euler', 'midpoint', or 'runge-kutta'. Return Fold.\n");
        Fnew_dev = Fold_dev;
    }

    // gather Fnew
    cudaErrorHandle(hipMemcpy(Fnew_compact, Fnew_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    modify_F(Fnew_compact, Fnew, false, &size_F);

    // free memory
    cudaErrorHandle(hipFree(Fold_dev));
    cudaErrorHandle(hipFree(Fnew_dev));
    cudaErrorHandle(hipFree(dF_dev));
    cudaErrorHandle(hipFree(size_F_dev));

    delete[] Fold_compact;
    delete[] Fnew_compact;
    delete[] MR_compact;
    delete[] u_compact;
    delete[] CG;
}

