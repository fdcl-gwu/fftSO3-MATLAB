#include "hip/hip_runtime.h"

#include "integrate.cuh"

#include <stdio.h>
#include <iostream>
#include <chrono>

#include <math.h>

#include "mex.h"

void mexFunction (int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
    ////////////////////////////
    // get arrays from Matlab //
    ////////////////////////////

    // get Fold from matlab
    hipDoubleComplex* Fold = (hipDoubleComplex*) mxGetComplexDoubles(prhs[0]);
    const mwSize* size_Fold = mxGetDimensions(prhs[0]);

    Size_F size_F;
    init_Size_F(&size_F, (int)size_Fold[2], (int)size_Fold[3]/2);

    hipDoubleComplex* Fold_compact = new hipDoubleComplex[size_F.nTot_compact];
    modify_F(Fold, Fold_compact, true, &size_F);

    Size_F* size_F_dev;
    cudaErrorHandle(hipMalloc(&size_F_dev, sizeof(Size_F)));
    cudaErrorHandle(hipMemcpy(size_F_dev, &size_F, sizeof(Size_F), hipMemcpyHostToDevice));

    // set up output Fnew
    plhs[0] = mxCreateUninitNumericArray(6, (size_t*) size_Fold, mxDOUBLE_CLASS, mxCOMPLEX);
    hipDoubleComplex* Fnew = (hipDoubleComplex*) mxGetComplexDoubles(plhs[0]);

    hipDoubleComplex* Fnew_compact = new hipDoubleComplex[size_F.nTot_compact];
    
    // get X from matlab
    hipDoubleComplex* X = (hipDoubleComplex*) mxGetComplexDoubles(prhs[1]);

    // get OJO from matlab
    hipDoubleComplex* OJO = (hipDoubleComplex*) mxGetComplexDoubles(prhs[2]);

    // get MR from matlab
    hipDoubleComplex* MR = (hipDoubleComplex*) mxGetComplexDoubles(prhs[3]);

    hipDoubleComplex* MR_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(MR, MR_compact, &size_F);

    // get dt from matlab
    double* dt = mxGetDoubles(prhs[4]);

    // get L from matlab
    double* L = mxGetDoubles(prhs[5]);

    // get u from matlab
    hipDoubleComplex* u = (hipDoubleComplex*) mxGetComplexDoubles(prhs[6]);

    hipDoubleComplex* u_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(u, u_compact, &size_F);

    // get CG from matlab
    double** CG = new double* [size_F.BR*size_F.BR];
    for (int l1 = 0; l1 < size_F.BR; l1++) {
        for (int l2 = 0; l2 < size_F.BR; l2++) {
            int ind_CG = l1+l2*size_F.BR;
            CG[ind_CG] = mxGetDoubles(mxGetCell(prhs[7], ind_CG));
        }
    }

    ////////////////////////////
    // circular_convolution X //
    ////////////////////////////

    // X_ijk = flip(flip(flip(X,1),2),3)
    // X_ijk = circshift(X_ijk,1,i)
    // X_ijk = circshift(X_ijk,2,j)
    // X_ijk = circshift(X_ijk,3,k)
    // dF{r,i,j,k,p} = Fold{r,m,n,l}.*X_ijk{m,n,l,p}

    std::cout << "Circular convolution with X begin\n";

    // set up GPU arrays
    hipDoubleComplex* Fold_dev;
    cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nx*size_F.nR_split * sizeof(hipDoubleComplex)));

    hipDoubleComplex* X_dev;
    cudaErrorHandle(hipMalloc(&X_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F.nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* X_ijk_dev;
    cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));

    hipDoubleComplex* dF3_dev;
    cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F.nR_split*sizeof(hipDoubleComplex)));

    // set up CPU arrays
    permute_F(Fold_compact, false, &size_F);

    hipDoubleComplex* dF3 = new hipDoubleComplex[3*size_F.nTot_compact];

    // set up cutensor
    hiptensorHandle_t handle_cutensor;
    cutensorInit(&handle_cutensor);

    hiptensorContractionPlan_t plan_conv[2];
    size_t worksize_conv[2] = {0,0};

    cutensor_initialize(&handle_cutensor, &plan_conv[0], &worksize_conv[0], Fold_dev, X_ijk_dev, dF3_dev, size_F.nR_split, &size_F);
    cutensor_initialize(&handle_cutensor, &plan_conv[1], &worksize_conv[1], Fold_dev, X_ijk_dev, dF3_dev, size_F.nR_remainder, &size_F);

    void* cutensor_workspace = nullptr;
    size_t worksize_max = worksize_conv[0]>worksize_conv[1] ? worksize_conv[0] : worksize_conv[1];
    if (worksize_max > 0) {
        cudaErrorHandle(hipMalloc(&cutensor_workspace, worksize_max));
    }

    hipDoubleComplex alpha_cutensor = make_hipDoubleComplex(-(double)1/size_F.nx,0);
    hipDoubleComplex beta_cutensor = make_hipDoubleComplex(0,0);

    // set up blocksize and gridsize
    dim3 blocksize_8(8, 8, 8);
    int gridnum_8 = ceil((double) size_F.const_2Bx/8);
    dim3 gridsize_8(gridnum_8, gridnum_8, gridnum_8);

    // calculate
    for (int is = 0; is < size_F.ns; is++) {
        int nR_split;
        if (is == size_F.ns-1)
            nR_split = size_F.nR_remainder;
        else
            nR_split = size_F.nR_split;

        cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact + is*size_F.nx*size_F.nR_split, size_F.nx*nR_split*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        for (int i = 0; i < size_F.const_2Bx; i++) {
            for (int j = 0; j < size_F.const_2Bx; j++) {
                for (int k = 0; k < size_F.const_2Bx; k++) {
                    flip_shift <<<gridsize_8, blocksize_8>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
                    cudaErrorHandle(hipGetLastError());

                    if (is == size_F.ns-1) {
                        cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[1], &alpha_cutensor, Fold_dev, X_ijk_dev,
                            &beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[1], 0));
                    } else {
                        cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[0], &alpha_cutensor, Fold_dev, X_ijk_dev,
                            &beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[0], 0));
                    }

                    for (int ip = 0; ip < 3; ip++) {
                        int ind_dF3 = is*size_F.nR_split + (i + j*size_F.const_2Bx + k*size_F.const_2Bxs)*size_F.nR_compact + ip*size_F.nTot_compact;
                        hipMemcpy(dF3+ind_dF3, dF3_dev+ip*nR_split, nR_split*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
                    }
                }
            }
        }
    }

    // free memory
    cudaErrorHandle(hipFree(X_dev));
    cudaErrorHandle(hipFree(X_ijk_dev));
    cudaErrorHandle(hipFree(dF3_dev));

    ////////////////
    // multiply u //
    ////////////////

    // dF(indmn,indmn,l,i,j,k,p) = -dF(indmn,indmn,l,i,j,k,p)*u(indmn,indmn,l,p)'

    // set up GPU arrays
    hipDoubleComplex* u_dev;
    cudaErrorHandle(hipMalloc(&u_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(u_dev, u_compact, 3*size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* dF_dev;
    cudaErrorHandle(hipMalloc(&dF_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex)));

    hipDoubleComplex* dF_dev_result;
    cudaErrorHandle(hipMalloc(&dF_dev_result, size_F.nTot_splitx*sizeof(hipDoubleComplex)));

    // set up CPU arrays
    permute_F(Fold_compact, true, &size_F);

    // set up cublas
    hipblasHandle_t handle_cublas;
    hipblasCreate(&handle_cublas);

    hipDoubleComplex alpha_cublas = make_hipDoubleComplex(1,0);
    hipDoubleComplex beta_cublas = make_hipDoubleComplex(0,0);

    // calculate
    for (int ip = 0; ip < 3; ip++) {
        for (int k = 0; k < size_F.const_2Bx; k++) {
            int ind_dF3 = k*size_F.nTot_splitx + ip*size_F.nTot_compact;
            cudaErrorHandle(hipMemcpy(dF_dev, dF3+ind_dF3, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

            for (int l = 0; l <= size_F.lmax; l++)
            {
                int ind_dF = l*(2*l-1)*(2*l+1)/3;
                long long int stride_dF = size_F.nR_compact;

                int ind_u = l*(2*l-1)*(2*l+1)/3 + ip*size_F.nR_compact;
                long long int stride_u = 0;

                cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
                    &alpha_cublas, dF_dev+ind_dF, 2*l+1, stride_dF,
                    u_dev+ind_u, 2*l+1, stride_u,
                    &beta_cublas, dF_dev_result+ind_dF, 2*l+1, stride_dF, size_F.const_2Bxs));
            }

            cudaErrorHandle(hipMemcpy(dF3+ind_dF3, dF_dev_result, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        }
    }

    // free memory
    cudaErrorHandle(hipFree(u_dev));
    cudaErrorHandle(hipFree(dF_dev_result));

    delete[] u_compact;

    /////////////
    // addup F //
    /////////////

    // dF = sum(dF,'p')

    // set up GPU arrays
    cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F.nTot_splitx*sizeof(hipDoubleComplex)));

    // set up CPU arrays
    hipDoubleComplex* dF = new hipDoubleComplex[size_F.nTot_compact];

    // set up blocksize and gridsize
    dim3 blocksize_512_nTot(512, 1, 1);
    dim3 gridsize_512_nTot(ceil((double) size_F.nTot_splitx/512), 1, 1);

    // calculate
    for (int k = 0; k < size_F.const_2Bx; k++) {
        for (int ip = 0; ip < 3; ip++) {
            int ind_dF3 = k*size_F.nTot_splitx + ip*size_F.nTot_compact;
            int ind_dF3_dev = ip*size_F.nTot_splitx;

            cudaErrorHandle(hipMemcpy(dF3_dev+ind_dF3_dev, dF3+ind_dF3, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        }

        addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
        cudaErrorHandle(hipGetLastError());

        int ind_dF = k*size_F.nTot_splitx;
        cudaErrorHandle(hipMemcpy(dF+ind_dF, dF3_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    }

    // free memory
    cudaErrorHandle(hipFree(dF3_dev));

    //////////////////////////////
    // circular convolution OJO //
    //////////////////////////////

    // OJO_ijk = flip(flip(flip(OJO,1),2),3)
    // OJO_ijk = circshift(OJO_ijk,1,i)
    // OJO_ijk = circshift(OJO_ijk,2,j)
    // OJO_ijk = circshift(OJO_ijk,3,k)
    // dF{r,i,j,k,p} = Fold{r,m,n,l}.*OJO_ijk{m,n,l,p}
    // dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)

    std::cout << "circular convolution with OJO begin\n";

    // set up GPU arrays
    hipDoubleComplex* OJO_dev;
    cudaErrorHandle(hipMalloc(&OJO_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(OJO_dev, OJO, 3*size_F.nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* OJO_ijk_dev;
    cudaErrorHandle(hipMalloc(&OJO_ijk_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));

    cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F.nR_split*sizeof(hipDoubleComplex)));

    // set up CPU arrays
    permute_F(Fold_compact, false, &size_F);

    // set up blocksize and gridsize
    dim3 blocksize_512_nR(512, 1, 1);
    dim3 gridsize_512_nR(ceil((double) size_F.nR_split/512), 1, 1);

    // calculate
    for (int is = 0; is < size_F.ns; is++) {
        int nR_split;
        if (is == size_F.ns-1) {
            nR_split = size_F.nR_remainder;
        } else {
            nR_split = size_F.nR_split;
        }

        gridsize_512_nR.x = ceil((double) nR_split/512);

        cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact + is*size_F.nx*size_F.nR_split, size_F.nx*nR_split*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        for (int i = 0; i < size_F.const_2Bx; i++) {
            for (int j = 0; j < size_F.const_2Bx; j++) {
                for (int k = 0; k < size_F.const_2Bx; k++) {
                    flip_shift <<<gridsize_8, blocksize_8>>> (OJO_dev, OJO_ijk_dev, i, j, k, size_F_dev);
                    cudaErrorHandle(hipGetLastError());

                    if (is == size_F.ns-1) {
                        cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[1], &alpha_cutensor, Fold_dev, OJO_ijk_dev,
                            &beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[1], 0));
                    } else {
                        cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[0], &alpha_cutensor, Fold_dev, OJO_ijk_dev,
                            &beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[0], 0));
                    }

                    double c[3];
                    deriv_x(c, i, size_F.Bx, *L);
                    deriv_x(c+1, j, size_F.Bx, *L);
                    deriv_x(c+2, k, size_F.Bx, *L);

                    mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF3_dev, c[0], nR_split, size_F_dev);
                    cudaErrorHandle(hipGetLastError());
                    mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF3_dev+nR_split, c[1], nR_split, size_F_dev);
                    cudaErrorHandle(hipGetLastError());
                    mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF3_dev+2*nR_split, c[2], nR_split, size_F_dev);
                    cudaErrorHandle(hipGetLastError());

                    for (int ip = 0; ip < 3; ip++) {
                        int ind_dF3 = is*size_F.nR_split + (i + j*size_F.const_2Bx + k*size_F.const_2Bxs)*size_F.nR_compact + ip*size_F.nTot_compact;
                        hipMemcpy(dF3+ind_dF3, dF3_dev+ip*nR_split, nR_split*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
                    }
                }
            }
        }
    }

    // free memory
    cudaErrorHandle(hipFree(OJO_dev));
    cudaErrorHandle(hipFree(OJO_ijk_dev));
    cudaErrorHandle(hipFree(dF3_dev));
    cudaErrorHandle(hipFree(Fold_dev));

    if (worksize_max > 0) {
        cudaErrorHandle(hipFree(cutensor_workspace));
    }

    /////////////
    // addup F //
    /////////////

    // dF = sum(dF,'p')

    // set up GPU arrays
    cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F.nTot_splitx*sizeof(hipDoubleComplex)));

    // calculate
    for (int k = 0; k < size_F.const_2Bx; k++) {
        for (int ip = 0; ip < 3; ip++) {
            int ind_dF3 = k*size_F.nTot_splitx + ip*size_F.nTot_compact;
            int ind_dF3_dev = ip*size_F.nTot_splitx;

            cudaErrorHandle(hipMemcpy(dF3_dev+ind_dF3_dev, dF3+ind_dF3, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        }

        addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
        cudaErrorHandle(hipGetLastError());

        int ind_dF = k*size_F.nTot_splitx;
        cudaErrorHandle(hipMemcpy(dF_dev, dF+ind_dF, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F_dev);
        cudaErrorHandle(hipGetLastError());

        cudaErrorHandle(hipMemcpy(dF+ind_dF, dF_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    }

    // free memory
    cudaErrorHandle(hipFree(dF3_dev));

    ///////////////////////
    // kronecker product //
    ///////////////////////

    std::cout << "kronecker product begin\n";

    // set up GPU arrays
    hipDoubleComplex** CG_dev = new hipDoubleComplex* [size_F.BR*size_F.BR];
    for (int l1 = 0; l1 <= size_F.lmax; l1++) {
        for (int l2 = 0; l2 <= size_F.lmax; l2++) {
            int m = (2*l1+1)*(2*l2+1);
            int ind_CG = l1+l2*size_F.BR;
            cudaErrorHandle(hipMalloc(&CG_dev[ind_CG], m*m*sizeof(hipDoubleComplex)));
            cudaErrorHandle(hipMemset(CG_dev[ind_CG], 0, m*m*sizeof(hipDoubleComplex)));

            double* CG_dev_d = (double*) CG_dev[ind_CG];
            cudaErrorHandle(hipMemcpy2D(CG_dev_d, 2*sizeof(double), CG[ind_CG], sizeof(double), sizeof(double), m*m, hipMemcpyHostToDevice));
        }
    }

    hipDoubleComplex* MR_dev;
    cudaErrorHandle(hipMalloc(&MR_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(MR_dev, MR_compact, 3*size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* FMR_dev;
    int m = (2*size_F.lmax+1) * (2*size_F.lmax+1);
    cudaErrorHandle(hipMalloc(&FMR_dev, 3*m*sizeof(hipDoubleComplex)));

    hipDoubleComplex* FMR_temp_dev;
    cudaErrorHandle(hipMalloc(&FMR_temp_dev, 3*size_F.const_2Bxs*sizeof(hipDoubleComplex)));

    hipDoubleComplex** Fold_strided = new hipDoubleComplex* [size_F.BR];
    for (int l = 0; l <= size_F.lmax; l++) {
        int m = (2*l+1)*(2*l+1);
        cudaErrorHandle(hipMalloc(&Fold_strided[l], m*size_F.const_2Bxs*sizeof(hipDoubleComplex)));
    }

    cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F.nTot_splitx*sizeof(hipDoubleComplex)));

    // set up CPU arrays
    permute_F(Fold_compact, true, &size_F);

    // get c
    double* c = new double[size_F.const_2Bx];
    for (int i = 0; i < size_F.const_2Bx; i++) {
        deriv_x(&c[i], i, size_F.Bx, *L);
    }

    double* c_dev;
    cudaErrorHandle(hipMalloc(&c_dev, size_F.const_2Bx*sizeof(double)));
    cudaErrorHandle(hipMemcpy(c_dev, c, size_F.const_2Bx*sizeof(double), hipMemcpyHostToDevice));

    // set up cutensor
    hiptensorContractionPlan_t* plan_FMR = new hiptensorContractionPlan_t [size_F.BR];
    size_t* worksize_FMR = new size_t [size_F.BR];

    for (int l1 = 0; l1 <= size_F.lmax; l1++) {
        cutensor_initFMR(&handle_cutensor, &plan_FMR[l1], &worksize_FMR[l1], Fold_strided[l1], FMR_dev, FMR_temp_dev, l1, size_F);
    }

    worksize_max = 0;
    for (int l = 0; l <= size_F.lmax; l++) {
        worksize_max = (worksize_FMR[l] > worksize_max) ? worksize_FMR[l] : worksize_max;
    }

    if (worksize_max > 0) {
        cudaErrorHandle(hipMalloc(&cutensor_workspace, worksize_max));
    }

    // set up blocksize and gridsize
    dim3 blocksize_addMFR(size_F.const_2Bx, 3, 1);
    dim3 gridsize_addMFR(size_F.const_2Bx, 1, 1);

    // calculate
    for (int k = 0; k < size_F.const_2Bx; k++) {
        int ind_Fold = k*size_F.nTot_splitx;
        for (int l = 0; l <= size_F.lmax; l++) {
            int ind = l*(2*l-1)*(2*l+1)/3;
            int m = (2*l+1)*(2*l+1);
            cudaErrorHandle(hipMemcpy2D(Fold_strided[l], m*sizeof(hipDoubleComplex), Fold_compact+ind_Fold+ind, size_F.nR_compact*sizeof(hipDoubleComplex),
                m*sizeof(hipDoubleComplex), size_F.const_2Bxs, hipMemcpyHostToDevice));
        }

        cudaErrorHandle(hipMemset(dF3_dev, 0, 3*size_F.nTot_splitx*sizeof(hipDoubleComplex)));

        for (int l = 0; l <= size_F.lmax; l++) {
            int ind_cumR = l*(2*l-1)*(2*l+1)/3;

            for (int l1 = 0; l1 <= size_F.lmax; l1++) {
                for (int l2 = 0; l2 <= size_F.lmax; l2++) {
                    if (abs(l1-l2)<=l && l1+l2>=l) {
                        int ind_MR = l2*(2*l2-1)*(2*l2+1)/3;
                        int ind_CG = l1+l2*size_F.BR;
                        int l12 = (2*l1+1)*(2*l2+1);

                        alpha_cutensor.x = (double) -l12/(2*l+1);

                        for (int m = -l; m <= l; m++) {
                            int ind_CG_m = (l*l-(l1-l2)*(l1-l2)+m+l)*l12;

                            for (int n = -l; n <= l; n++) {
                                int ind_CG_n = (l*l-(l1-l2)*(l1-l2)+n+l)*l12;
                                int ind_mnl = m+l + (n+l)*(2*l+1) + ind_cumR;

                                cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, 2*l1+1, 2*l2+1, 2*l2+1,
                                    &alpha_cublas, CG_dev[ind_CG]+ind_CG_m, 2*l2+1, 0, MR_dev+ind_MR, 2*l2+1, size_F.nR_compact,
                                    &beta_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), 3));

                                cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, 2*l1+1, 2*l1+1, 2*l2+1,
                                    &alpha_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), CG_dev[ind_CG]+ind_CG_n, 2*l2+1, 0,
                                    &beta_cublas, FMR_dev, 2*l1+1, (2*l1+1)*(2*l1+1), 3));

                                cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_FMR[l1], &alpha_cutensor, Fold_strided[l1],
                                    FMR_dev, &beta_cutensor, FMR_temp_dev, FMR_temp_dev, cutensor_workspace, worksize_FMR[l1], 0));

                                add_FMR <<<gridsize_addMFR, blocksize_addMFR>>> (dF3_dev, FMR_temp_dev, ind_mnl, size_F_dev);
                            }
                        }
                    }
                }
            }
        }

        for (int ip = 0; ip < 3; ip++) {
            int ind_dF3 = ind_Fold + ip*size_F.nTot_compact;
            int ind_dF3_dev = ip*size_F.nTot_splitx;
            cudaErrorHandle(hipMemcpy(dF3+ind_dF3, dF3_dev+ind_dF3_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        }
    }

    // take derivative about x
    dim3 blocksize_deriv(512,1,1);
    dim3 gridsize_deriv(ceil((double) size_F.nR_compact/512), size_F.const_2Bx, size_F.const_2Bx);

    for (int k = 0; k < size_F.const_2Bx; k++) {
        for (int ip = 0; ip < 3; ip++) {
            int ind_dF3 = k*size_F.nTot_splitx + ip*size_F.nTot_compact;

            cudaErrorHandle(hipMemcpy(dF3_dev, dF3+ind_dF3, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

            mulImg_FTot <<<gridsize_deriv, blocksize_deriv>>> (dF3_dev, c_dev, ip, k, size_F_dev);
            cudaErrorHandle(hipGetLastError());

            cudaErrorHandle(hipMemcpy(dF3+ind_dF3, dF3_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        }
    }

    // add up F
    for (int k = 0; k < size_F.const_2Bx; k++) {
        for (int ip = 0; ip < 3; ip++) {
            int ind_dF3 = k*size_F.nTot_splitx + ip*size_F.nTot_compact;
            int ind_dF3_dev = ip*size_F.nTot_splitx;

            cudaErrorHandle(hipMemcpy(dF3_dev+ind_dF3_dev, dF3+ind_dF3, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        }

        addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
        cudaErrorHandle(hipGetLastError());

        int ind_dF = k*size_F.nTot_splitx;
        cudaErrorHandle(hipMemcpy(dF_dev, dF+ind_dF, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F_dev);
        cudaErrorHandle(hipGetLastError());

        cudaErrorHandle(hipMemcpy(dF+ind_dF, dF_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    }

    // free memory
    cudaErrorHandle(hipFree(dF3_dev));
    cudaErrorHandle(hipFree(MR_dev));
    cudaErrorHandle(hipFree(FMR_dev));
    cudaErrorHandle(hipFree(FMR_temp_dev));
    cudaErrorHandle(hipFree(c_dev));

    if (worksize_max > 0) {
        cudaErrorHandle(hipFree(cutensor_workspace));
    }

    for (int l1 = 0; l1 <= size_F.lmax; l1++) {
        for (int l2 = 0; l2 <= size_F.lmax; l2++) {
            int ind_CG = l1+l2*size_F.BR;
            cudaErrorHandle(hipFree(CG_dev[ind_CG]));
        }
    }

    for (int l = 0; l <= size_F.lmax; l++) {
        cudaErrorHandle(hipFree(Fold_strided[l]));
    }

    delete[] c;
    delete[] CG_dev;
    delete[] Fold_strided;
    delete[] dF3;
    delete[] plan_FMR;
    delete[] worksize_FMR;

    delete[] MR_compact;
    delete[] CG;

    ///////////////
    // integrate //
    ///////////////

    // Fnew = Fold + dt*dF

    // set up GPU arrays
    hipDoubleComplex* Fnew_dev;
    cudaErrorHandle(hipMalloc(&Fnew_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex)));

    for (int k = 0; k < size_F.const_2Bx; k++) {
        int ind_F = k*size_F.nTot_splitx;
        cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        cudaErrorHandle(hipMemcpy(dF_dev, dF+ind_F, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F_dev);
        cudaErrorHandle(hipGetLastError());

        cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    }

    // gather Fnew
    modify_F(Fnew_compact, Fnew, false, &size_F);

    // free memory
    cudaErrorHandle(hipFree(Fold_dev));
    cudaErrorHandle(hipFree(Fnew_dev));
    cudaErrorHandle(hipFree(dF_dev));
    cudaErrorHandle(hipFree(size_F_dev));

    delete[] dF;
    delete[] Fold_compact;
    delete[] Fnew_compact;
}

