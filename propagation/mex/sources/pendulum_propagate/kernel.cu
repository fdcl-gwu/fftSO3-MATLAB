#include "hip/hip_runtime.h"

#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

#include <math.h>

#include "mex.h"

void mexFunction (int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
    ////////////////////////////
    // get arrays from Matlab //
    ////////////////////////////

    // get Fold from matlab
    hipDoubleComplex* Fold = (hipDoubleComplex*) mxGetComplexDoubles(prhs[0]);
    const mwSize* size_Fold = mxGetDimensions(prhs[0]);

    Size_F size_F;
    init_Size_F(&size_F, (int)size_Fold[2], (int)size_Fold[3]/2);

    hipDoubleComplex* Fold_compact = new hipDoubleComplex[size_F.nTot_compact];
    modify_F(Fold, Fold_compact, true, &size_F);

    Size_F* size_F_dev;
    cudaErrorHandle(hipMalloc(&size_F_dev, sizeof(Size_F)));
    cudaErrorHandle(hipMemcpy(size_F_dev, &size_F, sizeof(Size_F), hipMemcpyHostToDevice));

    // set up output Fnew
    plhs[0] = mxCreateUninitNumericArray(6, (size_t*) size_Fold, mxDOUBLE_CLASS, mxCOMPLEX);
    hipDoubleComplex* Fnew = (hipDoubleComplex*) mxGetComplexDoubles(plhs[0]);

    hipDoubleComplex* Fnew_compact = new hipDoubleComplex[size_F.nTot_compact];
    
    // get X from matlab
    hipDoubleComplex* X = (hipDoubleComplex*) mxGetComplexDoubles(prhs[1]);

    // get OJO from matlab
    hipDoubleComplex* OJO = (hipDoubleComplex*) mxGetComplexDoubles(prhs[2]);

    // get MR from matlab
    hipDoubleComplex* MR = (hipDoubleComplex*) mxGetComplexDoubles(prhs[3]);

    hipDoubleComplex* MR_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(MR, MR_compact, &size_F);

    // get dt from matlab
    double* dt = mxGetDoubles(prhs[4]);

    // get L from matlab
    double* L = mxGetDoubles(prhs[5]);

    // get u from matlab
    hipDoubleComplex* u = (hipDoubleComplex*) mxGetComplexDoubles(prhs[6]);

    hipDoubleComplex* u_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(u, u_compact, &size_F);

    // get CG from matlab
    double** CG = new double* [size_F.BR*size_F.BR];
    for (int l1 = 0; l1 < size_F.BR; l1++) {
        for (int l2 = 0; l2 < size_F.BR; l2++) {
            int ind_CG = l1+l2*size_F.BR;
            CG[ind_CG] = mxGetDoubles(mxGetCell(prhs[7], ind_CG));
        }
    }

    ////////////////////////////
    // circular_convolution X //
    ////////////////////////////

    // X_ijk = flip(flip(flip(X,1),2),3)
    // X_ijk = circshift(X_ijk,1,i)
    // X_ijk = circshift(X_ijk,2,j)
    // X_ijk = circshift(X_ijk,3,k)
    // dF{r,i,j,k,p} = Fold{r,m,n,l}.*X_ijk{m,n,l,p}

    mexPrintf("Circular convolution with X begin\n");

    // set up arrays
    hipDoubleComplex* Fold_dev;
    cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* X_dev;
    cudaErrorHandle(hipMalloc(&X_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F.nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* X_ijk_dev;
    cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));

    hipDoubleComplex* dF3_dev;
    cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F.nTot_compact*sizeof(hipDoubleComplex)));

    hipDoubleComplex* dF_temp_dev;
    cudaErrorHandle(hipMalloc(&dF_temp_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));

    // set up cutensor
    hiptensorHandle_t handle_cutensor;
    cutensorInit(&handle_cutensor);

    hiptensorContractionPlan_t plan;
    size_t worksize;

    cutensor_initialize(&handle_cutensor, &plan, &worksize, Fold_dev, X_ijk_dev, dF_temp_dev, size_F);

    void* work = nullptr;
    if (worksize > 0)
        cudaErrorHandle(hipMalloc(&work, worksize));

    hipDoubleComplex alpha_cutensor = make_hipDoubleComplex(0-(double)1/size_F.nx,0);
    hipDoubleComplex beta_cutensor = make_hipDoubleComplex(0,0);

    // set up blocksize and gridsize
    dim3 blocksize_8(8, 8, 8);
    int gridnum_8 = ceil((double) size_F.const_2Bx/8);
    dim3 gridsize_8(gridnum_8, gridnum_8, gridnum_8);

    // calculate
    for (int i = 0; i < size_F.const_2Bx; i++) {
        for (int j = 0; j < size_F.const_2Bx; j++) {
            for (int k = 0; k < size_F.const_2Bx; k++) {
                flip_shift <<<gridsize_8, blocksize_8>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
                cudaErrorHandle(hipGetLastError());

                hipDeviceSynchronize();

                cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan, (void*)&alpha_cutensor, Fold_dev, X_ijk_dev,
                    (void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize, 0));

                hipDeviceSynchronize();

                for (int n = 0; n < 3; n++) {
                    hipDoubleComplex* dF3_dev_ijkn = dF3_dev + i*size_F.nR_compact + 
                        j*(size_F.nR_compact*size_F.const_2Bx) + k*(size_F.nR_compact*size_F.const_2Bxs) + n*size_F.nTot_compact;
                    hipDoubleComplex* dF_temp_dev_n = dF_temp_dev + n*size_F.nR_compact;

                    cudaErrorHandle(hipMemcpy(dF3_dev_ijkn, dF_temp_dev_n, size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));
                }

                hipDeviceSynchronize();
            }
        }
    }

    // free memory
    cudaErrorHandle(hipFree(X_dev));
    cudaErrorHandle(hipFree(X_ijk_dev));
    cudaErrorHandle(hipFree(dF_temp_dev));
    if (worksize > 0)
        cudaErrorHandle(hipFree(work));

    ////////////////
    // multiply u //
    ////////////////

    // dF(indmn,indmn,l,i,j,k,p) = -dF(indmn,indmn,l,i,j,k,p)*u(indmn,indmn,l,p)'

    // set up arrays
    hipDoubleComplex* u_dev;
    cudaErrorHandle(hipMalloc(&u_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(u_dev, u_compact, 3*size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // set up cublas
    hipblasHandle_t handle_cublas;
    hipblasCreate(&handle_cublas);

    hipDoubleComplex alpha_cublas = make_hipDoubleComplex(1,0);
    hipDoubleComplex beta_cublas = make_hipDoubleComplex(0,0);

    // calculate
    for (int i = 0; i < 3; i++) {
        for (int l = 0; l <= size_F.lmax; l++)
        {
            int ind_dF = l*(2*l-1)*(2*l+1)/3 + i*size_F.nTot_compact;
            long long int stride_Fnew = size_F.nR_compact;

            int ind_u = l*(2*l-1)*(2*l+1)/3 + i*size_F.nR_compact;
            long long int stride_u = 0;

            cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
                &alpha_cublas, dF3_dev+ind_dF, 2*l+1, stride_Fnew,
                u_dev+ind_u, 2*l+1, stride_u,
                &beta_cublas, dF3_dev+ind_dF, 2*l+1, stride_Fnew, size_F.nx));

            hipDeviceSynchronize();
        }
    }

    // free memory
    cudaErrorHandle(hipFree(u_dev));

    /////////////
    // addup F //
    /////////////

    // dF = sum(dF,'p')

    // set up arrays
    hipDoubleComplex* dF_dev;
    cudaErrorHandle(hipMalloc(&dF_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

    // set up blocksize and gridsize
    dim3 blocksize_512_nTot(512, 1, 1);
    dim3 gridsize_512_nTot(ceil((double) size_F.nTot_compact/512), 1, 1);

    // calculate
    addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
    cudaErrorHandle(hipGetLastError());

    cudaErrorHandle(hipMemcpy(dF_dev, dF3_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

    //////////////////////////////
    // circular convolution OJO //
    //////////////////////////////

    // OJO_ijk = flip(flip(flip(OJO,1),2),3)
    // OJO_ijk = circshift(OJO_ijk,1,i)
    // OJO_ijk = circshift(OJO_ijk,2,j)
    // OJO_ijk = circshift(OJO_ijk,3,k)
    // dF{r,i,j,k,p} = Fold{r,m,n,l}.*OJO_ijk{m,n,l,p}
    // dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)

    mexPrintf("circular convolution with OJO begin\n");

    // set up arrays
    hipDoubleComplex* OJO_dev;
    cudaErrorHandle(hipMalloc(&OJO_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(OJO_dev, OJO, 3*size_F.nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    hipDoubleComplex* OJO_ijk_dev;
    cudaErrorHandle(hipMalloc(&OJO_ijk_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));

    cudaErrorHandle(hipMalloc(&dF_temp_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));

    // set up cutensor
    if (worksize > 0)
        cudaErrorHandle(hipMalloc(&work, worksize));

    // set up blocksize and gridsize
    dim3 blocksize_512_nR(512, 1, 1);
    dim3 gridsize_512_nR(ceil((double) size_F.nR_compact/512), 1, 1);

    // calculate
    for (int i = 0; i < size_F.const_2Bx; i++) {
        for (int j = 0; j < size_F.const_2Bx; j++) {
            for (int k = 0; k < size_F.const_2Bx; k++) {
                flip_shift <<<gridsize_8, blocksize_8>>> (OJO_dev, OJO_ijk_dev, i, j, k, size_F_dev);
                cudaErrorHandle(hipGetLastError());

                hipDeviceSynchronize();

                cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan, (void*)&alpha_cutensor, Fold_dev, OJO_ijk_dev,
                    (void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize, 0));

                double c[3];
                deriv_x(c, i, size_F.Bx, *L);
                deriv_x(c+1, j, size_F.Bx, *L);
                deriv_x(c+2, k, size_F.Bx, *L);

                mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev, c[0], size_F_dev);
                cudaErrorHandle(hipGetLastError());
                mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+size_F.nR_compact, c[1], size_F_dev);
                cudaErrorHandle(hipGetLastError());
                mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+2*size_F.nR_compact, c[2], size_F_dev);
                cudaErrorHandle(hipGetLastError());

                hipDeviceSynchronize();

                for (int n = 0; n < 3; n++) {
                    hipDoubleComplex* dF3_dev_ijkn = dF3_dev + i*size_F.nR_compact + 
                        j*(size_F.nR_compact*size_F.const_2Bx) + k*(size_F.nR_compact*size_F.const_2Bxs) + n*size_F.nTot_compact;
                    hipDoubleComplex* dF_temp_dev_n = dF_temp_dev + n*size_F.nR_compact;

                    cudaErrorHandle(hipMemcpy(dF3_dev_ijkn, dF_temp_dev_n, size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));
                }

                hipDeviceSynchronize();
            }
        }
    }

    // free memory
    cudaErrorHandle(hipFree(OJO_dev));
    cudaErrorHandle(hipFree(OJO_ijk_dev));
    cudaErrorHandle(hipFree(dF_temp_dev));
    if (worksize > 0)
        cudaErrorHandle(hipFree(work));

    /////////////
    // addup F //
    /////////////

    // dF = sum(dF,'p')

    // calculate
    addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
    cudaErrorHandle(hipGetLastError());

    add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F_dev);
    cudaErrorHandle(hipGetLastError());

    ///////////////////////
    // kronecker product //
    ///////////////////////

    mexPrintf("kronecker product begin\n");

    // set up arrays
    double** CG_dev = new double* [size_F.BR*size_F.BR];
    for (int l1 = 0; l1 <= size_F.lmax; l1++) {
        for (int l2 = 0; l2 <= size_F.lmax; l2++) {
            int m = (2*l1+1)*(2*l2+1);
            int ind_CG = l1+l2*size_F.BR;
            cudaErrorHandle(hipMalloc(&CG_dev[ind_CG], m*m*sizeof(double)));
            cudaErrorHandle(hipMemcpy(CG_dev[ind_CG], CG[ind_CG], m*m*sizeof(double), hipMemcpyHostToDevice));
        }
    }

    hipDoubleComplex* MR_dev;
    cudaErrorHandle(hipMalloc(&MR_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(MR_dev, MR_compact, 3*size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    double* FMR_real_dev;
    double* FMR_imag_dev;
    int m = (2*size_F.lmax+1) * (2*size_F.lmax+1);
    cudaErrorHandle(hipMalloc(&FMR_real_dev, m*m*size_F.const_2Bxs*sizeof(double)));
    cudaErrorHandle(hipMalloc(&FMR_imag_dev, m*m*size_F.const_2Bxs*sizeof(double)));

    double* FMR_real_temp_dev;
    double* FMR_imag_temp_dev;
    cudaErrorHandle(hipMalloc(&FMR_real_temp_dev, (2*size_F.lmax+1)*m*size_F.const_2Bxs*sizeof(double)));
    cudaErrorHandle(hipMalloc(&FMR_imag_temp_dev, (2*size_F.lmax+1)*m*size_F.const_2Bxs*sizeof(double)));

    cudaErrorHandle(hipMemset(dF3_dev, 0, 3*size_F.nTot_compact*sizeof(hipDoubleComplex)));

    // get c
    double* c = new double[size_F.const_2Bx];
    for (int i = 0; i < size_F.const_2Bx; i++) {
        deriv_x(&c[i], i, size_F.Bx, *L);
    }

    double* c_dev;
    cudaErrorHandle(hipMalloc(&c_dev, size_F.const_2Bx*sizeof(double)));
    cudaErrorHandle(hipMemcpy(c_dev, c, size_F.const_2Bx*sizeof(double), hipMemcpyHostToDevice));

    // set up cublas
    double alpha_cublas_real = 1;
    double beta_cublas_real = 0;

    // calculate
    for (int l = 0; l <= size_F.lmax; l++) {
        dim3 blocksize_add_FMR((2*l+1)*(2*l+1), 1, 1);
        dim3 gridsize_add_FMR(size_F.const_2Bxs, 1, 1);

        int ind_cumR = l*(2*l-1)*(2*l+1)/3;

        for (int l1 = 0; l1 <= size_F.lmax; l1++) {
            int ind_F_cumR = l1*(2*l1-1)*(2*l1+1)/3;

            for (int l2 = 0; l2 <= size_F.lmax; l2++) {
                int ind_MR_cumR = l2*(2*l2-1)*(2*l2+1)/3;

                int ind_CG = l1+l2*size_F.BR;
                int m = (2*l1+1)*(2*l2+1);
                int ind_CG_l1l2 = (l*l-(l1-l2)*(l1-l2))*m;

                if (abs(l1-l2)<=l && l1+l2>=l) {
                    // BR <= 16
                    dim3 blocksize_kron(2*l1+1, 2*l1+1, 1);
                    dim3 gridsize_kron(2*l2+1, 2*l2+1, size_F.const_2Bxs);

                    for (int k = 0; k < size_F.const_2Bx; k++) {
                        int ind_Fold = k*size_F.nR_compact*size_F.const_2Bxs;
                        for (int i = 0; i < 3; i++) {
                            int ind_MR = i*size_F.nR_compact;
                            int ind_dF = k*size_F.nR_compact*size_F.const_2Bxs + i*size_F.nTot_compact;

                            kron_FMR <<<gridsize_kron, blocksize_kron>>> (FMR_real_dev, FMR_imag_dev, Fold_dev+ind_Fold, MR_dev+ind_MR, ind_F_cumR, ind_MR_cumR, size_F_dev);
                            cudaErrorHandle(hipGetLastError());

                            // real part
                            alpha_cublas_real = -1;
                            cublasErrorHandle(hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                2*l+1, m, m, &alpha_cublas_real, CG_dev[ind_CG]+ind_CG_l1l2, m, 0,
                                FMR_real_dev, m, m*m, &beta_cublas_real, FMR_real_temp_dev, 2*l+1, (2*l+1)*m, size_F.const_2Bxs));

                            alpha_cublas_real = (double) m/(2*l+1);
                            cublasErrorHandle(hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                2*l+1, 2*l+1, m, &alpha_cublas_real, FMR_real_temp_dev, 2*l+1, (2*l+1)*m,
                                CG_dev[ind_CG]+ind_CG_l1l2, m, 0, &beta_cublas_real, FMR_real_dev, 2*l+1, (2*l+1)*(2*l+1), size_F.const_2Bxs));

                            // imaginary part
                            alpha_cublas_real = -1;
                            cublasErrorHandle(hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                2*l+1, m, m, &alpha_cublas_real, CG_dev[ind_CG]+ind_CG_l1l2, m, 0,
                                FMR_imag_dev, m, m*m, &beta_cublas_real, FMR_imag_temp_dev, 2*l+1, (2*l+1)*m, size_F.const_2Bxs));

                            alpha_cublas_real = (double) m/(2*l+1);
                            cublasErrorHandle(hipblasDgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                2*l+1, 2*l+1, m, &alpha_cublas_real, FMR_imag_temp_dev, 2*l+1, (2*l+1)*m,
                                CG_dev[ind_CG]+ind_CG_l1l2, m, 0, &beta_cublas_real, FMR_imag_dev, 2*l+1, (2*l+1)*(2*l+1), size_F.const_2Bxs));

                            // add
                            add_FMR <<<gridsize_add_FMR, blocksize_add_FMR>>> (dF3_dev+ind_dF, FMR_real_dev, FMR_imag_dev, ind_cumR, size_F_dev);

                            /* // test
                            if (l==1 && k==0 && i==0) {
                                mexPrintf("l1 = %i, l2 = %i\n", l1, l2);

                                double* FMR_real_test = new double[(2*l+1)*(2*l+1)*size_F.const_2Bxs];
                                double* FMR_imag_test = new double[(2*l+1)*(2*l+1)*size_F.const_2Bxs];

                                cudaErrorHandle(hipMemcpy(FMR_real_test, FMR_real_dev, (2*l+1)*(2*l+1)*size_F.const_2Bxs*sizeof(double), hipMemcpyDeviceToHost));
                                cudaErrorHandle(hipMemcpy(FMR_imag_test, FMR_imag_dev, (2*l+1)*(2*l+1)*size_F.const_2Bxs*sizeof(double), hipMemcpyDeviceToHost));

                                mexPrintf("(%e,%e) (%e,%e)\n", FMR_real_test[0], FMR_imag_test[0], FMR_real_test[1], FMR_imag_test[1]);
                                mexPrintf("(%e,%e) (%e,%e)\n", FMR_real_test[(2*l+1)*(2*l+1)*size_F.const_2Bxs-2], FMR_imag_test[(2*l+1)*(2*l+1)*size_F.const_2Bxs-2],
                                    FMR_real_test[(2*l+1)*(2*l+1)*size_F.const_2Bxs-1], FMR_imag_test[(2*l+1)*(2*l+1)*size_F.const_2Bxs-1]);

                                hipDoubleComplex* dF_test = new hipDoubleComplex[2];
                                cudaErrorHandle(hipMemcpy(dF_test, dF3_dev, 2*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

                                mexPrintf("(%e,%e) (%e,%e)\n", dF_test[0].x, dF_test[0].y, dF_test[1].x, dF_test[1].y);

                                delete[] dF_test;
                                delete[] FMR_real_test;
                                delete[] FMR_imag_test;
                            } */
                        }
                    }
                }
            }
        }
    }

    // take derivative about x
    dim3 blocksize_deriv(512,1,1);
    dim3 gridsize_deriv(ceil((double) size_F.nR_compact/512), size_F.const_2Bx, size_F.const_2Bxs);

    for (int i = 0; i < 3; i++) {
        mulImg_FTot <<<gridsize_deriv, blocksize_deriv>>> (dF3_dev+i*size_F.nTot_compact, c_dev, i, size_F_dev);
        cudaErrorHandle(hipGetLastError());
    }

    // addup F
    addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
    cudaErrorHandle(hipGetLastError());

    add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F_dev);
    cudaErrorHandle(hipGetLastError());

    // test
    /* hipDoubleComplex* dF3_test = new hipDoubleComplex[3*size_F.nTot_compact];
    cudaErrorHandle(hipMemcpy(dF3_test, dF3_dev, 3*size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    mexPrintf("(%e,%e) (%e,%e)\n", dF3_test[0].x, dF3_test[0].y, dF3_test[1].x, dF3_test[1].y);
    mexPrintf("(%e,%e) (%e,%e)\n", dF3_test[size_F.nTot_compact].x, dF3_test[size_F.nTot_compact].y, dF3_test[2*size_F.nTot_compact-1].x, dF3_test[2*size_F.nTot_compact-1].y);
    mexPrintf("(%e,%e) (%e,%e)\n", dF3_test[2*size_F.nTot_compact].x, dF3_test[2*size_F.nTot_compact].y, dF3_test[3*size_F.nTot_compact-1].x, dF3_test[3*size_F.nTot_compact-1].y);

    delete[] dF3_test; */

    /* // test
    hipDoubleComplex* dF_test = new hipDoubleComplex[size_F.nTot_compact];
    cudaErrorHandle(hipMemcpy(dF_test, dF_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    mexPrintf("(%e,%e) (%e,%e)\n", dF_test[0].x, dF_test[0].y, dF_test[1].x, dF_test[1].y);
    mexPrintf("(%e,%e) (%e,%e)\n", dF_test[size_F.nTot_compact-2].x, dF_test[size_F.nTot_compact-2].y, dF_test[size_F.nTot_compact-1].x, dF_test[size_F.nTot_compact-1].y);

    delete[] dF_test; */

    // free memory
    cudaErrorHandle(hipFree(dF3_dev));
    cudaErrorHandle(hipFree(MR_dev));
    cudaErrorHandle(hipFree(FMR_real_dev));
    cudaErrorHandle(hipFree(FMR_imag_dev));
    cudaErrorHandle(hipFree(c_dev));

    for (int l1 = 0; l1 <= size_F.lmax; l1++) {
        for (int l2 = 0; l2 <= size_F.lmax; l2++) {
            int ind_CG = l1+l2*size_F.BR;
            cudaErrorHandle(hipFree(CG_dev[ind_CG]));
        }
    }

    delete[] c;

    ///////////////
    // integrate //
    ///////////////

    // Fnew = Fold + dt*dF

    // set up arrays
    hipDoubleComplex* Fnew_dev;
    cudaErrorHandle(hipMalloc(&Fnew_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));

    integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F_dev);

    // gather Fnew
    cudaErrorHandle(hipMemcpy(Fnew_compact, Fnew_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    modify_F(Fnew_compact, Fnew, false, &size_F);

    // free memory
    cudaErrorHandle(hipFree(Fold_dev));
    cudaErrorHandle(hipFree(Fnew_dev));
    cudaErrorHandle(hipFree(dF_dev));
    cudaErrorHandle(hipFree(size_F_dev));
}

