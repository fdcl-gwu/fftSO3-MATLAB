#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

__global__ void flip_shift(const myComplex* X, myComplex* X_ijk, const int is, const int js, const int ks, const Size_F* size_F)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if (i < size_F[0].const_2Bx && j < size_F[0].const_2Bx && k < size_F[0].const_2Bx) {
		int iout = is-i;
		if (iout < 0)
			iout += size_F[0].const_2Bx;
		else if (iout >= size_F[0].const_2Bx)
			iout -= size_F[0].const_2Bx;

		int jout = js-j;
		if (jout < 0)
			jout += size_F[0].const_2Bx;
		else if (jout >= size_F[0].const_2Bx)
			jout -= size_F[0].const_2Bx;

		int kout = ks-k;
		if (kout < 0)
			kout += size_F[0].const_2Bx;
		else if (kout >= size_F[0].const_2Bx)
			kout -= size_F[0].const_2Bx;

		int X_ind = i + j*size_F[0].const_2Bx + k*size_F[0].const_2Bxs;
		int X_ijk_ind = iout + jout*size_F[0].const_2Bx + kout*size_F[0].const_2Bxs;

		for (int m = 0; m < 3; m++)
			X_ijk[X_ijk_ind + m*size_F[0].nx] = X[X_ind + m*size_F[0].nx];
	}
}

__global__ void addup_F(myComplex* dF, const int nTot)
{
	int ind1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind1 < nTot) {
		int ind2 = ind1 + nTot;
		int ind3 = ind2 + nTot;

		dF[ind1] = mycuCadd(dF[ind1], dF[ind2]);
		dF[ind1] = mycuCadd(dF[ind1], dF[ind3]);
	}
}

__global__ void add_F(myComplex* dF, const myComplex* dF_temp, const int nTot)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < nTot)
		dF[ind] = mycuCadd(dF[ind], dF_temp[ind]);
}

__global__ void mulImg_FR(myComplex* dF, const myReal c, const int nR)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < nR) {
		myReal y = dF[ind].y;
		dF[ind].y = dF[ind].x * c;
		dF[ind].x = -y * c;
	}
}

__global__ void add_FMR_small(myComplex* dF, const myComplex* FMR, const int ind_cumR, const Size_F* size_F)
{
	int ind_dF = ind_cumR + (threadIdx.x + threadIdx.y*size_F->const_2Bx + blockIdx.x*size_F->const_2Bxs)*size_F->nR_compact + blockIdx.y*size_F->nTot_compact;
	int ind_FMR = threadIdx.x + threadIdx.y*size_F->const_2Bx + blockIdx.x*size_F->const_2Bxs + blockIdx.y*size_F->nx;

	dF[ind_dF] = mycuCadd(dF[ind_dF], FMR[ind_FMR]);
}

__global__ void add_FMR_large(myComplex* dF, const myComplex* FMR, const int ind_cumR, const Size_F* size_F)
{
	int ind_dF = ind_cumR + (threadIdx.x + blockIdx.x*size_F->const_2Bx)*size_F->nR_compact + threadIdx.y*size_F->nTot_splitx;
	int ind_FMR = threadIdx.x + blockIdx.x*size_F->const_2Bx + threadIdx.y*size_F->const_2Bxs;

	dF[ind_dF] = mycuCadd(dF[ind_dF], FMR[ind_FMR]);
}

__global__ void mulImg_FTot_small(myComplex* dF, const myReal* c, const int dim, const Size_F* size_F)
{
	int ind_R = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind_R < size_F->nR_compact) {
		int ijk[3] = {};
		ijk[0] = blockIdx.y;

		if (dim != 0) {
			ijk[2] = (int) blockIdx.z / size_F->const_2Bx;
			ijk[1] = blockIdx.z % size_F->const_2Bx;
		}

		int ind_dF = ind_R + (ijk[0] + blockIdx.z*size_F->const_2Bx)*size_F->nR_compact;

		myReal y = dF[ind_dF].y;
		dF[ind_dF].y = dF[ind_dF].x * c[ijk[dim]];
		dF[ind_dF].x = -y * c[ijk[dim]];
	}
}

__global__ void mulImg_FTot_large(myComplex* dF, const myReal* c, const int dim, const int k, const Size_F* size_F)
{
	int ind_R = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind_R < size_F->nR_compact) {
		int ijk[3] = {};
		ijk[0] = blockIdx.y;
		ijk[1] = blockIdx.z;
		ijk[2] = k;

		int ind_dF = ind_R + (ijk[0] + ijk[1]*size_F->const_2Bx)*size_F->nR_compact;

		myReal y = dF[ind_dF].y;
		dF[ind_dF].y = dF[ind_dF].x * c[ijk[dim]];
		dF[ind_dF].x = -y * c[ijk[dim]];
	}
}

__global__ void get_c(myReal* c, const int i, const int j, const myReal* L, const myReal* G, const Size_F* size_F)
{
	if (i == j) {
		int ix = threadIdx.x;
		if (ix < size_F[0].Bx)
			c[ix] = -4*PI*PI * ix*ix * G[i+3*j] / (L[0]*L[0]);
		else
			c[ix] = -4*PI*PI * (ix-size_F[0].const_2Bx)*(ix-size_F[0].const_2Bx) * G[i+3*j] / (L[0]*L[0]);
	} else {
		int ix = threadIdx.x;
		int jx = threadIdx.y;

		myReal c1;
		if (ix < size_F[0].Bx)
			c1 = 2*PI * ix / L[0];
		else if (ix == size_F[0].Bx)
			c1 = 0;
		else
			c1 = 2*PI * (ix-size_F[0].const_2Bx) / L[0];

		myReal c2;
		if (jx < size_F[0].Bx)
			c2 = 2*PI * jx / L[0];
		else if (jx == size_F[0].Bx)
			c2 = 0;
		else
			c2 = 2*PI * (jx-size_F[0].const_2Bx) / L[0];

		int indc = ix + jx*size_F[0].const_2Bx;
		c[indc] = -c1*c2 * G[i+3*j];
	}
}

__global__ void get_biasRW_small(myComplex* dF_temp, const myComplex* Fold, const myReal* c, const int i, const int j, const Size_F* size_F)
{
	int indR = threadIdx.x + blockIdx.x*blockDim.x;
	if (indR < size_F[0].nR_compact) {
		unsigned int indx = blockIdx.y;
		int ijk[3];

		ijk[2] = (int) indx / size_F[0].const_2Bxs;
		int ijx = indx % size_F[0].const_2Bxs;
		ijk[1] = (int) ijx / size_F[0].const_2Bx;
		ijk[0] = ijx % size_F[0].const_2Bx;

		int ind = indR + indx*size_F[0].nR_compact;

		if (i==j) {
			dF_temp[ind].x = Fold[ind].x * c[ijk[i]];
			dF_temp[ind].y = Fold[ind].y * c[ijk[i]];
		} else {
			int indc = ijk[i] + ijk[j]*size_F[0].const_2Bx;
			dF_temp[ind].x = Fold[ind].x * c[indc];
			dF_temp[ind].y = Fold[ind].y * c[indc];
		}
	}
}

__global__ void integrate_Fnew(myComplex* Fnew, const myComplex* Fold, const myComplex* dF, const myReal dt, const int nTot)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < nTot)
	{
		Fnew[ind].x = Fold[ind].x + dt*dF[ind].x;
		Fnew[ind].y = Fold[ind].y + dt*dF[ind].y;
	}
}

__host__ void modify_F(const myComplex* F, myComplex* F_modify, bool reduce,Size_F* size_F)
{
	if (reduce) {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F_reduced] = F[ind_F];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	} else {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F] = F[ind_F_reduced];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	}
}

__host__ void permute_F(myComplex* F, bool R_first, const Size_F* size_F)
{
	myComplex* Fp = new myComplex[size_F->nTot_compact];
	if (R_first) {
		for (int iR = 0; iR < size_F->nR_compact; iR++) {
			for (int i = 0; i < size_F->const_2Bx; i++) {
				for (int j = 0; j < size_F->const_2Bx; j++) {
					for (int k = 0; k < size_F->const_2Bx; k++) {
						int ind_F = i + j*size_F->const_2Bx + k*size_F->const_2Bxs + iR*size_F->nx;
						int ind_Fp = iR + (i + j*size_F->const_2Bx + k*size_F->const_2Bxs) * size_F->nR_compact;

						Fp[ind_Fp] = F[ind_F];
					}
				}
			}
		}
	} else {
		for (int iR = 0; iR < size_F->nR_compact; iR++) {
			for (int i = 0; i < size_F->const_2Bx; i++) {
				for (int j = 0; j < size_F->const_2Bx; j++) {
					for (int k = 0; k < size_F->const_2Bx; k++) {
						int ind_F = iR + (i + j*size_F->const_2Bx + k*size_F->const_2Bxs) * size_F->nR_compact;
						int ind_Fp = i + j*size_F->const_2Bx + k*size_F->const_2Bxs + iR*size_F->nx;

						Fp[ind_Fp] = F[ind_F];
					}
				}
			}
		}
	}

	memcpy(F, Fp, size_F->nTot_compact * sizeof(myComplex));
	delete[] Fp;
}

__host__ void modify_u(const myComplex* u, myComplex* u_modify, Size_F* size_F)
{
	int ind_u_reduced = 0;
	for (int i = 0; i < 3; i++) {
		for (int l = 0; l <= size_F[0].lmax; l++) {
			for (int m = -l; m <= l; m++) {
				for (int n = -l; n <= l; n++) {
					int ind_u = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + l*size_F[0].l_cum1 + i*size_F[0].l_cum2;
					u_modify[ind_u_reduced] = u[ind_u];

					ind_u_reduced++;
				}
			}
		}
	}
}

__host__ void deriv_x(myReal* c, const int n, const int B, const myReal L)
{
	if (n < B)
		*c = 2*PI*n/L;
	else if (n == B)
		*c = 0;
	else
		*c = 2*PI*(n-2*B)/L;
}

__host__ void get_dF_small(myComplex* dF, const myComplex* F, const myComplex* X, const myComplex* OJO, const myComplex* MR, const myReal* b,
	const myReal* G, const myReal* L, const myComplex* u, const myReal* const* CG, const Size_F* size_F, const Size_F* size_F_dev)
{
	////////////////////////////
	// circular_convolution X //
	////////////////////////////

	// X_ijk = flip(flip(flip(X,1),2),3)
	// X_ijk = circshift(X_ijk,1,i)
	// X_ijk = circshift(X_ijk,2,j)
	// X_ijk = circshift(X_ijk,3,k)
	// dF{r,i,j,k,p} = F{r,m,n,l}.*X_ijk{m,n,l,p}
	// dF(indmn,indmn,l,i,j,k,p) = -dF(indmn,indmn,l,i,j,k,p)*u(indmn,indmn,l,p)'
	// dF = sum(dF,'p')

	// set up arrays
	myComplex* F_dev;
	cudaErrorHandle(hipMalloc(&F_dev, size_F->nTot_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(F_dev, F, size_F->nTot_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* X_dev;
	cudaErrorHandle(hipMalloc(&X_dev, 3*size_F->nx*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F->nx*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* X_ijk_dev;
	cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F->nx*sizeof(myComplex)));

	myComplex* dF3_dev;
	cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F->nTot_compact*sizeof(myComplex)));

	myComplex* dF_temp_dev;
	cudaErrorHandle(hipMalloc(&dF_temp_dev, 3*size_F->nR_compact*sizeof(myComplex)));

	myComplex* u_dev;
	cudaErrorHandle(hipMalloc(&u_dev, 3*size_F->nR_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(u_dev, u, 3*size_F->nR_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* dF_dev;
	cudaErrorHandle(hipMalloc(&dF_dev, size_F->nTot_compact*sizeof(myComplex)));

	// set up cublas
	hipblasHandle_t handle_cublas;
	hipblasCreate(&handle_cublas);

	myComplex alpha_cublas = make_myComplex(1,0);
	myComplex beta_cublas = make_myComplex(0,0);

	// set up cutensor
	hiptensorHandle_t handle_cutensor;
	cutensorInit(&handle_cutensor);

	hiptensorContractionPlan_t plan_conv;
	size_t worksize_conv;

	cutensor_initConv(&handle_cutensor, &plan_conv, &worksize_conv, F_dev, X_ijk_dev, dF_temp_dev, size_F->nR_compact, true, size_F);

	void* work = nullptr;
	if (worksize_conv > 0)
		cudaErrorHandle(hipMalloc(&work, worksize_conv));

	myComplex alpha_cutensor = make_myComplex(0-(myReal)1/size_F->nx,0);
	myComplex beta_cutensor = make_myComplex(0,0);

	// set up blocksize and gridsize
	dim3 blocksize_8(8, 8, 8);
	int gridnum_8 = (int) size_F->const_2Bx/8 + 1;
	dim3 gridsize_8(gridnum_8, gridnum_8, gridnum_8);

	dim3 blocksize_512_nTot(512, 1, 1);
	dim3 gridsize_512_nTot((int)size_F->nTot_compact/512+1, 1, 1);

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			for (int k = 0; k < size_F->const_2Bx; k++) {
				flip_shift <<<gridsize_8, blocksize_8>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
				cudaErrorHandle(hipGetLastError());

				cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, X_ijk_dev,
					(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

				for (int n = 0; n < 3; n++) {
					myComplex* dF3_dev_ijkn = dF3_dev + i*size_F->nR_compact + 
						j*(size_F->nR_compact*size_F->const_2Bx) + k*(size_F->nR_compact*size_F->const_2Bxs) + n*size_F->nTot_compact;
					myComplex* dF_temp_dev_n = dF_temp_dev + n*size_F->nR_compact;

					cudaErrorHandle(hipMemcpy(dF3_dev_ijkn, dF_temp_dev_n, size_F->nR_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));
				}
			}
		}
	}

	for (int ip = 0; ip < 3; ip++) {
		for (int l = 0; l <= size_F->lmax; l++)
		{
			int ind_dF = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nTot_compact;
			long long int stride_Fnew = size_F->nR_compact;

			int ind_u = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nR_compact;
			long long int stride_u = 0;

			cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
				&alpha_cublas, dF3_dev+ind_dF, 2*l+1, stride_Fnew,
				u_dev+ind_u, 2*l+1, stride_u,
				&beta_cublas, dF3_dev+ind_dF, 2*l+1, stride_Fnew, size_F->nx));
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	cudaErrorHandle(hipMemcpy(dF_dev, dF3_dev, size_F->nTot_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));

	// free memory
	cudaErrorHandle(hipFree(u_dev));

	//////////////////////////////
	// circular convolution OJO //
	//////////////////////////////

	// OJO_ijk = flip(flip(flip(OJO,1),2),3)
	// OJO_ijk = circshift(OJO_ijk,1,i)
	// OJO_ijk = circshift(OJO_ijk,2,j)
	// OJO_ijk = circshift(OJO_ijk,3,k)
	// dF{r,i,j,k,p} = Fold{r,m,n,l}.*OJO_ijk{m,n,l,p}
	// dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)
	// dF = sum(dF,'p')

	// set up arrays
	myComplex* OJO_dev;
	cudaErrorHandle(hipMalloc(&OJO_dev, 3*size_F->nx*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(OJO_dev, OJO, 3*size_F->nx*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* OJO_ijk_dev;
	cudaErrorHandle(hipMalloc(&OJO_ijk_dev, 3*size_F->nx*sizeof(myComplex)));

	// set up blocksize and gridsize
	dim3 blocksize_512_nR(512, 1, 1);
	dim3 gridsize_512_nR((int)size_F->nR_compact/512+1, 1, 1);

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			for (int k = 0; k < size_F->const_2Bx; k++) {
				flip_shift <<<gridsize_8, blocksize_8>>> (OJO_dev, OJO_ijk_dev, i, j, k, size_F_dev);
				cudaErrorHandle(hipGetLastError());

				cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, OJO_ijk_dev,
					(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

				myReal c[3];
				deriv_x(c, i, size_F->Bx, *L);
				deriv_x(c+1, j, size_F->Bx, *L);
				deriv_x(c+2, k, size_F->Bx, *L);

				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev, c[0], size_F->nR_compact);
				cudaErrorHandle(hipGetLastError());
				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+size_F->nR_compact, c[1], size_F->nR_compact);
				cudaErrorHandle(hipGetLastError());
				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+2*size_F->nR_compact, c[2], size_F->nR_compact);
				cudaErrorHandle(hipGetLastError());

				for (int ip = 0; ip < 3; ip++) {
					myComplex* dF3_dev_ijkp = dF3_dev + i*size_F->nR_compact + 
						j*(size_F->nR_compact*size_F->const_2Bx) + k*(size_F->nR_compact*size_F->const_2Bxs) + ip*size_F->nTot_compact;
					myComplex* dF_temp_dev_p = dF_temp_dev + ip*size_F->nR_compact;

					cudaErrorHandle(hipMemcpy(dF3_dev_ijkp, dF_temp_dev_p, size_F->nR_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));
				}
			}
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(OJO_dev));
	cudaErrorHandle(hipFree(OJO_ijk_dev));

	//////////////////////////////
	// circular convolutions bX //
	//////////////////////////////

	// bX_ijk = flip(flip(flip(-b*X,1),2),3)
	// bX_ijk = circshift(bX_ijk,1,i)
	// bX_ijk = circshift(bX_ijk,2,j)
	// bX_ijk = circshift(bX_ijk,3,k)
	// dF{r,i,j,k,p} = Fold{r,m,n,l}.*bX_ijk{m,n,l,p}
	// dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)
	// dF = sum(dF,'p')

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			for (int k = 0; k < size_F->const_2Bx; k++) {
				flip_shift <<<gridsize_8, blocksize_8>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
				cudaErrorHandle(hipGetLastError());

				cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, X_ijk_dev,
					(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

				myReal c[3];
				deriv_x(c, i, size_F->Bx, *L);
				deriv_x(c+1, j, size_F->Bx, *L);
				deriv_x(c+2, k, size_F->Bx, *L);

				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev, -c[0]*b[0], size_F->nR_compact);
				cudaErrorHandle(hipGetLastError());
				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+size_F->nR_compact, -c[1]*b[1], size_F->nR_compact);
				cudaErrorHandle(hipGetLastError());
				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+2*size_F->nR_compact, -c[2]*b[2], size_F->nR_compact);
				cudaErrorHandle(hipGetLastError());

				for (int ip = 0; ip < 3; ip++) {
					myComplex* dF3_dev_ijkp = dF3_dev + i*size_F->nR_compact + 
						j*(size_F->nR_compact*size_F->const_2Bx) + k*(size_F->nR_compact*size_F->const_2Bxs) + ip*size_F->nTot_compact;
					myComplex* dF_temp_dev_p = dF_temp_dev + ip*size_F->nR_compact;

					cudaErrorHandle(hipMemcpy(dF3_dev_ijkp, dF_temp_dev_p, size_F->nR_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));
				}
			}
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(X_dev));
	cudaErrorHandle(hipFree(X_ijk_dev));
	cudaErrorHandle(hipFree(dF_temp_dev));
	if (worksize_conv > 0)
		cudaErrorHandle(hipFree(work));

	///////////////////////
	// kronecker product //
	///////////////////////

	// set up arrays
	myComplex** CG_dev = new myComplex* [size_F->BR*size_F->BR];
	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		for (int l2 = 0; l2 <= size_F->lmax; l2++) {
			int m = (2*l1+1)*(2*l2+1);
			int ind_CG = l1+l2*size_F->BR;
			cudaErrorHandle(hipMalloc(&CG_dev[ind_CG], m*m*sizeof(myComplex)));
			cudaErrorHandle(hipMemset(CG_dev[ind_CG], 0, m*m*sizeof(myComplex)));

			myReal* CG_dev_d = (myReal*) CG_dev[ind_CG];
			cudaErrorHandle(hipMemcpy2D(CG_dev_d, 2*sizeof(myReal), CG[ind_CG], sizeof(myReal), sizeof(myReal), m*m, hipMemcpyHostToDevice));
		}
	}

	myComplex** F_strided = new myComplex* [size_F->BR];
	for (int l = 0; l <= size_F->lmax; l++) {
		int ind = l*(2*l-1)*(2*l+1)/3;
		int m = (2*l+1)*(2*l+1);
		cudaErrorHandle(hipMalloc(&F_strided[l], m*size_F->nx*sizeof(myComplex)));
		cudaErrorHandle(hipMemcpy2D(F_strided[l], m*sizeof(myComplex), F+ind, size_F->nR_compact*sizeof(myComplex),
			m*sizeof(myComplex), size_F->nx, hipMemcpyHostToDevice));
	}

	myComplex* MR_dev;
	cudaErrorHandle(hipMalloc(&MR_dev, 3*size_F->nR_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(MR_dev, MR, 3*size_F->nR_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* FMR_dev;
	int m = (2*size_F->lmax+1) * (2*size_F->lmax+1);
	cudaErrorHandle(hipMalloc(&FMR_dev, 3*m*sizeof(myComplex)));

	myComplex* FMR_temp_dev;
	cudaErrorHandle(hipMalloc(&FMR_temp_dev, 3*size_F->nx*sizeof(myComplex)));

	cudaErrorHandle(hipMemset(dF3_dev, 0, 3*size_F->nTot_compact*sizeof(myComplex)));

	// get c
	myReal* c = new myReal[size_F->const_2Bx];
	for (int i = 0; i < size_F->const_2Bx; i++) {
		deriv_x(&c[i], i, size_F->Bx, *L);
	}

	myReal* c_dev;
	cudaErrorHandle(hipMalloc(&c_dev, size_F->const_2Bx*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(c_dev, c, size_F->const_2Bx*sizeof(myReal), hipMemcpyHostToDevice));

	// set up cutensor
	hiptensorContractionPlan_t* plan_FMR = new hiptensorContractionPlan_t [size_F->BR];
	size_t* worksize_FMR = new size_t [size_F->BR];

	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		cutensor_initFMR(&handle_cutensor, &plan_FMR[l1], &worksize_FMR[l1], F_strided[l1], FMR_dev, FMR_temp_dev, l1, true, size_F);
	}

	size_t worksize_FMR_max = 0;
	for (int l = 0; l <= size_F->lmax; l++) {
		worksize_FMR_max = (worksize_FMR[l] > worksize_FMR_max) ? worksize_FMR[l] : worksize_FMR_max;
	}

	if (worksize_FMR_max > 0) {
		cudaErrorHandle(hipMalloc(&work, worksize_FMR_max));
	}

	// set up blocksize and gridsize
	dim3 blocksize_addMFR(size_F->const_2Bx, size_F->const_2Bx, 1);
	dim3 gridsize_addMFR(size_F->const_2Bx, 3, 1);

	dim3 blocksize_deriv(512,1,1);
	dim3 gridsize_deriv((int)size_F->nR_compact/512+1, size_F->const_2Bx, size_F->const_2Bxs);

	// calculate
	for (int l = 0; l <= size_F->lmax; l++) {
		int ind_cumR = l*(2*l-1)*(2*l+1)/3;

		for (int l1 = 0; l1 <= size_F->lmax; l1++) {
			for (int l2 = 0; l2 <= size_F->lmax; l2++) {
				if (abs(l1-l2)<=l && l1+l2>=l) {
					int ind_MR = l2*(2*l2-1)*(2*l2+1)/3;
					int ind_CG = l1+l2*size_F->BR;
					int l12 = (2*l1+1)*(2*l2+1);

					alpha_cutensor.x = (myReal) -l12/(2*l+1);

					for (int m = -l; m <= l; m++) {
						int ind_CG_m = (l*l-(l1-l2)*(l1-l2)+m+l)*l12;

						for (int n = -l; n <= l; n++) {
							int ind_CG_n = (l*l-(l1-l2)*(l1-l2)+n+l)*l12;
							int ind_mnl = m+l + (n+l)*(2*l+1) + ind_cumR;

							cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, 2*l1+1, 2*l2+1, 2*l2+1,
								&alpha_cublas, CG_dev[ind_CG]+ind_CG_m, 2*l2+1, 0, MR_dev+ind_MR, 2*l2+1, size_F->nR_compact,
								&beta_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), 3));

							cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, 2*l1+1, 2*l1+1, 2*l2+1,
								&alpha_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), CG_dev[ind_CG]+ind_CG_n, 2*l2+1, 0,
								&beta_cublas, FMR_dev, 2*l1+1, (2*l1+1)*(2*l1+1), 3));

							cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_FMR[l1], &alpha_cutensor, F_strided[l1],
								FMR_dev, &beta_cutensor, FMR_temp_dev, FMR_temp_dev, work, worksize_FMR[l1], 0));

							add_FMR_small <<<gridsize_addMFR, blocksize_addMFR>>> (dF3_dev, FMR_temp_dev, ind_mnl, size_F_dev);
						}
					}
				}
			}
		}
	}

	for (int i = 0; i < 3; i++) {
		mulImg_FTot_small <<<gridsize_deriv, blocksize_deriv>>> (dF3_dev+i*size_F->nTot_compact, c_dev, i, size_F_dev);
		cudaErrorHandle(hipGetLastError());
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(MR_dev));
	cudaErrorHandle(hipFree(FMR_dev));
	cudaErrorHandle(hipFree(FMR_temp_dev));
	cudaErrorHandle(hipFree(c_dev));

	if (worksize_FMR_max > 0) {
		cudaErrorHandle(hipFree(work));
	}

	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		for (int l2 = 0; l2 <= size_F->lmax; l2++) {
			int ind_CG = l1+l2*size_F->BR;
			cudaErrorHandle(hipFree(CG_dev[ind_CG]));
		}
	}

	for (int l = 0; l <= size_F->lmax; l++) {
		cudaErrorHandle(hipFree(F_strided[l]));
	}

	delete[] c;
	delete[] plan_FMR;
	delete[] worksize_FMR;
	delete[] CG_dev;
	delete[] F_strided;

	///////////////////////
	// random walk noise //
	///////////////////////

	// set up arrays
	cudaErrorHandle(hipMalloc(&c_dev, size_F->const_2Bxs*sizeof(myReal)));

	myReal* G_dev;
	cudaErrorHandle(hipMalloc(&G_dev, 9*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(G_dev, G, 9*sizeof(myReal), hipMemcpyHostToDevice));

	myReal* L_dev;
	cudaErrorHandle(hipMalloc(&L_dev, sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(L_dev, L, sizeof(myReal), hipMemcpyHostToDevice));

	// set up block and grid sizes
	dim3 blocksize_512_nR_nx(512, 1, 1);
	dim3 gridsize_512_nR_nx((int)size_F->nR_compact/512+1, size_F->nx, 1);

	// calculate
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			if (i == j) {
				dim3 blocksize_c(size_F->const_2Bx, 1, 1);
				get_c <<<1, blocksize_c>>> (c_dev, i, j, L_dev, G_dev, size_F_dev);
				cudaErrorHandle(hipGetLastError());
			}
			else {
				dim3 blocksize_c(size_F->const_2Bx, size_F->const_2Bx, 1);
				get_c <<<1, blocksize_c>>> (c_dev, i, j, L_dev, G_dev, size_F_dev);
				cudaErrorHandle(hipGetLastError());
			}

			get_biasRW_small <<<gridsize_512_nR_nx, blocksize_512_nR_nx>>> (dF3_dev, F_dev, c_dev, i, j, size_F_dev);
			cudaErrorHandle(hipGetLastError());

			add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F->nTot_compact);
			cudaErrorHandle(hipGetLastError());
		}
	}

	// free memory
	cudaErrorHandle(hipFree(c_dev));
	cudaErrorHandle(hipFree(G_dev));
	cudaErrorHandle(hipFree(L_dev));
	cudaErrorHandle(hipFree(F_dev));
	cudaErrorHandle(hipFree(dF3_dev));

	// return
	cudaErrorHandle(hipMemcpy(dF, dF_dev, size_F->nTot_compact*sizeof(myComplex), hipMemcpyDeviceToHost));

	cudaErrorHandle(hipFree(dF_dev));
}

__host__ void get_dF_large(myComplex* dF, myComplex* F, const myComplex* X, const myComplex* OJO, const myComplex* MR,
	const myReal* L, const myComplex* u, const myReal* const* CG, const Size_F* size_F, const Size_F* size_F_dev)
{
	////////////////////////////
	// circular_convolution X //
	////////////////////////////

	// X_ijk = flip(flip(flip(X,1),2),3)
	// X_ijk = circshift(X_ijk,1,i)
	// X_ijk = circshift(X_ijk,2,j)
	// X_ijk = circshift(X_ijk,3,k)
	// dF{r,i,j,k,p} = Fold{r,m,n,l}.*X_ijk{m,n,l,p}
	// dF(indmn,indmn,l,i,j,k,p) = -dF(indmn,indmn,l,i,j,k,p)*u(indmn,indmn,l,p)'
	// dF = sum(dF,'p')

	// set up GPU arrays
	myComplex* F_dev;
	if (size_F->nx*size_F->nR_split > size_F->nTot_splitx) {
		cudaErrorHandle(hipMalloc(&F_dev, size_F->nx*size_F->nR_split*sizeof(myComplex)));
	} else {
		cudaErrorHandle(hipMalloc(&F_dev, size_F->nTot_splitx*sizeof(myComplex)));
	}

	myComplex* X_dev;
	cudaErrorHandle(hipMalloc(&X_dev, 3*size_F->nx*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F->nx*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* X_ijk_dev;
	cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F->nx*sizeof(myComplex)));

	myComplex* dF3_dev;
	cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F->nTot_splitx*sizeof(myComplex)));

	myComplex* u_dev;
	cudaErrorHandle(hipMalloc(&u_dev, 3*size_F->nR_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(u_dev, u, 3*size_F->nR_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	// set up CPU arrays
	permute_F(F, false, size_F);

	myComplex* dF3 = new myComplex[3*size_F->nTot_compact];

	// set up cutensor
	hiptensorHandle_t handle_cutensor;
	cutensorInit(&handle_cutensor);

	hiptensorContractionPlan_t plan_conv[2];
	size_t worksize_conv[2] = {0,0};

	cutensor_initConv(&handle_cutensor, &plan_conv[0], &worksize_conv[0], F_dev, X_ijk_dev, dF3_dev, size_F->nR_split, false, size_F);
	cutensor_initConv(&handle_cutensor, &plan_conv[1], &worksize_conv[1], F_dev, X_ijk_dev, dF3_dev, size_F->nR_remainder, false, size_F);

	void* cutensor_workspace = nullptr;
	size_t worksize_max = worksize_conv[0]>worksize_conv[1] ? worksize_conv[0] : worksize_conv[1];
	if (worksize_max > 0) {
		cudaErrorHandle(hipMalloc(&cutensor_workspace, worksize_max));
	}

	myComplex alpha_cutensor = make_myComplex(-(myReal)1/size_F->nx,0);
	myComplex beta_cutensor = make_myComplex(0,0);

	// set up cublas
	hipblasHandle_t handle_cublas;
	hipblasCreate(&handle_cublas);

	myComplex alpha_cublas = make_myComplex(1,0);
	myComplex beta_cublas = make_myComplex(0,0);

	// set up blocksize and gridsize
	dim3 blocksize_8(8, 8, 8);
	int gridnum_8 = (int)size_F->const_2Bx/8+1;
	dim3 gridsize_8(gridnum_8, gridnum_8, gridnum_8);

	dim3 blocksize_512_nTot(512, 1, 1);
	dim3 gridsize_512_nTot((int)size_F->nTot_splitx/512+1, 1, 1);

	// calculate
	for (int is = 0; is < size_F->ns; is++) {
		int nR_split;
		if (is == size_F->ns-1)
			nR_split = size_F->nR_remainder;
		else
			nR_split = size_F->nR_split;

		cudaErrorHandle(hipMemcpy(F_dev, F+is*size_F->nx*size_F->nR_split, size_F->nx*nR_split*sizeof(myComplex), hipMemcpyHostToDevice));

		for (int i = 0; i < size_F->const_2Bx; i++) {
			for (int j = 0; j < size_F->const_2Bx; j++) {
				for (int k = 0; k < size_F->const_2Bx; k++) {
					flip_shift <<<gridsize_8, blocksize_8>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
					cudaErrorHandle(hipGetLastError());

					if (is == size_F->ns-1) {
						cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[1], &alpha_cutensor, F_dev, X_ijk_dev,
							&beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[1], 0));
					} else {
						cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[0], &alpha_cutensor, F_dev, X_ijk_dev,
							&beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[0], 0));
					}

					for (int ip = 0; ip < 3; ip++) {
						int ind_dF3 = is*size_F->nR_split + (i + j*size_F->const_2Bx + k*size_F->const_2Bxs)*size_F->nR_compact + ip*size_F->nTot_compact;
						hipMemcpy(dF3+ind_dF3, dF3_dev+ip*nR_split, nR_split*sizeof(myComplex), hipMemcpyDeviceToHost);
					}
				}
			}
		}
	}

	// set up CPU arrays
	permute_F(F, true, size_F);

	// multiply u
	for (int k = 0; k < size_F->const_2Bx; k++) {
		for (int ip = 0; ip < 3; ip++) {
			int ind_dF3 = k*size_F->nTot_splitx + ip*size_F->nTot_compact;
			int ind_dF3_dev = ip*size_F->nTot_splitx;

			cudaErrorHandle(hipMemcpy(F_dev, dF3+ind_dF3, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyHostToDevice));

			for (int l = 0; l <= size_F->lmax; l++)
			{
				int ind_dF = l*(2*l-1)*(2*l+1)/3;
				long long int stride_dF = size_F->nR_compact;

				int ind_u = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nR_compact;
				long long int stride_u = 0;

				cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
					&alpha_cublas, F_dev+ind_dF, 2*l+1, stride_dF,
					u_dev+ind_u, 2*l+1, stride_u,
					&beta_cublas, dF3_dev+ind_dF3_dev+ind_dF, 2*l+1, stride_dF, size_F->const_2Bxs));
			}
		}

		// addup dF
		addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_splitx);
		cudaErrorHandle(hipGetLastError());

		int ind_dF = k*size_F->nTot_splitx;
		cudaErrorHandle(hipMemcpy(dF+ind_dF, dF3_dev, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyDeviceToHost));
	}

	// free memory
	cudaErrorHandle(hipFree(X_dev));
	cudaErrorHandle(hipFree(X_ijk_dev));
	cudaErrorHandle(hipFree(u_dev));

	//////////////////////////////
	// circular convolution OJO //
	//////////////////////////////

	// OJO_ijk = flip(flip(flip(OJO,1),2),3)
	// OJO_ijk = circshift(OJO_ijk,1,i)
	// OJO_ijk = circshift(OJO_ijk,2,j)
	// OJO_ijk = circshift(OJO_ijk,3,k)
	// dF{r,i,j,k,p} = Fold{r,m,n,l}.*OJO_ijk{m,n,l,p}
	// dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)
	// dF = sum(dF,'p')

	// set up GPU arrays
	myComplex* OJO_dev;
	cudaErrorHandle(hipMalloc(&OJO_dev, 3*size_F->nx*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(OJO_dev, OJO, 3*size_F->nx*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* OJO_ijk_dev;
	cudaErrorHandle(hipMalloc(&OJO_ijk_dev, 3*size_F->nx*sizeof(myComplex)));

	// set up CPU arrays
	permute_F(F, false, size_F);

	// set up blocksize and gridsize
	dim3 blocksize_512_nR(512, 1, 1);
	dim3 gridsize_512_nR((int)size_F->nR_split/512+1, 1, 1);

	// calculate
	for (int is = 0; is < size_F->ns; is++) {
		int nR_split;
		if (is == size_F->ns-1) {
			nR_split = size_F->nR_remainder;
		} else {
			nR_split = size_F->nR_split;
		}

		gridsize_512_nR.x = (int)nR_split/512+1;

		cudaErrorHandle(hipMemcpy(F_dev, F+is*size_F->nx*size_F->nR_split, size_F->nx*nR_split*sizeof(myComplex), hipMemcpyHostToDevice));

		for (int i = 0; i < size_F->const_2Bx; i++) {
			for (int j = 0; j < size_F->const_2Bx; j++) {
				for (int k = 0; k < size_F->const_2Bx; k++) {
					flip_shift <<<gridsize_8, blocksize_8>>> (OJO_dev, OJO_ijk_dev, i, j, k, size_F_dev);
					cudaErrorHandle(hipGetLastError());

					if (is == size_F->ns-1) {
						cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[1], &alpha_cutensor, F_dev, OJO_ijk_dev,
							&beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[1], 0));
					} else {
						cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv[0], &alpha_cutensor, F_dev, OJO_ijk_dev,
							&beta_cutensor, dF3_dev, dF3_dev, cutensor_workspace, worksize_conv[0], 0));
					}

					myReal c[3];
					deriv_x(c, i, size_F->Bx, *L);
					deriv_x(c+1, j, size_F->Bx, *L);
					deriv_x(c+2, k, size_F->Bx, *L);

					mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF3_dev, c[0], nR_split);
					cudaErrorHandle(hipGetLastError());
					mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF3_dev+nR_split, c[1], nR_split);
					cudaErrorHandle(hipGetLastError());
					mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF3_dev+2*nR_split, c[2], nR_split);
					cudaErrorHandle(hipGetLastError());

					for (int ip = 0; ip < 3; ip++) {
						int ind_dF3 = is*size_F->nR_split + (i + j*size_F->const_2Bx + k*size_F->const_2Bxs)*size_F->nR_compact + ip*size_F->nTot_compact;
						hipMemcpy(dF3+ind_dF3, dF3_dev+ip*nR_split, nR_split*sizeof(myComplex), hipMemcpyDeviceToHost);
					}
				}
			}
		}
	}

	for (int k = 0; k < size_F->const_2Bx; k++) {
		for (int ip = 0; ip < 3; ip++) {
			int ind_dF3 = k*size_F->nTot_splitx + ip*size_F->nTot_compact;
			int ind_dF3_dev = ip*size_F->nTot_splitx;

			cudaErrorHandle(hipMemcpy(dF3_dev+ind_dF3_dev, dF3+ind_dF3, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyHostToDevice));
		}

		addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_splitx);
		cudaErrorHandle(hipGetLastError());

		int ind_dF = k*size_F->nTot_splitx;
		cudaErrorHandle(hipMemcpy(dF3_dev+size_F->nTot_splitx, dF+ind_dF, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyHostToDevice));

		add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev+size_F->nTot_splitx, dF3_dev, size_F->nTot_splitx);
		cudaErrorHandle(hipGetLastError());

		cudaErrorHandle(hipMemcpy(dF+ind_dF, dF3_dev+size_F->nTot_splitx, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyDeviceToHost));
	}

	// free memory
	cudaErrorHandle(hipFree(OJO_dev));
	cudaErrorHandle(hipFree(OJO_ijk_dev));
	cudaErrorHandle(hipFree(F_dev));

	if (worksize_max > 0) {
		cudaErrorHandle(hipFree(cutensor_workspace));
	}

	///////////////////////
	// kronecker product //
	///////////////////////

	// set up GPU arrays
	myComplex** CG_dev = new myComplex* [size_F->BR*size_F->BR];
	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		for (int l2 = 0; l2 <= size_F->lmax; l2++) {
			int m = (2*l1+1)*(2*l2+1);
			int ind_CG = l1+l2*size_F->BR;
			cudaErrorHandle(hipMalloc(&CG_dev[ind_CG], m*m*sizeof(myComplex)));
			cudaErrorHandle(hipMemset(CG_dev[ind_CG], 0, m*m*sizeof(myComplex)));

			myReal* CG_dev_d = (myReal*) CG_dev[ind_CG];
			cudaErrorHandle(hipMemcpy2D(CG_dev_d, 2*sizeof(myReal), CG[ind_CG], sizeof(myReal), sizeof(myReal), m*m, hipMemcpyHostToDevice));
		}
	}

	myComplex* MR_dev;
	cudaErrorHandle(hipMalloc(&MR_dev, 3*size_F->nR_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(MR_dev, MR, 3*size_F->nR_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* FMR_dev;
	int m = (2*size_F->lmax+1) * (2*size_F->lmax+1);
	cudaErrorHandle(hipMalloc(&FMR_dev, 3*m*sizeof(myComplex)));

	myComplex* FMR_temp_dev;
	cudaErrorHandle(hipMalloc(&FMR_temp_dev, 3*size_F->const_2Bxs*sizeof(myComplex)));

	myComplex** F_strided = new myComplex* [size_F->BR];
	for (int l = 0; l <= size_F->lmax; l++) {
		int m = (2*l+1)*(2*l+1);
		cudaErrorHandle(hipMalloc(&F_strided[l], m*size_F->const_2Bxs*sizeof(myComplex)));
	}

	// set up CPU arrays
	permute_F(F, true, size_F);

	// get c
	myReal* c = new myReal[size_F->const_2Bx];
	for (int i = 0; i < size_F->const_2Bx; i++) {
		deriv_x(&c[i], i, size_F->Bx, *L);
	}

	myReal* c_dev;
	cudaErrorHandle(hipMalloc(&c_dev, size_F->const_2Bx*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(c_dev, c, size_F->const_2Bx*sizeof(myReal), hipMemcpyHostToDevice));

	// set up cutensor
	hiptensorContractionPlan_t* plan_FMR = new hiptensorContractionPlan_t [size_F->BR];
	size_t* worksize_FMR = new size_t [size_F->BR];

	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		cutensor_initFMR(&handle_cutensor, &plan_FMR[l1], &worksize_FMR[l1], F_strided[l1], FMR_dev, FMR_temp_dev, l1, false, size_F);
	}

	worksize_max = 0;
	for (int l = 0; l <= size_F->lmax; l++) {
		worksize_max = (worksize_FMR[l] > worksize_max) ? worksize_FMR[l] : worksize_max;
	}

	if (worksize_max > 0) {
		cudaErrorHandle(hipMalloc(&cutensor_workspace, worksize_max));
	}

	// set up blocksize and gridsize
	dim3 blocksize_addMFR(size_F->const_2Bx, 3, 1);
	dim3 gridsize_addMFR(size_F->const_2Bx, 1, 1);

	dim3 blocksize_deriv(512,1,1);
	dim3 gridsize_deriv((int)size_F->nR_compact/512+1, size_F->const_2Bx, size_F->const_2Bx);

	// calculate
	for (int k = 0; k < size_F->const_2Bx; k++) {
		int ind_Fold = k*size_F->nTot_splitx;
		for (int l = 0; l <= size_F->lmax; l++) {
			int ind = l*(2*l-1)*(2*l+1)/3;
			int m = (2*l+1)*(2*l+1);
			cudaErrorHandle(hipMemcpy2D(F_strided[l], m*sizeof(myComplex), F+ind_Fold+ind, size_F->nR_compact*sizeof(myComplex),
				m*sizeof(myComplex), size_F->const_2Bxs, hipMemcpyHostToDevice));
		}

		cudaErrorHandle(hipMemset(dF3_dev, 0, 3*size_F->nTot_splitx*sizeof(myComplex)));

		for (int l = 0; l <= size_F->lmax; l++) {
			int ind_cumR = l*(2*l-1)*(2*l+1)/3;

			for (int l1 = 0; l1 <= size_F->lmax; l1++) {
				for (int l2 = 0; l2 <= size_F->lmax; l2++) {
					if (abs(l1-l2)<=l && l1+l2>=l) {
						int ind_MR = l2*(2*l2-1)*(2*l2+1)/3;
						int ind_CG = l1+l2*size_F->BR;
						int l12 = (2*l1+1)*(2*l2+1);

						alpha_cutensor.x = (myReal) -l12/(2*l+1);

						for (int m = -l; m <= l; m++) {
							int ind_CG_m = (l*l-(l1-l2)*(l1-l2)+m+l)*l12;

							for (int n = -l; n <= l; n++) {
								int ind_CG_n = (l*l-(l1-l2)*(l1-l2)+n+l)*l12;
								int ind_mnl = m+l + (n+l)*(2*l+1) + ind_cumR;

								cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, 2*l1+1, 2*l2+1, 2*l2+1,
									&alpha_cublas, CG_dev[ind_CG]+ind_CG_m, 2*l2+1, 0, MR_dev+ind_MR, 2*l2+1, size_F->nR_compact,
									&beta_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), 3));

								cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, 2*l1+1, 2*l1+1, 2*l2+1,
									&alpha_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), CG_dev[ind_CG]+ind_CG_n, 2*l2+1, 0,
									&beta_cublas, FMR_dev, 2*l1+1, (2*l1+1)*(2*l1+1), 3));

								cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_FMR[l1], &alpha_cutensor, F_strided[l1],
									FMR_dev, &beta_cutensor, FMR_temp_dev, FMR_temp_dev, cutensor_workspace, worksize_FMR[l1], 0));

								add_FMR_large <<<gridsize_addMFR, blocksize_addMFR>>> (dF3_dev, FMR_temp_dev, ind_mnl, size_F_dev);
							}
						}
					}
				}
			}
		}

		for (int ip = 0; ip < 3; ip++) {
			int ind_dF3 = ind_Fold + ip*size_F->nTot_compact;
			int ind_dF3_dev = ip*size_F->nTot_splitx;
			cudaErrorHandle(hipMemcpy(dF3+ind_dF3, dF3_dev+ind_dF3_dev, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyDeviceToHost));
		}
	}

	for (int k = 0; k < size_F->const_2Bx; k++) {
		// take derivative about x
		for (int ip = 0; ip < 3; ip++) {
			int ind_dF3 = k*size_F->nTot_splitx + ip*size_F->nTot_compact;
			int ind_dF3_dev = ip*size_F->nTot_splitx;

			cudaErrorHandle(hipMemcpy(dF3_dev+ind_dF3_dev, dF3+ind_dF3, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyHostToDevice));

			mulImg_FTot_large <<<gridsize_deriv, blocksize_deriv>>> (dF3_dev+ind_dF3_dev, c_dev, ip, k, size_F_dev);
			cudaErrorHandle(hipGetLastError());
		}

		// addup dF
		addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F->nTot_splitx);
		cudaErrorHandle(hipGetLastError());

		int ind_dF = k*size_F->nTot_splitx;
		cudaErrorHandle(hipMemcpy(dF3_dev+size_F->nTot_splitx, dF+ind_dF, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyHostToDevice));

		add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev+size_F->nTot_splitx, dF3_dev, size_F->nTot_splitx);
		cudaErrorHandle(hipGetLastError());

		cudaErrorHandle(hipMemcpy(dF+ind_dF, dF3_dev+size_F->nTot_splitx, size_F->nTot_splitx*sizeof(myComplex), hipMemcpyDeviceToHost));
	}

	// free memory
	cudaErrorHandle(hipFree(dF3_dev));
	cudaErrorHandle(hipFree(MR_dev));
	cudaErrorHandle(hipFree(FMR_dev));
	cudaErrorHandle(hipFree(FMR_temp_dev));
	cudaErrorHandle(hipFree(c_dev));

	if (worksize_max > 0) {
		cudaErrorHandle(hipFree(cutensor_workspace));
	}

	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		for (int l2 = 0; l2 <= size_F->lmax; l2++) {
			int ind_CG = l1+l2*size_F->BR;
			cudaErrorHandle(hipFree(CG_dev[ind_CG]));
		}
	}

	for (int l = 0; l <= size_F->lmax; l++) {
		cudaErrorHandle(hipFree(F_strided[l]));
	}

	delete[] c;
	delete[] CG_dev;
	delete[] F_strided;
	delete[] dF3;
	delete[] plan_FMR;
	delete[] worksize_FMR;
}

