#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

__global__ void flip_shift(const hipDoubleComplex* X, hipDoubleComplex* X_ijk, const int is, const int js, const int ks, const Size_F* size_F)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if (i < size_F[0].const_2Bx && j < size_F[0].const_2Bx && k < size_F[0].const_2Bx) {
		int iout = is-i;
		if (iout < 0)
			iout += size_F[0].const_2Bx;
		else if (iout >= size_F[0].const_2Bx)
			iout -= size_F[0].const_2Bx;

		int jout = js-j;
		if (jout < 0)
			jout += size_F[0].const_2Bx;
		else if (jout >= size_F[0].const_2Bx)
			jout -= size_F[0].const_2Bx;

		int kout = ks-k;
		if (kout < 0)
			kout += size_F[0].const_2Bx;
		else if (kout >= size_F[0].const_2Bx)
			kout -= size_F[0].const_2Bx;

		int X_ind = i + j*size_F[0].const_2Bx + k*size_F[0].const_2Bxs;
		int X_ijk_ind = iout + jout*size_F[0].const_2Bx + kout*size_F[0].const_2Bxs;

		for (int m = 0; m < 3; m++)
			X_ijk[X_ijk_ind + m*size_F[0].nx] = X[X_ind + m*size_F[0].nx];
	}
}

__global__ void addup_F(hipDoubleComplex* dF, Size_F* size_F)
{
	int ind1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind1 < size_F[0].nTot_compact) {
		int ind2 = ind1 + size_F[0].nTot_compact;
		int ind3 = ind2 + size_F[0].nTot_compact;

		dF[ind1] = hipCadd(dF[ind1], dF[ind2]);
		dF[ind1] = hipCadd(dF[ind1], dF[ind3]);
	}
}

__global__ void add_F(hipDoubleComplex* dF, const hipDoubleComplex* dF_temp, const Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nTot_compact)
		dF[ind] = hipCadd(dF[ind], dF_temp[ind]);
}

__global__ void mulImg_FR(hipDoubleComplex* dF, const double c, const Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nR_compact) {
		double y = dF[ind].y;
		dF[ind].y = dF[ind].x * c;
		dF[ind].x = -y * c;
	}
}

__global__ void kron_FMR(double* FMR_real, double* FMR_imag, const hipDoubleComplex* F, const hipDoubleComplex* MR,
	const int ind_F_cumR, const int ind_MR_cumR, const Size_F* size_F)
{
	int ind_F = threadIdx.x + threadIdx.y*blockDim.x + ind_F_cumR + blockIdx.z*size_F->nR_compact;
	int ind_MR = blockIdx.x + blockIdx.y*gridDim.x + ind_MR_cumR;
	
	int i_FMR = blockIdx.x + threadIdx.x*gridDim.x;
	int j_FMR = blockIdx.y + threadIdx.y*gridDim.y;
	int m = blockDim.x*gridDim.x;
	int ind_FMR = i_FMR + j_FMR*m + blockIdx.z*m*m;

	FMR_real[ind_FMR] = F[ind_F].x*MR[ind_MR].x - F[ind_F].y*MR[ind_MR].y;
	FMR_imag[ind_FMR] = F[ind_F].x*MR[ind_MR].y + F[ind_F].y*MR[ind_MR].x;
}

__global__ void add_FMR(hipDoubleComplex* dF, const double* FMR_real, const double* FMR_imag, const int ind_cumR, const Size_F* size_F)
{
	int ind_dF = threadIdx.x + ind_cumR + blockIdx.x*size_F->nR_compact;
	int ind_FMR = threadIdx.x + blockIdx.x*blockDim.x;

	dF[ind_dF].x += FMR_real[ind_FMR];
	dF[ind_dF].y += FMR_imag[ind_FMR];
}

__global__ void mulImg_FTot(hipDoubleComplex* dF, const double* c, const int dim, const Size_F* size_F)
{
	int ind_R = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind_R < size_F->nR_compact) {
		int ijk[3] = {};
		ijk[0] = blockIdx.y;

		if (dim != 0) {
			ijk[2] = (int) blockIdx.z / size_F->const_2Bx;
			ijk[1] = blockIdx.z % size_F->const_2Bx;
		}

		int ind_dF = ind_R + (ijk[0] + blockIdx.z*size_F->const_2Bx)*size_F->nR_compact;

		double y = dF[ind_dF].y;
		dF[ind_dF].y = dF[ind_dF].x * c[ijk[dim]];
		dF[ind_dF].x = -y * c[ijk[dim]];
	}
}

__global__ void integrate_Fnew(hipDoubleComplex* Fnew, const hipDoubleComplex* Fold, const hipDoubleComplex* dF, const double dt, const Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nTot_compact)
	{
		Fnew[ind].x = Fold[ind].x + dt*dF[ind].x;
		Fnew[ind].y = Fold[ind].y + dt*dF[ind].y;
	}
}

__host__ void modify_F(const hipDoubleComplex* F, hipDoubleComplex* F_modify, bool reduce,Size_F* size_F)
{
	if (reduce) {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F_reduced] = F[ind_F];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	} else {
		int ind_F_reduced = 0;
		for (int k = 0; k < size_F[0].const_2Bx; k++) {
			for (int j = 0; j < size_F[0].const_2Bx; j++) {
				for (int i = 0; i < size_F[0].const_2Bx; i++) {
					for (int l = 0; l <= size_F[0].lmax; l++) {
						for (int m = -l; m <= l; m++) {
							for (int n = -l; n <= l; n++) {
								int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
									l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3 + k*size_F[0].l_cum4;
								F_modify[ind_F] = F[ind_F_reduced];

								ind_F_reduced++;
							}
						}
					}
				}
			}
		}
	}
}

__host__ void modify_u(const hipDoubleComplex* u, hipDoubleComplex* u_modify, Size_F* size_F)
{
	int ind_u_reduced = 0;
	for (int i = 0; i < 3; i++) {
		for (int l = 0; l <= size_F[0].lmax; l++) {
			for (int m = -l; m <= l; m++) {
				for (int n = -l; n <= l; n++) {
					int ind_u = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + l*size_F[0].l_cum1 + i*size_F[0].l_cum2;
					u_modify[ind_u_reduced] = u[ind_u];

					ind_u_reduced++;
				}
			}
		}
	}
}

__host__ void deriv_x(double* c, const int n, const int B, const double L)
{
	if (n < B)
		*c = 2*PI*n/L;
	else if (n == B)
		*c = 0;
	else
		*c = 2*PI*(n-2*B)/L;
}

__host__ void cudaErrorHandle(const hipError_t& err)
{
	if (err != hipSuccess) {
		std::cout << "Cuda Error: " << hipGetErrorString(err) << std::endl;
	}
}

__host__ void cutensorErrorHandle(const hiptensorStatus_t& err)
{
	if (err != HIPTENSOR_STATUS_SUCCESS) {
		std::cout << "cuTensor Error: " << hiptensorGetErrorString(err) << std::endl;
	}
}

__host__ void cublasErrorHandle(const hipblasStatus_t& err)
{
	if (err != HIPBLAS_STATUS_SUCCESS) {
		std::cout << "cuBlas Error: " << err << std::endl;
	}
}

__host__ void cutensor_initialize(hiptensorHandle_t* handle, hiptensorContractionPlan_t* plan, size_t* worksize,
	hipDoubleComplex* Fold_dev, hipDoubleComplex* X_ijk_dev, hipDoubleComplex* dF_temp_dev, Size_F size_F)
{
	int mode_Fold[4] = {'r','i','j','k'};
	int mode_X[4] = {'i','j','k','p'};
	int mode_dF[2] = {'r','p'};

	int64_t extent_Fold[4] = {size_F.nR_compact, size_F.const_2Bx, size_F.const_2Bx, size_F.const_2Bx};
	int64_t extent_X[4] = {size_F.const_2Bx, size_F.const_2Bx, size_F.const_2Bx, 3};
	int64_t extent_dF[2] = {size_F.nR_compact, 3};

	hiptensorTensorDescriptor_t desc_Fold;
	hiptensorTensorDescriptor_t desc_X;
	hiptensorTensorDescriptor_t desc_temp;
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_Fold,
		4, extent_Fold, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_X,
		4, extent_X, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_temp,
		2, extent_dF, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));

	uint32_t alignmentRequirement_Fold;
	uint32_t alignmentRequirement_X;
	uint32_t alignmentRequirement_temp;
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		Fold_dev, &desc_Fold, &alignmentRequirement_Fold));
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		X_ijk_dev, &desc_X, &alignmentRequirement_X));
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		dF_temp_dev, &desc_temp, &alignmentRequirement_temp));

	cutensorContractionDescriptor_t desc;
	cutensorErrorHandle(cutensorInitContractionDescriptor(handle, &desc,
		&desc_Fold, mode_Fold, alignmentRequirement_Fold,
		&desc_X, mode_X, alignmentRequirement_X,
		&desc_temp, mode_dF, alignmentRequirement_temp,
		&desc_temp, mode_dF, alignmentRequirement_temp,
		HIPTENSOR_COMPUTE_DESC_64F));

	cutensorContractionFind_t find;
	cutensorErrorHandle(cutensorInitContractionFind(handle, &find, HIPTENSOR_ALGO_DEFAULT));

	cutensorErrorHandle(cutensorContractionGetWorkspace(handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, worksize));

	cutensorErrorHandle(cutensorInitContractionPlan(handle, plan, &desc, &find, *worksize));
}

__host__ void init_Size_F(Size_F* size_F, int BR, int Bx)
{
	size_F->BR = BR;
	size_F->Bx = Bx;
	size_F->lmax = BR-1;

	size_F->nR = (2*size_F->lmax+1) * (2*size_F->lmax+1) * (size_F->lmax+1);
	size_F->nx = (2*Bx) * (2*Bx) * (2*Bx);
	size_F->nTot = size_F->nR * size_F->nx;

	size_F->nR_compact = (size_F->lmax+1) * (2*size_F->lmax+1) * (2*size_F->lmax+3) / 3;
	size_F->nTot_compact = size_F->nR_compact * size_F->nx;

	size_F->const_2Bx = 2*Bx;
	size_F->const_2Bxs = (2*Bx) * (2*Bx);
	size_F->const_2lp1 = 2*size_F->lmax+1;
	size_F->const_lp1 = size_F->lmax+1;
	size_F->const_2lp1s = (2*size_F->lmax+1) * (2*size_F->lmax+1);

	size_F->l_cum0 = size_F->const_2lp1;
	size_F->l_cum1 = size_F->l_cum0*size_F->const_2lp1;
	size_F->l_cum2 = size_F->l_cum1*size_F->const_lp1;
	size_F->l_cum3 = size_F->l_cum2*size_F->const_2Bx;
	size_F->l_cum4 = size_F->l_cum3*size_F->const_2Bx;
}

