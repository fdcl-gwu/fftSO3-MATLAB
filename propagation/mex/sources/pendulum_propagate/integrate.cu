#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

__global__ void flip_shift(const hipDoubleComplex* X, hipDoubleComplex* X_ijk, const int is, const int js, const int ks, const Size_F* size_F)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if (i < size_F[0].const_2Bx && j < size_F[0].const_2Bx && k < size_F[0].const_2Bx) {
		int iout = is-i;
		if (iout < 0)
			iout += size_F[0].const_2Bx;
		else if (iout >= size_F[0].const_2Bx)
			iout -= size_F[0].const_2Bx;

		int jout = js-j;
		if (jout < 0)
			jout += size_F[0].const_2Bx;
		else if (jout >= size_F[0].const_2Bx)
			jout -= size_F[0].const_2Bx;

		int kout = ks-k;
		if (kout < 0)
			kout += size_F[0].const_2Bx;
		else if (kout >= size_F[0].const_2Bx)
			kout -= size_F[0].const_2Bx;

		int X_ind = i + j*size_F[0].const_2Bx + k*size_F[0].const_2Bxs;
		int X_ijk_ind = iout + jout*size_F[0].const_2Bx + kout*size_F[0].const_2Bxs;

		for (int m = 0; m < 3; m++)
			X_ijk[X_ijk_ind + m*size_F[0].nx] = X[X_ind + m*size_F[0].nx];
	}
}

__global__ void addup_F(hipDoubleComplex* dF, const Size_F* size_F)
{
	int ind1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind1 < size_F[0].nTot_compact) {
		int ind2 = ind1 + size_F[0].nTot_compact;
		int ind3 = ind2 + size_F[0].nTot_compact;

		dF[ind1] = hipCadd(dF[ind1], dF[ind2]);
		dF[ind1] = hipCadd(dF[ind1], dF[ind3]);
	}
}

__global__ void add_F(hipDoubleComplex* dF, const hipDoubleComplex* dF_temp, const Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nTot_compact)
		dF[ind] = hipCadd(dF[ind], dF_temp[ind]);
}

__global__ void mulImg_FR(hipDoubleComplex* dF, const double c, const Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nR_compact) {
		double y = dF[ind].y;
		dF[ind].y = dF[ind].x * c;
		dF[ind].x = -y * c;
	}
}

__global__ void add_FMR(hipDoubleComplex* dF, const hipDoubleComplex* FMR, const int ind_cumR, const Size_F* size_F)
{
	int ind_dF = ind_cumR + (threadIdx.x + threadIdx.y*size_F->const_2Bx + blockIdx.x*size_F->const_2Bxs)*size_F->nR_compact + blockIdx.y*size_F->nTot_compact;
	int ind_FMR = threadIdx.x + threadIdx.y*size_F->const_2Bx + blockIdx.x*size_F->const_2Bxs + blockIdx.y*size_F->nx;

	dF[ind_dF] = hipCadd(dF[ind_dF], FMR[ind_FMR]);
}

__global__ void mulImg_FTot(hipDoubleComplex* dF, const double* c, const int dim, const Size_F* size_F)
{
	int ind_R = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind_R < size_F->nR_compact) {
		int ijk[3] = {};
		ijk[0] = blockIdx.y;

		if (dim != 0) {
			ijk[2] = (int) blockIdx.z / size_F->const_2Bx;
			ijk[1] = blockIdx.z % size_F->const_2Bx;
		}

		int ind_dF = ind_R + (ijk[0] + blockIdx.z*size_F->const_2Bx)*size_F->nR_compact;

		double y = dF[ind_dF].y;
		dF[ind_dF].y = dF[ind_dF].x * c[ijk[dim]];
		dF[ind_dF].x = -y * c[ijk[dim]];
	}
}

__global__ void integrate_Fnew(hipDoubleComplex* Fnew, hipDoubleComplex* Fold, const hipDoubleComplex* dF, const double dt, const Size_F* size_F)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < size_F[0].nTot_compact)
	{
		Fnew[ind].x = Fold[ind].x + dt*dF[ind].x;
		Fnew[ind].y = Fold[ind].y + dt*dF[ind].y;
	}
}

__host__ void deriv_x(double* c, const int n, const int B, const double L)
{
	if (n < B)
		*c = 2*PI*n/L;
	else if (n == B)
		*c = 0;
	else
		*c = 2*PI*(n-2*B)/L;
}

__host__ void get_dF(hipDoubleComplex* dF, const hipDoubleComplex* F, const hipDoubleComplex* X, const hipDoubleComplex* OJO, const hipDoubleComplex* MR,
	const double* L, const hipDoubleComplex* u, const double* const* CG, const Size_F* size_F, const Size_F* size_F_dev)
{
	////////////////////////////
	// circular_convolution X //
	////////////////////////////

	// X_ijk = flip(flip(flip(X,1),2),3)
	// X_ijk = circshift(X_ijk,1,i)
	// X_ijk = circshift(X_ijk,2,j)
	// X_ijk = circshift(X_ijk,3,k)
	// dF{r,i,j,k,p} = F{r,m,n,l}.*X_ijk{m,n,l,p}
	// dF(indmn,indmn,l,i,j,k,p) = -dF(indmn,indmn,l,i,j,k,p)*u(indmn,indmn,l,p)'
	// dF = sum(dF,'p')

	// set up arrays
	hipDoubleComplex* F_dev;
	cudaErrorHandle(hipMalloc(&F_dev, size_F->nTot_compact*sizeof(hipDoubleComplex)));
	cudaErrorHandle(hipMemcpy(F_dev, F, size_F->nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	hipDoubleComplex* X_dev;
	cudaErrorHandle(hipMalloc(&X_dev, 3*size_F->nx*sizeof(hipDoubleComplex)));
	cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F->nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	hipDoubleComplex* X_ijk_dev;
	cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F->nx*sizeof(hipDoubleComplex)));

	hipDoubleComplex* dF3_dev;
	cudaErrorHandle(hipMalloc(&dF3_dev, 3*size_F->nTot_compact*sizeof(hipDoubleComplex)));

	hipDoubleComplex* dF_temp_dev;
	cudaErrorHandle(hipMalloc(&dF_temp_dev, 3*size_F->nR_compact*sizeof(hipDoubleComplex)));

	hipDoubleComplex* u_dev;
	cudaErrorHandle(hipMalloc(&u_dev, 3*size_F->nR_compact*sizeof(hipDoubleComplex)));
	cudaErrorHandle(hipMemcpy(u_dev, u, 3*size_F->nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	hipDoubleComplex* dF_dev;
	cudaErrorHandle(hipMalloc(&dF_dev, size_F->nTot_compact*sizeof(hipDoubleComplex)));

	// set up cublas
	hipblasHandle_t handle_cublas;
	hipblasCreate(&handle_cublas);

	hipDoubleComplex alpha_cublas = make_hipDoubleComplex(1,0);
	hipDoubleComplex beta_cublas = make_hipDoubleComplex(0,0);

	// set up cutensor
	hiptensorHandle_t handle_cutensor;
	cutensorInit(&handle_cutensor);

	hiptensorContractionPlan_t plan_conv;
	size_t worksize_conv;

	cutensor_initConv(&handle_cutensor, &plan_conv, &worksize_conv, F_dev, X_ijk_dev, dF_temp_dev, size_F);

	void* work = nullptr;
	if (worksize_conv > 0)
		cudaErrorHandle(hipMalloc(&work, worksize_conv));

	hipDoubleComplex alpha_cutensor = make_hipDoubleComplex(0-(double)1/size_F->nx,0);
	hipDoubleComplex beta_cutensor = make_hipDoubleComplex(0,0);

	// set up blocksize and gridsize
	dim3 blocksize_8(8, 8, 8);
	int gridnum_8 = (int) size_F->const_2Bx/8 + 1;
	dim3 gridsize_8(gridnum_8, gridnum_8, gridnum_8);

	dim3 blocksize_512_nTot(512, 1, 1);
	dim3 gridsize_512_nTot((int)size_F->nTot_compact/512+1, 1, 1);

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			for (int k = 0; k < size_F->const_2Bx; k++) {
				flip_shift <<<gridsize_8, blocksize_8>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
				cudaErrorHandle(hipGetLastError());

				hipDeviceSynchronize();

				cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, X_ijk_dev,
					(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

				hipDeviceSynchronize();

				for (int n = 0; n < 3; n++) {
					hipDoubleComplex* dF3_dev_ijkn = dF3_dev + i*size_F->nR_compact + 
						j*(size_F->nR_compact*size_F->const_2Bx) + k*(size_F->nR_compact*size_F->const_2Bxs) + n*size_F->nTot_compact;
					hipDoubleComplex* dF_temp_dev_n = dF_temp_dev + n*size_F->nR_compact;

					cudaErrorHandle(hipMemcpy(dF3_dev_ijkn, dF_temp_dev_n, size_F->nR_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));
				}

				hipDeviceSynchronize();
			}
		}
	}

	for (int ip = 0; ip < 3; ip++) {
		for (int l = 0; l <= size_F->lmax; l++)
		{
			int ind_dF = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nTot_compact;
			long long int stride_Fnew = size_F->nR_compact;

			int ind_u = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nR_compact;
			long long int stride_u = 0;

			cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
				&alpha_cublas, dF3_dev+ind_dF, 2*l+1, stride_Fnew,
				u_dev+ind_u, 2*l+1, stride_u,
				&beta_cublas, dF3_dev+ind_dF, 2*l+1, stride_Fnew, size_F->nx));

			hipDeviceSynchronize();
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
	cudaErrorHandle(hipGetLastError());

	cudaErrorHandle(hipMemcpy(dF_dev, dF3_dev, size_F->nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

	// free memory
	cudaErrorHandle(hipFree(X_dev));
	cudaErrorHandle(hipFree(X_ijk_dev));
	cudaErrorHandle(hipFree(u_dev));

	//////////////////////////////
	// circular convolution OJO //
	//////////////////////////////

	// OJO_ijk = flip(flip(flip(OJO,1),2),3)
	// OJO_ijk = circshift(OJO_ijk,1,i)
	// OJO_ijk = circshift(OJO_ijk,2,j)
	// OJO_ijk = circshift(OJO_ijk,3,k)
	// dF{r,i,j,k,p} = Fold{r,m,n,l}.*OJO_ijk{m,n,l,p}
	// dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)
	// dF = sum(dF,'p')

	// set up arrays
	hipDoubleComplex* OJO_dev;
	cudaErrorHandle(hipMalloc(&OJO_dev, 3*size_F->nx*sizeof(hipDoubleComplex)));
	cudaErrorHandle(hipMemcpy(OJO_dev, OJO, 3*size_F->nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	hipDoubleComplex* OJO_ijk_dev;
	cudaErrorHandle(hipMalloc(&OJO_ijk_dev, 3*size_F->nx*sizeof(hipDoubleComplex)));

	// set up blocksize and gridsize
	dim3 blocksize_512_nR(512, 1, 1);
	dim3 gridsize_512_nR((int)size_F->nR_compact/512+1, 1, 1);

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			for (int k = 0; k < size_F->const_2Bx; k++) {
				flip_shift <<<gridsize_8, blocksize_8>>> (OJO_dev, OJO_ijk_dev, i, j, k, size_F_dev);
				cudaErrorHandle(hipGetLastError());

				hipDeviceSynchronize();

				cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, OJO_ijk_dev,
					(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

				double c[3];
				deriv_x(c, i, size_F->Bx, *L);
				deriv_x(c+1, j, size_F->Bx, *L);
				deriv_x(c+2, k, size_F->Bx, *L);

				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev, c[0], size_F_dev);
				cudaErrorHandle(hipGetLastError());
				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+size_F->nR_compact, c[1], size_F_dev);
				cudaErrorHandle(hipGetLastError());
				mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+2*size_F->nR_compact, c[2], size_F_dev);
				cudaErrorHandle(hipGetLastError());

				hipDeviceSynchronize();

				for (int ip = 0; ip < 3; ip++) {
					hipDoubleComplex* dF3_dev_ijkp = dF3_dev + i*size_F->nR_compact + 
						j*(size_F->nR_compact*size_F->const_2Bx) + k*(size_F->nR_compact*size_F->const_2Bxs) + ip*size_F->nTot_compact;
					hipDoubleComplex* dF_temp_dev_p = dF_temp_dev + ip*size_F->nR_compact;

					cudaErrorHandle(hipMemcpy(dF3_dev_ijkp, dF_temp_dev_p, size_F->nR_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));
				}

				hipDeviceSynchronize();
			}
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F_dev);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(OJO_dev));
	cudaErrorHandle(hipFree(OJO_ijk_dev));
	cudaErrorHandle(hipFree(dF_temp_dev));
	if (worksize_conv > 0)
		cudaErrorHandle(hipFree(work));

	///////////////////////
	// kronecker product //
	///////////////////////

	// set up arrays
	hipDoubleComplex** CG_dev = new hipDoubleComplex* [size_F->BR*size_F->BR];
	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		for (int l2 = 0; l2 <= size_F->lmax; l2++) {
			int m = (2*l1+1)*(2*l2+1);
			int ind_CG = l1+l2*size_F->BR;
			cudaErrorHandle(hipMalloc(&CG_dev[ind_CG], m*m*sizeof(hipDoubleComplex)));
			cudaErrorHandle(hipMemset(CG_dev[ind_CG], 0, m*m*sizeof(hipDoubleComplex)));

			double* CG_dev_d = (double*) CG_dev[ind_CG];
			cudaErrorHandle(hipMemcpy2D(CG_dev_d, 2*sizeof(double), CG[ind_CG], sizeof(double), sizeof(double), m*m, hipMemcpyHostToDevice));
		}
	}

	hipDoubleComplex** F_strided = new hipDoubleComplex* [size_F->BR];
	for (int l = 0; l <= size_F->lmax; l++) {
		int ind = l*(2*l-1)*(2*l+1)/3;
		int m = (2*l+1)*(2*l+1);
		cudaErrorHandle(hipMalloc(&F_strided[l], m*size_F->nx*sizeof(hipDoubleComplex)));
		cudaErrorHandle(hipMemcpy2D(F_strided[l], m*sizeof(hipDoubleComplex), F+ind, size_F->nR_compact*sizeof(hipDoubleComplex),
			m*sizeof(hipDoubleComplex), size_F->nx, hipMemcpyHostToDevice));
	}

	hipDoubleComplex* MR_dev;
	cudaErrorHandle(hipMalloc(&MR_dev, 3*size_F->nR_compact*sizeof(hipDoubleComplex)));
	cudaErrorHandle(hipMemcpy(MR_dev, MR, 3*size_F->nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

	hipDoubleComplex* FMR_dev;
	int m = (2*size_F->lmax+1) * (2*size_F->lmax+1);
	cudaErrorHandle(hipMalloc(&FMR_dev, 3*m*sizeof(hipDoubleComplex)));

	hipDoubleComplex* FMR_temp_dev;
	cudaErrorHandle(hipMalloc(&FMR_temp_dev, 3*size_F->nx*sizeof(hipDoubleComplex)));

	cudaErrorHandle(hipMemset(dF3_dev, 0, 3*size_F->nTot_compact*sizeof(hipDoubleComplex)));

	// get c
	double* c = new double[size_F->const_2Bx];
	for (int i = 0; i < size_F->const_2Bx; i++) {
		deriv_x(&c[i], i, size_F->Bx, *L);
	}

	double* c_dev;
	cudaErrorHandle(hipMalloc(&c_dev, size_F->const_2Bx*sizeof(double)));
	cudaErrorHandle(hipMemcpy(c_dev, c, size_F->const_2Bx*sizeof(double), hipMemcpyHostToDevice));

	// set up cutensor
	hiptensorContractionPlan_t* plan_FMR = new hiptensorContractionPlan_t [size_F->BR];
	size_t* worksize_FMR = new size_t [size_F->BR];

	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		cutensor_initFMR(&handle_cutensor, &plan_FMR[l1], &worksize_FMR[l1], F_strided[l1], FMR_dev, FMR_temp_dev, l1, size_F);
	}

	size_t worksize_FMR_max = 0;
	for (int l = 0; l <= size_F->lmax; l++) {
		worksize_FMR_max = (worksize_FMR[l] > worksize_FMR_max) ? worksize_FMR[l] : worksize_FMR_max;
	}

	if (worksize_FMR_max > 0) {
		cudaErrorHandle(hipMalloc(&work, worksize_FMR_max));
	}

	// set up blocksize and gridsize
	dim3 blocksize_addMFR(size_F->const_2Bx, size_F->const_2Bx, 1);
	dim3 gridsize_addMFR(size_F->const_2Bx, 3, 1);

	dim3 blocksize_deriv(512,1,1);
	dim3 gridsize_deriv((int)size_F->nR_compact/512+1, size_F->const_2Bx, size_F->const_2Bxs);

	// calculate
	for (int l = 0; l <= size_F->lmax; l++) {
		int ind_cumR = l*(2*l-1)*(2*l+1)/3;

		for (int l1 = 0; l1 <= size_F->lmax; l1++) {
			for (int l2 = 0; l2 <= size_F->lmax; l2++) {
				if (abs(l1-l2)<=l && l1+l2>=l) {
					int ind_MR = l2*(2*l2-1)*(2*l2+1)/3;
					int ind_CG = l1+l2*size_F->BR;
					int l12 = (2*l1+1)*(2*l2+1);

					alpha_cutensor.x = (double) -l12/(2*l+1);

					for (int m = -l; m <= l; m++) {
						int ind_CG_m = (l*l-(l1-l2)*(l1-l2)+m+l)*l12;

						for (int n = -l; n <= l; n++) {
							int ind_CG_n = (l*l-(l1-l2)*(l1-l2)+n+l)*l12;
							int ind_mnl = m+l + (n+l)*(2*l+1) + ind_cumR;

							cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, 2*l1+1, 2*l2+1, 2*l2+1,
								&alpha_cublas, CG_dev[ind_CG]+ind_CG_m, 2*l2+1, 0, MR_dev+ind_MR, 2*l2+1, size_F->nR_compact,
								&beta_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), 3));

							cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, 2*l1+1, 2*l1+1, 2*l2+1,
								&alpha_cublas, FMR_temp_dev, 2*l1+1, (2*l1+1)*(2*l2+1), CG_dev[ind_CG]+ind_CG_n, 2*l2+1, 0,
								&beta_cublas, FMR_dev, 2*l1+1, (2*l1+1)*(2*l1+1), 3));

							cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_FMR[l1], &alpha_cutensor, F_strided[l1],
								FMR_dev, &beta_cutensor, FMR_temp_dev, FMR_temp_dev, work, worksize_FMR[l1], 0));

							add_FMR <<<gridsize_addMFR, blocksize_addMFR>>> (dF3_dev, FMR_temp_dev, ind_mnl, size_F_dev);
						}
					}
				}
			}
		}
	}

	for (int i = 0; i < 3; i++) {
		mulImg_FTot <<<gridsize_deriv, blocksize_deriv>>> (dF3_dev+i*size_F->nTot_compact, c_dev, i, size_F_dev);
		cudaErrorHandle(hipGetLastError());
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF3_dev, size_F_dev);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF3_dev, size_F_dev);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(dF3_dev));
	cudaErrorHandle(hipFree(MR_dev));
	cudaErrorHandle(hipFree(FMR_dev));
	cudaErrorHandle(hipFree(FMR_temp_dev));
	cudaErrorHandle(hipFree(c_dev));
	cudaErrorHandle(hipFree(F_dev));

	if (worksize_FMR_max > 0) {
		cudaErrorHandle(hipFree(work));
	}

	for (int l1 = 0; l1 <= size_F->lmax; l1++) {
		for (int l2 = 0; l2 <= size_F->lmax; l2++) {
			int ind_CG = l1+l2*size_F->BR;
			cudaErrorHandle(hipFree(CG_dev[ind_CG]));
		}
	}

	for (int l = 0; l <= size_F->lmax; l++) {
		cudaErrorHandle(hipFree(F_strided[l]));
	}

	delete[] c;
	delete[] plan_FMR;
	delete[] worksize_FMR;
	delete[] CG_dev;
	delete[] F_strided;

	// return
	cudaErrorHandle(hipMemcpy(dF, dF_dev, size_F->nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

	cudaErrorHandle(hipFree(dF_dev));
}

