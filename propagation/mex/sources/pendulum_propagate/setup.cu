#include "setup.cuh"

#include <stdio.h>
#include <iostream>


__host__ void cudaErrorHandle(const hipError_t& err)
{
	if (err != hipSuccess) {
		std::cout << "Cuda Error: " << hipGetErrorString(err) << std::endl;
	}
}

__host__ void cutensorErrorHandle(const hiptensorStatus_t& err)
{
	if (err != HIPTENSOR_STATUS_SUCCESS) {
		std::cout << "cuTensor Error: " << hiptensorGetErrorString(err) << std::endl;
	}
}

__host__ void cublasErrorHandle(const hipblasStatus_t& err)
{
	if (err != HIPBLAS_STATUS_SUCCESS) {
		std::cout << "cuBlas Error: " << err << std::endl;
	}
}

__host__ void cutensor_initConv(hiptensorHandle_t* handle, hiptensorContractionPlan_t* plan, size_t* worksize,
	const void* Fold_dev, const void* X_dev, const void* dF_dev, const int nR_split, const bool issmall, const Size_F* size_F)
{
	int mode_Fold[4];
	int mode_X[4] = {'i','j','k','p'};
	int mode_dF[2] = {'r','p'};

	int64_t extent_Fold[4];
	int64_t extent_X[4] = {size_F->const_2Bx, size_F->const_2Bx, size_F->const_2Bx, 3};
	int64_t extent_dF[2] = {nR_split, 3};

	if (issmall) {
		mode_Fold[0] = 'r';
		mode_Fold[1] = 'i';
		mode_Fold[2] = 'j';
		mode_Fold[3] = 'k';

		extent_Fold[0] = nR_split;
		extent_Fold[1] = size_F->const_2Bx;
		extent_Fold[2] = size_F->const_2Bx;
		extent_Fold[3] = size_F->const_2Bx;
	} else {
		mode_Fold[0] = 'i';
		mode_Fold[1] = 'j';
		mode_Fold[2] = 'k';
		mode_Fold[3] = 'r';

		extent_Fold[0] = size_F->const_2Bx;
		extent_Fold[1] = size_F->const_2Bx;
		extent_Fold[2] = size_F->const_2Bx;
		extent_Fold[3] = nR_split;
	}

	hiptensorTensorDescriptor_t desc_Fold;
	hiptensorTensorDescriptor_t desc_X;
	hiptensorTensorDescriptor_t desc_dF;
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_Fold,
		4, extent_Fold, NULL, mycutensor_datatype, HIPTENSOR_OP_IDENTITY));
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_X,
		4, extent_X, NULL, mycutensor_datatype, HIPTENSOR_OP_IDENTITY));
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_dF,
		2, extent_dF, NULL, mycutensor_datatype, HIPTENSOR_OP_IDENTITY));

	uint32_t alignmentRequirement_Fold;
	uint32_t alignmentRequirement_X;
	uint32_t alignmentRequirement_temp;
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		Fold_dev, &desc_Fold, &alignmentRequirement_Fold));
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		X_dev, &desc_X, &alignmentRequirement_X));
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		dF_dev, &desc_dF, &alignmentRequirement_temp));

	cutensorContractionDescriptor_t desc;
	cutensorErrorHandle(cutensorInitContractionDescriptor(handle, &desc,
		&desc_Fold, mode_Fold, alignmentRequirement_Fold,
		&desc_X, mode_X, alignmentRequirement_X,
		&desc_dF, mode_dF, alignmentRequirement_temp,
		&desc_dF, mode_dF, alignmentRequirement_temp,
		mycutensor_computetype));

	cutensorContractionFind_t find;
	cutensorErrorHandle(cutensorInitContractionFind(handle, &find, HIPTENSOR_ALGO_DEFAULT));

	cutensorErrorHandle(cutensorContractionGetWorkspace(handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, worksize));

	cutensorErrorHandle(cutensorInitContractionPlan(handle, plan, &desc, &find, *worksize));
}

__host__ void cutensor_initFMR(hiptensorHandle_t* handle, hiptensorContractionPlan_t* plan, size_t* worksize,
	const void* Fold_dev, const void* MR_dev, const void* FMR_dev, const int l, const bool issmall, const Size_F* size_F)
{
	int mode_Fold[2] = {'r','x'};
	int mode_MR[2] = {'r','p'};
	int mode_FMR[2] = {'x','p'};

	int m = (2*l+1)*(2*l+1);

	int64_t extent_Fold[2];
	int64_t extent_MR[2] = {m, 3};
	int64_t extent_FMR[2];

	if (issmall) {
		extent_Fold[0] = m;
		extent_Fold[1] = size_F->nx;

		extent_FMR[0] = size_F->nx;
		extent_FMR[1] = 3;
	} else {
		extent_Fold[0] = m;
		extent_Fold[1] = size_F->const_2Bxs;

		extent_FMR[0] = size_F->const_2Bxs;
		extent_FMR[1] = 3;
	}

	hiptensorTensorDescriptor_t desc_Fold;
	hiptensorTensorDescriptor_t desc_MR;
	hiptensorTensorDescriptor_t desc_FMR;
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_Fold,
		2, extent_Fold, NULL, mycutensor_datatype, HIPTENSOR_OP_IDENTITY));
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_MR,
		2, extent_MR, NULL, mycutensor_datatype, HIPTENSOR_OP_IDENTITY));
	cutensorErrorHandle(hiptensorInitTensorDescriptor(handle, &desc_FMR,
		2, extent_FMR, NULL, mycutensor_datatype, HIPTENSOR_OP_IDENTITY));

	uint32_t alignmentRequirement_Fold;
	uint32_t alignmentRequirement_MR;
	uint32_t alignmentRequirement_FMR;
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		Fold_dev, &desc_Fold, &alignmentRequirement_Fold));
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		MR_dev, &desc_MR, &alignmentRequirement_MR));
	cutensorErrorHandle(cutensorGetAlignmentRequirement(handle,
		FMR_dev, &desc_FMR, &alignmentRequirement_FMR));

	cutensorContractionDescriptor_t desc;
	cutensorErrorHandle(cutensorInitContractionDescriptor(handle, &desc,
		&desc_Fold, mode_Fold, alignmentRequirement_Fold,
		&desc_MR, mode_MR, alignmentRequirement_MR,
		&desc_FMR, mode_FMR, alignmentRequirement_FMR,
		&desc_FMR, mode_FMR, alignmentRequirement_FMR,
		mycutensor_computetype));

	cutensorContractionFind_t find;
	cutensorErrorHandle(cutensorInitContractionFind(handle, &find, HIPTENSOR_ALGO_DEFAULT));

	cutensorErrorHandle(cutensorContractionGetWorkspace(handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, worksize));

	cutensorErrorHandle(cutensorInitContractionPlan(handle, plan, &desc, &find, *worksize));
}

__host__ void init_Size_F(Size_F* size_F, int BR, int Bx)
{
	size_F->BR = BR;
	size_F->Bx = Bx;
	size_F->lmax = BR-1;

	size_F->nR = (2*size_F->lmax+1) * (2*size_F->lmax+1) * (size_F->lmax+1);
	size_F->nx = (2*Bx) * (2*Bx) * (2*Bx);
	size_F->nTot = size_F->nR * size_F->nx;

	size_F->nR_compact = (size_F->lmax+1) * (2*size_F->lmax+1) * (2*size_F->lmax+3) / 3;
	size_F->nTot_compact = size_F->nR_compact * size_F->nx;

	size_F->const_2Bx = 2*Bx;
	size_F->const_2Bxs = (2*Bx) * (2*Bx);
	size_F->const_2lp1 = 2*size_F->lmax+1;
	size_F->const_lp1 = size_F->lmax+1;
	size_F->const_2lp1s = (2*size_F->lmax+1) * (2*size_F->lmax+1);

	size_F->l_cum0 = size_F->const_2lp1;
	size_F->l_cum1 = size_F->l_cum0*size_F->const_2lp1;
	size_F->l_cum2 = size_F->l_cum1*size_F->const_lp1;
	size_F->l_cum3 = size_F->l_cum2*size_F->const_2Bx;
	size_F->l_cum4 = size_F->l_cum3*size_F->const_2Bx;

	size_F->ns = size_F->const_2lp1;
	size_F->nR_split = (int) size_F->nR_compact / (size_F->ns-1);
	size_F->nR_remainder = size_F->nR_compact % (size_F->ns-1);

	size_F->nTot_splitx = size_F->nR_compact * size_F->const_2Bxs;
}

