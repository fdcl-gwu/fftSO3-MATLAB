
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

#include <string.h>

#include "mex.h"

void mexFunction (int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
    ////////////////////////////
    // get arrays from Matlab //
    ////////////////////////////

    // get Fold from matlab
    hipComplex* Fold = (hipComplex*) mxGetComplexSingles(prhs[0]);
    const mwSize* size_Fold = mxGetDimensions(prhs[0]);

    Size_F size_F;
    init_Size_F(&size_F, (int)size_Fold[2], (int)size_Fold[3]/2);

    hipComplex* Fold_compact = new hipComplex[size_F.nTot_compact];
    modify_F(Fold, Fold_compact, true, &size_F);

    Size_F* size_F_dev;
    cudaErrorHandle(hipMalloc(&size_F_dev, sizeof(Size_F)));
    cudaErrorHandle(hipMemcpy(size_F_dev, &size_F, sizeof(Size_F), hipMemcpyHostToDevice));

    // set up output Fnew
    plhs[0] = mxCreateUninitNumericArray(6, (size_t*) size_Fold, mxSINGLE_CLASS, mxCOMPLEX);
    hipComplex* Fnew = (hipComplex*) mxGetComplexSingles(plhs[0]);

    hipComplex* Fnew_compact = new hipComplex[size_F.nTot_compact];
    
    // get X from matlab
    hipComplex* X = (hipComplex*) mxGetComplexSingles(prhs[1]);

    // get OJO from matlab
    hipComplex* OJO = (hipComplex*) mxGetComplexSingles(prhs[2]);

    // get MR from matlab
    hipComplex* MR = (hipComplex*) mxGetComplexSingles(prhs[3]);

    hipComplex* MR_compact = new hipComplex[3*size_F.nR_compact];
    modify_u(MR, MR_compact, &size_F);

    // get dt from matlab
    float* dt = mxGetSingles(prhs[4]);

    // get L from matlab
    float* L = mxGetSingles(prhs[5]);

    // get u from matlab
    hipComplex* u = (hipComplex*) mxGetComplexSingles(prhs[6]);

    hipComplex* u_compact = new hipComplex[3*size_F.nR_compact];
    modify_u(u, u_compact, &size_F);

    // get CG from matlab
    float** CG = new float* [size_F.BR*size_F.BR];
    for (int l1 = 0; l1 < size_F.BR; l1++) {
        for (int l2 = 0; l2 < size_F.BR; l2++) {
            int ind_CG = l1+l2*size_F.BR;
            CG[ind_CG] = mxGetSingles(mxGetCell(prhs[7], ind_CG));
        }
    }
    
    // get method from matlab
    char* method;
    method = mxArrayToString(prhs[8]);

    //////////////////
    // calculate dF //
    //////////////////

    // if the problem is too large, split arrays
    bool issmall = (size_F.BR<=10 && size_F.Bx<=10);

    // set up arrays
    hipComplex* dF1;
    hipComplex* dF2;
    hipComplex* dF3;
    hipComplex* dF4;

    hipComplex* Fold_dev;

    if (issmall) {
        // set up arrays
        cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot_compact*sizeof(hipComplex)));
        cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));

        // set up blocksize and gridsize
        dim3 blocksize_512_nTot(512, 1, 1);
        dim3 gridsize_512_nTot((int)size_F.nTot_compact/512+1, 1, 1);

        // calculate
        // dF1
        dF1 = new hipComplex[size_F.nTot_compact];
        get_dF_small(dF1, Fold_compact, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

        if (stricmp(method,"midpoint") == 0 || stricmp(method,"runge-kutta") == 0) {
            // dF2
            hipComplex* F2_dev;
            cudaErrorHandle(hipMalloc(&F2_dev, size_F.nTot_compact*sizeof(hipComplex)));

            hipComplex* dF1_dev;
            cudaErrorHandle(hipMalloc(&dF1_dev, size_F.nTot_compact*sizeof(hipComplex)));
            cudaErrorHandle(hipMemcpy(dF1_dev, dF1, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));

            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F2_dev, Fold_dev, dF1_dev, dt[0]/2, size_F.nTot_compact);

            hipComplex* F2 = new hipComplex[size_F.nTot_compact];
            cudaErrorHandle(hipMemcpy(F2, F2_dev, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyDeviceToHost));

            dF2 = new hipComplex[size_F.nTot_compact];
            get_dF_small(dF2, F2, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

            delete[] F2;
            cudaErrorHandle(hipFree(F2_dev));
            cudaErrorHandle(hipFree(dF1_dev));
        }

        if (stricmp(method,"runge-kutta") == 0) {
            // dF3
            hipComplex* F3_dev;
            cudaErrorHandle(hipMalloc(&F3_dev, size_F.nTot_compact*sizeof(hipComplex)));

            hipComplex* dF2_dev;
            cudaErrorHandle(hipMalloc(&dF2_dev, size_F.nTot_compact*sizeof(hipComplex)));
            cudaErrorHandle(hipMemcpy(dF2_dev, dF2, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));

            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F3_dev, Fold_dev, dF2_dev, dt[0]/2, size_F.nTot_compact);

            hipComplex* F3 = new hipComplex[size_F.nTot_compact];
            cudaErrorHandle(hipMemcpy(F3, F3_dev, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyDeviceToHost));

            dF3 = new hipComplex[size_F.nTot_compact];
            get_dF_small(dF3, F3, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

            delete[] F3;
            cudaErrorHandle(hipFree(F3_dev));
            cudaErrorHandle(hipFree(dF2_dev));

            // dF4
            hipComplex* F4_dev;
            cudaErrorHandle(hipMalloc(&F4_dev, size_F.nTot_compact*sizeof(hipComplex)));

            hipComplex* dF3_dev;
            cudaErrorHandle(hipMalloc(&dF3_dev, size_F.nTot_compact*sizeof(hipComplex)));
            cudaErrorHandle(hipMemcpy(dF3_dev, dF3, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));

            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F4_dev, Fold_dev, dF3_dev, dt[0], size_F.nTot_compact);

            hipComplex* F4 = new hipComplex[size_F.nTot_compact];
            cudaErrorHandle(hipMemcpy(F4, F4_dev, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyDeviceToHost));

            dF4 = new hipComplex[size_F.nTot_compact];
            get_dF_small(dF4, F4, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

            delete[] F4;
            cudaErrorHandle(hipFree(F4_dev));
            cudaErrorHandle(hipFree(dF3_dev));
        }
    } else {
        // set up arrays
        cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot_splitx*sizeof(hipComplex)));

        // set up blocksize and gridsize
        dim3 blocksize_512_nTot(512, 1, 1);
        dim3 gridsize_512_nTot((int)size_F.nTot_splitx/512+1, 1, 1);
    
        // calculate
        // dF1
        dF1 = new hipComplex[size_F.nTot_compact];
        get_dF_large(dF1, Fold_compact, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

        if (stricmp(method,"midpoint") == 0 || stricmp(method,"runge-kutta") == 0) {
            // dF2
            hipComplex* F2_dev;
            cudaErrorHandle(hipMalloc(&F2_dev, size_F.nTot_splitx*sizeof(hipComplex)));

            hipComplex* dF1_dev;
            cudaErrorHandle(hipMalloc(&dF1_dev, size_F.nTot_splitx*sizeof(hipComplex)));

            hipComplex* F2 = new hipComplex[size_F.nTot_compact];

            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF1_dev, dF1+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F2_dev, Fold_dev, dF1_dev, dt[0]/2, size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(F2+ind_F, F2_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            dF2 = new hipComplex[size_F.nTot_compact];
            get_dF_large(dF2, F2, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

            delete[] F2;
            cudaErrorHandle(hipFree(F2_dev));
            cudaErrorHandle(hipFree(dF1_dev));
        }

        if (stricmp(method,"runge-kutta") == 0) {
            // dF3
            hipComplex* F3_dev;
            cudaErrorHandle(hipMalloc(&F3_dev, size_F.nTot_splitx*sizeof(hipComplex)));

            hipComplex* dF2_dev;
            cudaErrorHandle(hipMalloc(&dF2_dev, size_F.nTot_splitx*sizeof(hipComplex)));

            hipComplex* F3 = new hipComplex[size_F.nTot_compact];

            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF2_dev, dF2+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F3_dev, Fold_dev, dF2_dev, dt[0]/2, size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(F3+ind_F, F3_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            dF3 = new hipComplex[size_F.nTot_compact];
            get_dF_large(dF3, F3, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

            delete[] F3;
            cudaErrorHandle(hipFree(F3_dev));
            cudaErrorHandle(hipFree(dF2_dev));

            // dF4
            hipComplex* F4_dev;
            cudaErrorHandle(hipMalloc(&F4_dev, size_F.nTot_splitx*sizeof(hipComplex)));

            hipComplex* dF3_dev;
            cudaErrorHandle(hipMalloc(&dF3_dev, size_F.nTot_splitx*sizeof(hipComplex)));

            hipComplex* F4 = new hipComplex[size_F.nTot_compact];

            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF3_dev, dF3+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (F4_dev, Fold_dev, dF3_dev, dt[0], size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(F4+ind_F, F4_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            dF4 = new hipComplex[size_F.nTot_compact];
            get_dF_large(dF4, F4, X, OJO, MR_compact, L, u_compact, CG, &size_F, size_F_dev);

            delete[] F4;
            cudaErrorHandle(hipFree(F4_dev));
            cudaErrorHandle(hipFree(dF3_dev));
        }
    }

    // free memory
    delete[] MR_compact;
    delete[] u_compact;
    delete[] CG;

    ///////////////
    // integrate //
    ///////////////

    // Fnew = Fold + dt*dF1 (euler)
    // Fnew = Fold + dt*dF2 (midpoint)
    // Fnew = Fold + dt/3*dF1 + dt/6*dF2 + dt/6*dF3 + dt/3*dF4 (runge-kutta)

    // set up GPU arrays
    hipComplex* Fnew_dev;
    hipComplex* dF_dev;

    // calculate
    if (issmall) {
        // set up arrays
        cudaErrorHandle(hipMalloc(&Fnew_dev, size_F.nTot_compact*sizeof(hipComplex)));
        cudaErrorHandle(hipMalloc(&dF_dev, size_F.nTot_compact*sizeof(hipComplex)));

        // set up blocksize and gridsize
        dim3 blocksize_512_nTot(512, 1, 1);
        dim3 gridsize_512_nTot((int)size_F.nTot_compact/512+1, 1, 1);

        // calculate
        if (stricmp(method,"euler") == 0) {
            cudaErrorHandle(hipMemcpy(dF_dev, dF1, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));
            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F.nTot_compact);

            delete[] dF1;
        } else if (stricmp(method,"midpoint") == 0) {
            cudaErrorHandle(hipMemcpy(dF_dev, dF2, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));
            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F.nTot_compact);

            delete[] dF1;
            delete[] dF2;
        } else if (stricmp(method,"runge-kutta") == 0) {
            cudaErrorHandle(hipMemcpy(dF_dev, dF1, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));
            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0]/6, size_F.nTot_compact);

            cudaErrorHandle(hipMemcpy(dF_dev, dF2, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));
            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fnew_dev, dF_dev, dt[0]/3, size_F.nTot_compact);

            cudaErrorHandle(hipMemcpy(dF_dev, dF3, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));
            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fnew_dev, dF_dev, dt[0]/3, size_F.nTot_compact);

            cudaErrorHandle(hipMemcpy(dF_dev, dF4, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyHostToDevice));
            integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fnew_dev, dF_dev, dt[0]/6, size_F.nTot_compact);

            delete[] dF1;
            delete[] dF2;
            delete[] dF3;
            delete[] dF4;
        } else {
            mexPrintf("'method' must be 'euler', 'midpoint', or 'runge-kutta'. Return Fold.\n");
            Fnew_dev = Fold_dev;
        }

        cudaErrorHandle(hipMemcpy(Fnew_compact, Fnew_dev, size_F.nTot_compact*sizeof(hipComplex), hipMemcpyDeviceToHost));
    } else {
        // set up arrays
        cudaErrorHandle(hipMalloc(&Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex)));
        cudaErrorHandle(hipMalloc(&dF_dev, size_F.nTot_splitx*sizeof(hipComplex)));

        // set up blocksize and gridsize
        dim3 blocksize_512_nTot(512, 1, 1);
        dim3 gridsize_512_nTot((int)size_F.nTot_splitx/512+1, 1, 1);

        // calculate
        if (stricmp(method,"euler") == 0) {
            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF_dev, dF1+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            delete[] dF1;
        } else if (stricmp(method,"midpoint") == 0) {
            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF_dev, dF2+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0], size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            delete[] dF1;
            delete[] dF2;
        } else if (stricmp(method,"runge-kutta") == 0) {
            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF_dev, dF1+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0]/6, size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fnew_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF_dev, dF2+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0]/3, size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fnew_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF_dev, dF3+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0]/3, size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            for (int k = 0; k < size_F.const_2Bx; k++) {
                int ind_F = k*size_F.nTot_splitx;
                cudaErrorHandle(hipMemcpy(Fold_dev, Fnew_compact+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));
                cudaErrorHandle(hipMemcpy(dF_dev, dF4+ind_F, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyHostToDevice));

                integrate_Fnew <<<gridsize_512_nTot, blocksize_512_nTot>>> (Fnew_dev, Fold_dev, dF_dev, dt[0]/6, size_F.nTot_splitx);
                cudaErrorHandle(hipGetLastError());

                cudaErrorHandle(hipMemcpy(Fnew_compact+ind_F, Fnew_dev, size_F.nTot_splitx*sizeof(hipComplex), hipMemcpyDeviceToHost));
            }

            delete[] dF1;
            delete[] dF2;
            delete[] dF3;
            delete[] dF4;
        } else {
            mexPrintf("'method' must be 'euler', 'midpoint', or 'runge-kutta'. Return Fold.\n");
            Fnew_dev = Fold_dev;
        }
    }

    // gather Fnew
    modify_F(Fnew_compact, Fnew, false, &size_F);

    // free memory
    cudaErrorHandle(hipFree(Fold_dev));
    cudaErrorHandle(hipFree(Fnew_dev));
    cudaErrorHandle(hipFree(dF_dev));
    cudaErrorHandle(hipFree(size_F_dev));

    delete[] Fold_compact;
    delete[] Fnew_compact;
}

