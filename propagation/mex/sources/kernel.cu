#include "hip/hip_runtime.h"

#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

#include <math.h>

#include "mex.h"

void mexFunction (int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
    ///////////////////
    // set up arrays //
    ///////////////////

    // get Fold from matlab
    hipDoubleComplex* Fold = (hipDoubleComplex*) mxGetComplexDoubles(prhs[0]);
    const mwSize* size_Fold = mxGetDimensions(prhs[0]);

    Size_F size_F;
    init_Size_F(&size_F, (int)size_Fold[2], (int)size_Fold[3]/2);

    hipDoubleComplex* Fold_compact = new hipDoubleComplex[size_F.nTot_compact];
    modify_F(Fold, Fold_compact, true, &size_F);

    hipDoubleComplex* Fold_dev;
    cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(Fold_dev, Fold_compact, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    Size_F* size_F_dev;
    cudaErrorHandle(hipMalloc(&size_F_dev, sizeof(size_F)));
    cudaErrorHandle(hipMemcpy(size_F_dev, &size_F, sizeof(size_F), hipMemcpyHostToDevice));

    // set up output Fnew
    plhs[0] = mxCreateUninitNumericArray(6, (size_t*) size_Fold, mxDOUBLE_CLASS, mxCOMPLEX);
    hipDoubleComplex* Fnew = (hipDoubleComplex*) mxGetComplexDoubles(plhs[0]);

    hipDoubleComplex* Fnew_compact = new hipDoubleComplex[size_F.nTot_compact];

    hipDoubleComplex* Fnew_dev;
    cudaErrorHandle(hipMalloc(&Fnew_dev, 3*size_F.nTot_compact*sizeof(hipDoubleComplex)));

    hipDoubleComplex* Fnew_buffer_dev;
    cudaErrorHandle(hipMalloc(&Fnew_buffer_dev, 3*size_F.nTot_compact*sizeof(hipDoubleComplex)));
    
    // get X from matlab
    hipDoubleComplex* X = (hipDoubleComplex*) mxGetComplexDoubles(prhs[1]);

    hipDoubleComplex* X_dev;
    cudaErrorHandle(hipMalloc(&X_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F.nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // get dt from matlab
    double* dt = mxGetDoubles(prhs[2]);

    // get u from matlab
    hipDoubleComplex* u = (hipDoubleComplex*) mxGetComplexDoubles(prhs[3]);

    hipDoubleComplex* u_compact = new hipDoubleComplex[3*size_F.nR_compact];
    modify_u(u, u_compact, &size_F);

    hipDoubleComplex* u_dev;
    cudaErrorHandle(hipMalloc(&u_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(u_dev, u_compact, 3*size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // set up temporary variables for calculation
    hipDoubleComplex* temp_dev;
    cudaErrorHandle(hipMalloc(&temp_dev, 3*size_F.nR_compact*sizeof(hipDoubleComplex)));

    hipDoubleComplex* X_ijk_dev;
    cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));

    /////////////////////////////////
    // set up block and grid sizes //
    /////////////////////////////////

    // flip the circshift X
    dim3 blocksize_X(8, 8, 8);
    int gridnum_X = ceil((double) size_F.const_2Bx/8);
    dim3 gridsize_X(gridnum_X, gridnum_X, gridnum_X);

    // dddup_F
    dim3 blocksize_add(512, 1, 1);
    dim3 gridsize_add(ceil((double) size_F.nR_compact/512), size_F.nx, 1);

    ////////////////////
    // set up tensors //
    ////////////////////
    int mode_Fold[4] = {'r','i','j','k'};
    int mode_X[4] = {'i','j','k','p'};
    int mode_temp[2] = {'r','p'};

    int64_t extent_Fold[4] = {size_F.nR_compact, size_F.const_2Bx, size_F.const_2Bx, size_F.const_2Bx};
    int64_t extent_X[4] = {size_F.const_2Bx, size_F.const_2Bx, size_F.const_2Bx, 3};
    int64_t extent_temp[2] = {size_F.nR_compact, 3};

    hiptensorHandle_t handle_cutensor;
    cutensorInit(&handle_cutensor);

    hiptensorTensorDescriptor_t desc_Fold;
    hiptensorTensorDescriptor_t desc_X;
    hiptensorTensorDescriptor_t desc_temp;
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle_cutensor, &desc_Fold,
        4, extent_Fold, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle_cutensor, &desc_X,
        4, extent_X, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle_cutensor, &desc_temp,
        2, extent_temp, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));

    uint32_t alignmentRequirement_Fold;
    uint32_t alignmentRequirement_X;
    uint32_t alignmentRequirement_temp;
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle_cutensor,
        Fold_dev, &desc_Fold, &alignmentRequirement_Fold));
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle_cutensor,
        X_dev, &desc_X, &alignmentRequirement_X));
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle_cutensor,
        temp_dev, &desc_temp, &alignmentRequirement_temp));

    cutensorContractionDescriptor_t desc;
    cutensorErrorHandle(cutensorInitContractionDescriptor(&handle_cutensor, &desc,
        &desc_Fold, mode_Fold, alignmentRequirement_Fold,
        &desc_X, mode_X, alignmentRequirement_X,
        &desc_temp, mode_temp, alignmentRequirement_temp,
        &desc_temp, mode_temp, alignmentRequirement_temp,
        HIPTENSOR_COMPUTE_DESC_32F));

    cutensorContractionFind_t find;
    cutensorErrorHandle(cutensorInitContractionFind(&handle_cutensor, &find, HIPTENSOR_ALGO_DEFAULT));

    size_t worksize = 0;
    cutensorErrorHandle(cutensorContractionGetWorkspace(&handle_cutensor, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize));

    void* work = nullptr;
    if (worksize > 0) {
        cudaErrorHandle(hipMalloc(&work, worksize));
    }

    hiptensorContractionPlan_t plan;
    cutensorErrorHandle(cutensorInitContractionPlan(&handle_cutensor, &plan, &desc, &find, worksize));

    hipDoubleComplex alpha = make_hipDoubleComplex((double)1/size_F.nx,0);
    hipDoubleComplex beta = make_hipDoubleComplex(0,0);

    ///////////////////
    // set up cublas //
    ///////////////////

    hipblasHandle_t handle_cublas;
    hipblasCreate(&handle_cublas);

    hipDoubleComplex alpha_cublas = make_hipDoubleComplex(-1,0);

    ///////////////
    // calculate //
    ///////////////

    // circular convolution
    for (int i = 0; i < size_F.const_2Bx; i++) {
        for (int j = 0; j < size_F.const_2Bx; j++) {
            for (int k = 0; k < size_F.const_2Bx; k++) {

                flip_shift <<<gridsize_X, blocksize_X>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
                cudaErrorHandle(hipGetLastError());

                hipDeviceSynchronize();

                cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan, (void*)&alpha, Fold_dev, X_ijk_dev,
                    (void*)&beta, temp_dev, temp_dev, work, worksize, 0));

                hipDeviceSynchronize();

                for (int n = 0; n < 3; n++) {
                    hipDoubleComplex* Fnew_dev_ijkn = Fnew_dev + i*size_F.nR_compact + 
                        j*(size_F.nR_compact*size_F.const_2Bx) + k*(size_F.nR_compact*size_F.const_2Bxs) + n*size_F.nTot_compact;
                    hipDoubleComplex* temp_dev_n = temp_dev + n*size_F.nR_compact;

                    cudaErrorHandle(hipMemcpy(Fnew_dev_ijkn, temp_dev_n, size_F.nR_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));
                }

                hipDeviceSynchronize();
            }
        }
    }

    // multiply u
    for (int i = 0; i < 3; i++) {
        for (int l = 0; l <= size_F.lmax; l++)
        {
            int ind_Fnew = l*(2*l-1)*(2*l+1)/3 + i*size_F.nTot_compact;
            long long int stride_Fnew = size_F.nR_compact;

            int ind_u = l*(2*l-1)*(2*l+1)/3 + i*size_F.nR_compact;
            long long int stride_u = 0;

            cublasErrorHandle(hipblasZgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
                &alpha_cublas, Fnew_dev+ind_Fnew, 2*l+1, stride_Fnew,
                u_dev+ind_u, 2*l+1, stride_u,
                &beta, Fnew_dev+ind_Fnew, 2*l+1, stride_Fnew, size_F.nx));

            hipDeviceSynchronize();
        }
    }

    // addup F
    addup_F <<<gridsize_add, blocksize_add>>> (Fnew_dev, Fold_dev, dt[0], size_F_dev);
    cudaErrorHandle(hipGetLastError());

    hipDeviceSynchronize();

    // gather Fnew
    cudaErrorHandle(hipMemcpy(Fnew_compact, Fnew_dev, size_F.nTot_compact*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    modify_F(Fnew_compact, Fnew, false, &size_F);

    hipFree(Fold_dev);
    hipFree(Fnew_dev);
    hipFree(Fnew_buffer_dev);
    hipFree(X_dev);
    hipFree(X_ijk_dev);
    hipFree(u_dev);

    delete[] Fold_compact;
    delete[] Fnew_compact;
}

