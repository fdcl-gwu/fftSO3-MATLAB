#include "hip/hip_runtime.h"

#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

#include <math.h>

#include "mex.h"

void mexFunction (int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
    ///////////////////
    // set up arrays //
    ///////////////////

    // get Fold from matlab
    hipDoubleComplex* Fold = (hipDoubleComplex*) mxGetComplexDoubles(prhs[0]);
    const mwSize* size_Fold = mxGetDimensions(prhs[0]);

    Size_F size_F;
    init_Size_F(&size_F, (int)size_Fold[2], (int)size_Fold[3]/2);

    hipDoubleComplex* Fold_dev;
    cudaErrorHandle(hipMalloc(&Fold_dev, size_F.nTot*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(Fold_dev, Fold, size_F.nTot*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    Size_F* size_F_dev;
    cudaErrorHandle(hipMalloc(&size_F_dev, sizeof(size_F)));
    cudaErrorHandle(hipMemcpy(size_F_dev, &size_F, sizeof(size_F), hipMemcpyHostToDevice));

    // set up output Fnew
    plhs[0] = mxCreateUninitNumericArray(6, (size_t*) size_Fold, mxDOUBLE_CLASS, mxCOMPLEX);
    hipDoubleComplex* Fnew = (hipDoubleComplex*) mxGetComplexDoubles(plhs[0]);

    hipDoubleComplex* Fnew_dev;
    cudaErrorHandle(hipMalloc(&Fnew_dev, size_F.nTot*sizeof(hipDoubleComplex)));
    
    // get X from matlab
    hipDoubleComplex* X = (hipDoubleComplex*) mxGetComplexDoubles(prhs[1]);

    hipDoubleComplex* X_dev;
    cudaErrorHandle(hipMalloc(&X_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(X_dev, X, 3*size_F.nx*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // get dt from matlab
    double* dt = mxGetDoubles(prhs[2]);

    // get u from matlab
    hipDoubleComplex* u = (hipDoubleComplex*) mxGetComplexDoubles(prhs[3]);

    hipDoubleComplex* u_dev;
    cudaErrorHandle(hipMalloc(&u_dev, 3*size_F.nR*sizeof(hipDoubleComplex)));
    cudaErrorHandle(hipMemcpy(u_dev, u, 3*size_F.nR*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // set up temporary variables for calculation
    hipDoubleComplex* temp_dev;
    cudaErrorHandle(hipMalloc(&temp_dev, 3*size_F.nR*sizeof(hipDoubleComplex)));

    hipDoubleComplex* X_ijk_dev;
    cudaErrorHandle(hipMalloc(&X_ijk_dev, 3*size_F.nx*sizeof(hipDoubleComplex)));

    /////////////////////////////////
    // set up block and grid sizes //
    /////////////////////////////////

    // flip the circshift X
    dim3 blocksize_X(8, 8, 8);
    int gridnum_X = ceil((double) size_F.const_2Bx/8);
    dim3 gridsize_X(gridnum_X, gridnum_X, gridnum_X);

    // multiply u
    dim3 blocksize_u(size_F.const_2lp1, size_F.const_2lp1, 1);
    dim3 gridsize_u(size_F.const_lp1, 1, 1);

    // add dF
    dim3 blocksize_add(512,1,1);
    dim3 gridsize_add(ceil((double) size_F.nTot/512),1,1);

    ////////////////////
    // set up tensors //
    ////////////////////
    int mode_Fold[6] = {'m','n','l','i','j','k'};
    int mode_X[4] = {'i','j','k','p'};
    int mode_temp[4] = {'m','n','l','p'};

    int64_t extent_Fold[6] = {size_F.const_2lp1, size_F.const_2lp1, size_F.const_lp1, size_F.const_2Bx, size_F.const_2Bx, size_F.const_2Bx};
    int64_t extent_X[4] = {size_F.const_2Bx, size_F.const_2Bx, size_F.const_2Bx, 3};
    int64_t extent_temp[4] = {size_F.const_2lp1, size_F.const_2lp1, size_F.const_lp1, 3};

    hiptensorHandle_t handle;
    cutensorInit(&handle);

    hiptensorTensorDescriptor_t desc_Fold;
    hiptensorTensorDescriptor_t desc_X;
    hiptensorTensorDescriptor_t desc_temp;
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle, &desc_Fold,
        6, extent_Fold, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle, &desc_X,
        4, extent_X, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle, &desc_temp,
        4, extent_temp, NULL, HIP_C_64F, HIPTENSOR_OP_IDENTITY));

    uint32_t alignmentRequirement_Fold;
    uint32_t alignmentRequirement_X;
    uint32_t alignmentRequirement_temp;
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle,
        Fold_dev, &desc_Fold, &alignmentRequirement_Fold));
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle,
        X_dev, &desc_X, &alignmentRequirement_X));
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle,
        temp_dev, &desc_temp, &alignmentRequirement_temp));

    cutensorContractionDescriptor_t desc;
    cutensorErrorHandle(cutensorInitContractionDescriptor(&handle, &desc,
        &desc_Fold, mode_Fold, alignmentRequirement_Fold,
        &desc_X, mode_X, alignmentRequirement_X,
        &desc_temp, mode_temp, alignmentRequirement_temp,
        &desc_temp, mode_temp, alignmentRequirement_temp,
        HIPTENSOR_COMPUTE_DESC_32F));

    cutensorContractionFind_t find;
    cutensorErrorHandle(cutensorInitContractionFind(&handle, &find, HIPTENSOR_ALGO_DEFAULT));

    size_t worksize = 0;
    cutensorErrorHandle(cutensorContractionGetWorkspace(&handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize));

    void* work = nullptr;
    if (worksize > 0) {
        cudaErrorHandle(hipMalloc(&work, worksize));
    }

    hiptensorContractionPlan_t plan;
    cutensorErrorHandle(cutensorInitContractionPlan(&handle, &plan, &desc, &find, worksize));

    hipDoubleComplex alpha = make_hipDoubleComplex((double)1/size_F.nx,0);
    hipDoubleComplex beta = make_hipDoubleComplex(0,0);

    ///////////////
    // calculate //
    ///////////////

    for (int i = 0; i < size_F.const_2Bx; i++) {
        for (int j = 0; j < size_F.const_2Bx; j++) {
            for (int k = 0; k < size_F.const_2Bx; k++) {

                flip_shift <<<gridsize_X, blocksize_X>>> (X_dev, X_ijk_dev, i, j, k, size_F_dev);
                cudaErrorHandle(hipGetLastError());

                cutensorErrorHandle(hiptensorContraction(&handle, &plan, (void*)&alpha, Fold_dev, X_ijk_dev,
                    (void*)&beta, temp_dev, temp_dev, work, worksize, 0));

                hipDeviceSynchronize();

                derivate <<<gridsize_u, blocksize_u>>> (temp_dev, u_dev, Fnew_dev, i, j, k, size_F_dev);
                cudaErrorHandle(hipGetLastError());

                hipDeviceSynchronize();
            }
        }
    }

    add_dF <<<gridsize_add, blocksize_add>>> (Fnew_dev, Fold_dev, dt[0], size_F_dev);
    cudaErrorHandle(hipGetLastError());

    cudaErrorHandle(hipMemcpy(Fnew, Fnew_dev, size_F.nTot*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    hipFree(Fold_dev);
    hipFree(Fnew_dev);
    hipFree(X_dev);
    hipFree(X_ijk_dev);
    hipFree(u_dev);
}

